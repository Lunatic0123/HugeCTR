#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "HugeCTR/include/optimizers/nesterov_optimizer.hpp"

namespace HugeCTR {

namespace {

template <typename T>
__global__ void nesterov_kernel(int len, float* weight, const T* wgrad, float* accum, 
                                float lr, float mu, float scaler) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < len) {
    float accum_old = accum[i];
    float accum_new = mu * accum_old - lr * static_cast<float>(wgrad[i]) / scaler;
    accum[i] = accum_new;
    weight[i] += (-mu * accum_old + (1 + mu) * accum_new);
  }
}

}  // namespace

template <typename T>
NesterovOptimizer<T>::NesterovOptimizer(const std::shared_ptr<GeneralBuffer<float>>& weight,
                                     const std::shared_ptr<GeneralBuffer<T>>& wgrad,
                                     int device_id,
                                     float learning_rate, float momentum_factor,
                                     float scaler)
    : Optimizer(weight, device_id, learning_rate, scaler),
      accum_(weight->get_num_elements(), device_id),
      mu_(momentum_factor),
      wgrad_(wgrad) {
  accum_.reset_sync();
  if (weight_->get_num_elements() != wgrad_->get_num_elements()) {
    CK_THROW_(Error_t::WrongInput, "weight_ and wgrad_ have different lengths");
  }
}

template <typename T>
void NesterovOptimizer<T>::update(hipStream_t stream) {
  CudaDeviceContext context(device_id_);

  const int len = weight_->get_num_elements();
  const int block_dim = 256;
  const int grid_dim = (len - 1) / block_dim + 1;

  float* weight = weight_->get_ptr_with_offset(0);
  const T* wgrad = wgrad_->get_ptr_with_offset(0);
  float* accum = accum_.get_ptr_with_offset(0);

  nesterov_kernel<<<grid_dim, block_dim, 0, stream>>>(len, weight, wgrad, accum,
                                                      lr_, mu_, scaler_);

#ifndef NDEBUG
  hipDeviceSynchronize();
  CK_CUDA_THROW_(hipGetLastError());
#endif
}

template class NesterovOptimizer<float>;
template class NesterovOptimizer<__half>;

}  // namespace HugeCTR
