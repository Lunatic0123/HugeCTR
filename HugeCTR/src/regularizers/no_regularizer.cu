#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <regularizers/no_regularizer.hpp>
#include <utils.cuh>
#include <utility>

#ifndef NDEBUG
#include <iostream>
#endif

namespace HugeCTR {
template <typename T>
NoRegularizer<T>::NoRegularizer(const std::shared_ptr<GeneralBuffer<float>>& weight_buff,
                                const std::shared_ptr<GeneralBuffer<T>>& wgrad_buff,
                                const int batch_size, const int device_id)
    : Regularizer<T>(weight_buff, wgrad_buff, batch_size, device_id) {}

template <typename T>
void NoRegularizer<T>::do_compute_rterm(const float* weight, float* rterm, int num_elements,
                                        hipStream_t stream) {
  *rterm = 0.0f;
}

template <typename T>
void NoRegularizer<T>::do_initialize_wgrad(const float* weight, T* wgrad, int num_elements,
                                           hipStream_t stream) {
  int n_blocks = Regularizer<T>::get_n_sms() * 4;
  int block_size = 512;
  initialize_array<<<n_blocks, block_size, 0, stream>>>(wgrad, num_elements, T(0.0f));
}

template class NoRegularizer<__half>;
template class NoRegularizer<float>;

}  // namespace HugeCTR
