#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "HugeCTR/include/layers/fully_connected_layer_half.hpp"
#include "HugeCTR/include/utils.cuh"

namespace HugeCTR {

FullyConnectedLayerHalf::FullyConnectedLayerHalf(const GeneralBufferPtr<float>& master_weights_buff,
                                                 const GeneralBufferPtr<__half>& weights_buff,
                                                 const GeneralBufferPtr<__half>& weights_grad_buff,
                                                 const GeneralBufferPtr<__half>& blobs_buff,
                                                 const TensorPtr<__half>& bottom_tensor,
                                                 const TensorPtr<__half>& top_tensor,
                                                 TensorFormat_t weight_tensor_format,
                                                 hipblasHandle_t const& cublas_handle, int device_id,
                                                 std::vector<Initializer_t> initializer_types)
    : Layer(device_id, initializer_types),
      cublas_handle_(cublas_handle),
      falgo_b_(CUBLAS_GEMM_DEFAULT_TENSOR_OP),
      falgo_k_(CUBLAS_GEMM_DEFAULT_TENSOR_OP),
      balgo_b_(CUBLAS_GEMM_DEFAULT_TENSOR_OP),
      balgo_k_(CUBLAS_GEMM_DEFAULT_TENSOR_OP),
      balgo_x_(CUBLAS_GEMM_DEFAULT_TENSOR_OP) {
  const auto& bottom_tensor_dim = bottom_tensor->get_dims();
  const auto& top_tensor_dim = top_tensor->get_dims();

  if (bottom_tensor_dim.size() != 2 || top_tensor_dim.size() != 2) {
    CK_THROW_(Error_t::WrongInput, "input or output tensor doesn't has two dimensions");
  }

  assert(weight_tensor_format == TensorFormat_t::HW);
  assert(bottom_tensor->get_format() == TensorFormat_t::HW);
  assert(top_tensor->get_format() == TensorFormat_t::HW);

  size_t m = bottom_tensor_dim[0];
  size_t n = top_tensor_dim[1];
  size_t k = bottom_tensor_dim[1];

  std::vector<size_t> kernel_dim = {k, n};
  std::vector<size_t> bias_dim = {1, n};
  std::vector<size_t> identity_dim = {1, m};

  weights_.emplace_back(new Tensor<float>(kernel_dim, master_weights_buff, weight_tensor_format));
  weights_.emplace_back(new Tensor<float>(bias_dim, master_weights_buff, weight_tensor_format));

  weights_half_.emplace_back(new Tensor<__half>(kernel_dim, weights_buff, weight_tensor_format));
  weights_half_.emplace_back(new Tensor<__half>(bias_dim, weights_buff, weight_tensor_format));

  weights_grad_.emplace_back(
      new Tensor<__half>(kernel_dim, weights_grad_buff, weight_tensor_format));
  weights_grad_.emplace_back(new Tensor<__half>(bias_dim, weights_grad_buff, weight_tensor_format));

  identity_tensor_.reset(new Tensor<__half>(identity_dim, blobs_buff, TensorFormat_t::HW));

  bottom_tensor_ = bottom_tensor;
  top_tensor_ = top_tensor;
}

void FullyConnectedLayerHalf::fprop(hipStream_t stream) {
  CudaDeviceContext context(get_device_id());
  CK_CUBLAS_THROW_(hipblasSetStream(cublas_handle_, stream));

  const __half* kernel = weights_half_[0]->get_ptr();
  const __half* bias = weights_half_[1]->get_ptr();
  const __half* bottom = bottom_tensor_->get_ptr();
  const __half* identity = identity_tensor_->get_ptr();
  __half* top = top_tensor_->get_ptr();

  const auto& bottom_tensor_dim = bottom_tensor_->get_dims();
  const auto& top_tensor_dim = top_tensor_->get_dims();

  size_t m = bottom_tensor_dim[0];
  size_t n = top_tensor_dim[1];
  size_t k = bottom_tensor_dim[1];

  const float alpha = 1.0f;
  const float beta_b = 0.0f;
  const float beta_k = 1.0f;

  CK_CUBLAS_THROW_(hipblasGemmEx(cublas_handle_, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, 1, &alpha, bias,
                                HIP_R_16F, n, identity, HIP_R_16F, 1, &beta_b, top, HIP_R_16F, n,
                                HIP_R_32F, falgo_b_));

  CK_CUBLAS_THROW_(hipblasGemmEx(cublas_handle_, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, &alpha, kernel,
                                HIP_R_16F, n, bottom, HIP_R_16F, k, &beta_k, top, HIP_R_16F, n,
                                HIP_R_32F, falgo_k_));

#ifndef NDEBUG
  hipDeviceSynchronize();
  CK_CUDA_THROW_(hipGetLastError());
#endif
}

void FullyConnectedLayerHalf::bprop(hipStream_t stream) {
  CudaDeviceContext context(get_device_id());
  CK_CUBLAS_THROW_(hipblasSetStream(cublas_handle_, stream));

  const __half* kernel = weights_half_[0]->get_ptr();
  const __half* top = top_tensor_->get_ptr();
  const __half* identity = identity_tensor_->get_ptr();
  __half* kernel_grad = weights_grad_[0]->get_ptr();
  __half* bias_grad = weights_grad_[1]->get_ptr();
  __half* bottom = bottom_tensor_->get_ptr();

  const auto& bottom_tensor_dim = bottom_tensor_->get_dims();
  const auto& top_tensor_dim = top_tensor_->get_dims();

  int m = bottom_tensor_dim[0];
  int n = top_tensor_dim[1];
  int k = bottom_tensor_dim[1];

  const float alpha = 1.0f;
  const float beta_b = 0.0f;
  const float beta_k = 1.0f;
  const float beta_x = 0.0f;

  CK_CUBLAS_THROW_(hipblasGemmEx(cublas_handle_, HIPBLAS_OP_N, HIPBLAS_OP_N, n, 1, m, &alpha, top,
                                HIP_R_16F, n, identity, HIP_R_16F, m, &beta_b, bias_grad,
                                HIP_R_16F, n, HIP_R_32F, balgo_b_));

  CK_CUBLAS_THROW_(hipblasGemmEx(cublas_handle_, HIPBLAS_OP_N, HIPBLAS_OP_T, n, k, m, &alpha, top,
                                HIP_R_16F, n, bottom, HIP_R_16F, k, &beta_k, kernel_grad,
                                HIP_R_16F, n, HIP_R_32F, balgo_k_));

  CK_CUBLAS_THROW_(hipblasGemmEx(cublas_handle_, HIPBLAS_OP_T, HIPBLAS_OP_N, k, m, n, &alpha, kernel,
                                HIP_R_16F, n, top, HIP_R_16F, n, &beta_x, bottom, HIP_R_16F, k,
                                HIP_R_32F, balgo_x_));

#ifndef NDEBUG
  hipDeviceSynchronize();
  CK_CUDA_THROW_(hipGetLastError());
#endif
}

void FullyConnectedLayerHalf::initialize() {
  CudaDeviceContext context(get_device_id());

  // CUDA stream to be used for cublas on this device
  hipStream_t stream;
  CK_CUDA_THROW_(hipStreamCreate(&stream));

  __half* identity = identity_tensor_->get_ptr();
  const auto& bottom_tensor_dim = bottom_tensor_->get_dims();
  size_t m = bottom_tensor_dim[0];

  // Initialize identity vector
  initialize_array<<<(m - 1) / 1024 + 1, 1024, 0, stream>>>(identity, m, __float2half(1.0f));

  CK_CUDA_THROW_(hipStreamDestroy(stream));
}

void FullyConnectedLayerHalf::search_algorithm() {
  // Set to the CUDA device where this layer assigned to
  CudaDeviceContext context(get_device_id());
  const size_t repeat_num = 100;

  // CUDA stream to be used for cublas on this device
  hipStream_t stream;
  CK_CUDA_THROW_(hipStreamCreate(&stream));

  // Set stream to cublas handler
  CK_CUBLAS_THROW_(hipblasSetStream(cublas_handle_, stream));

  // Device Tensors to be used
  __half* bottom = bottom_tensor_->get_ptr();
  __half* top = top_tensor_->get_ptr();
  __half* identity = identity_tensor_->get_ptr();
  __half* kernel = weights_half_[0]->get_ptr();
  __half* bias = weights_half_[1]->get_ptr();
  __half* kernel_grad = weights_grad_[0]->get_ptr();
  __half* bias_grad = weights_grad_[1]->get_ptr();

  // Tensor dim
  const auto& bottom_tensor_dim = bottom_tensor_->get_dims();
  const auto& top_tensor_dim = top_tensor_->get_dims();

  size_t m = bottom_tensor_dim[0];
  size_t n = top_tensor_dim[1];
  size_t k = bottom_tensor_dim[1];

  // Initialize identity vector
  initialize_array<<<(m - 1) / 1024 + 1, 1024, 0, stream>>>(identity, m, __float2half(1.0f));

  // Record time for each algorithm
  float shortestTime = std::numeric_limits<float>::max();
  float time;
  hipEvent_t start, stop;
  CK_CUDA_THROW_(hipEventCreate(&start));
  CK_CUDA_THROW_(hipEventCreate(&stop));

  // Start, end for search
  const hipblasGemmAlgo_t startAlgo = CUBLAS_GEMM_DEFAULT_TENSOR_OP;
  const hipblasGemmAlgo_t endAlgo = CUBLAS_GEMM_ALGO15_TENSOR_OP;

  // Search all the algorithm for falgo_b_
  for (int testAlgo = startAlgo; testAlgo <= endAlgo; testAlgo++) {
    hipblasStatus_t status = HIPBLAS_STATUS_SUCCESS;

    const float alpha = 1.0f;
    const float beta = 0.0f;

    // Record start event
    CK_CUDA_THROW_(hipEventRecord(start, stream));
    for (size_t i = 0; i < repeat_num && status == HIPBLAS_STATUS_SUCCESS; ++i) {
      status = hipblasGemmEx(cublas_handle_, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, 1, &alpha, bias,
                            HIP_R_16F, n, identity, HIP_R_16F, 1, &beta, top, HIP_R_16F, n,
                            HIP_R_32F, static_cast<hipblasGemmAlgo_t>(testAlgo));
    }
    CK_CUDA_THROW_(hipEventRecord(stop, stream));
    CK_CUDA_THROW_(hipEventSynchronize(stop));
    CK_CUDA_THROW_(hipEventElapsedTime(&time, start, stop));
    // Avg Time(ms) for this alorithm for fprop GEMM
    time = time / repeat_num;
    // Skip if the algorithm is supported for fprop configuration
    if (status != HIPBLAS_STATUS_SUCCESS) {
      // printf("The algorithms %d is not supported for fprop_b, skipped.\n", testAlgo);
      continue;
    }
    // Record the optimal time and algorithm
    if (time < shortestTime) {
      shortestTime = time;
      falgo_b_ = static_cast<hipblasGemmAlgo_t>(testAlgo);
    }
  }

  // Reset shortestTime
  shortestTime = std::numeric_limits<float>::max();

  // Search all the algorithm for falgo_k_
  for (int testAlgo = startAlgo; testAlgo <= endAlgo; testAlgo++) {
    hipblasStatus_t status = HIPBLAS_STATUS_SUCCESS;

    const float alpha = 1.0f;
    const float beta = 1.0f;

    // Record start event
    CK_CUDA_THROW_(hipEventRecord(start, stream));
    for (size_t i = 0; i < repeat_num && status == HIPBLAS_STATUS_SUCCESS; ++i) {
      status = hipblasGemmEx(cublas_handle_, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, &alpha, kernel,
                            HIP_R_16F, n, bottom, HIP_R_16F, k, &beta, top, HIP_R_16F, n,
                            HIP_R_32F, static_cast<hipblasGemmAlgo_t>(testAlgo));
    }
    CK_CUDA_THROW_(hipEventRecord(stop, stream));
    CK_CUDA_THROW_(hipEventSynchronize(stop));
    CK_CUDA_THROW_(hipEventElapsedTime(&time, start, stop));
    // Avg Time(ms) for this alorithm for fprop GEMM
    time = time / repeat_num;
    // Skip if the algorithm is supported for fprop configuration
    if (status != HIPBLAS_STATUS_SUCCESS) {
      // printf("The algorithms %d is not supported for fprop, skipped.\n", testAlgo);
      continue;
    }
    // Record the optimal time and algorithm
    if (time < shortestTime) {
      shortestTime = time;
      falgo_k_ = static_cast<hipblasGemmAlgo_t>(testAlgo);
    }
  }

  // Reset shortestTime
  shortestTime = std::numeric_limits<float>::max();

  // Search all the algorithm for balgo_b_
  for (int testAlgo = startAlgo; testAlgo <= endAlgo; testAlgo++) {
    hipblasStatus_t status = HIPBLAS_STATUS_SUCCESS;

    const float alpha = 1.0f;
    const float beta = 0.0f;

    // Record start event
    CK_CUDA_THROW_(hipEventRecord(start, stream));
    for (size_t i = 0; i < repeat_num && status == HIPBLAS_STATUS_SUCCESS; ++i) {
      status = hipblasGemmEx(cublas_handle_, HIPBLAS_OP_N, HIPBLAS_OP_N, n, 1, m, &alpha, top,
                            HIP_R_16F, n, identity, HIP_R_16F, m, &beta, bias_grad, HIP_R_16F, n,
                            HIP_R_32F, static_cast<hipblasGemmAlgo_t>(testAlgo));
    }
    CK_CUDA_THROW_(hipEventRecord(stop, stream));
    CK_CUDA_THROW_(hipEventSynchronize(stop));
    CK_CUDA_THROW_(hipEventElapsedTime(&time, start, stop));
    // Avg Time(ms) for this alorithm for fprop GEMM
    time = time / repeat_num;
    // Skip if the algorithm is supported for fprop configuration
    if (status != HIPBLAS_STATUS_SUCCESS) {
      // printf("The algorithms %d is not supported for bprop_W, skipped.\n", testAlgo);
      continue;
    }
    // Record the optimal time and algorithm
    if (time < shortestTime) {
      shortestTime = time;
      balgo_b_ = static_cast<hipblasGemmAlgo_t>(testAlgo);
    }
  }

  // Reset shortestTime
  shortestTime = std::numeric_limits<float>::max();

  // Search all the algorithm for balgo_k_
  for (int testAlgo = startAlgo; testAlgo <= endAlgo; testAlgo++) {
    hipblasStatus_t status = HIPBLAS_STATUS_SUCCESS;

    const float alpha = 1.0f;
    const float beta = 1.0f;

    // Record start event
    CK_CUDA_THROW_(hipEventRecord(start, stream));
    for (size_t i = 0; i < repeat_num && status == HIPBLAS_STATUS_SUCCESS; ++i) {
      status = hipblasGemmEx(cublas_handle_, HIPBLAS_OP_N, HIPBLAS_OP_T, n, k, m, &alpha, top,
                            HIP_R_16F, n, bottom, HIP_R_16F, k, &beta, kernel_grad, HIP_R_16F, n,
                            HIP_R_32F, static_cast<hipblasGemmAlgo_t>(testAlgo));
    }
    CK_CUDA_THROW_(hipEventRecord(stop, stream));
    CK_CUDA_THROW_(hipEventSynchronize(stop));
    CK_CUDA_THROW_(hipEventElapsedTime(&time, start, stop));
    // Avg Time(ms) for this alorithm for fprop GEMM
    time = time / repeat_num;
    // Skip if the algorithm is supported for fprop configuration
    if (status != HIPBLAS_STATUS_SUCCESS) {
      // printf("The algorithms %d is not supported for bprop_W, skipped.\n", testAlgo);
      continue;
    }
    // Record the optimal time and algorithm
    if (time < shortestTime) {
      shortestTime = time;
      balgo_k_ = static_cast<hipblasGemmAlgo_t>(testAlgo);
    }
  }

  // Reset shortestTime
  shortestTime = std::numeric_limits<float>::max();

  // Search all the algorithm for balgo_x_
  for (int testAlgo = startAlgo; testAlgo <= endAlgo; testAlgo++) {
    hipblasStatus_t status = HIPBLAS_STATUS_SUCCESS;

    const float alpha = 1.0f;
    const float beta = 0.0f;

    // Record start event
    CK_CUDA_THROW_(hipEventRecord(start, stream));
    for (size_t i = 0; i < repeat_num && status == HIPBLAS_STATUS_SUCCESS; ++i) {
      status = hipblasGemmEx(cublas_handle_, HIPBLAS_OP_T, HIPBLAS_OP_N, k, m, n, &alpha, kernel,
                            HIP_R_16F, n, top, HIP_R_16F, n, &beta, bottom, HIP_R_16F, k,
                            HIP_R_32F, static_cast<hipblasGemmAlgo_t>(testAlgo));
    }

    CK_CUDA_THROW_(hipEventRecord(stop, stream));
    CK_CUDA_THROW_(hipEventSynchronize(stop));
    CK_CUDA_THROW_(hipEventElapsedTime(&time, start, stop));
    // Avg Time(ms) for this alorithm for fprop GEMM
    time = time / repeat_num;
    // Skip if the algorithm is supported for fprop configuration
    if (status != HIPBLAS_STATUS_SUCCESS) {
      // printf("The algorithms %d is not supported for bprop_Xn, skipped.\n", testAlgo);
      continue;
    }
    // Record the optimal time and algorithm
    if (time < shortestTime) {
      shortestTime = time;
      balgo_x_ = static_cast<hipblasGemmAlgo_t>(testAlgo);
    }
  }

  // Print selection information
  // printf(
  //     "The algorithm selection for falgo_b_, falgo_k_, balgo_b_, balgo_k_, balgo_x_ are: %d, %d,
  //     "
  //     "%d, %d and %d.\n",
  //     (int)falgo_b_ - CUBLAS_GEMM_DEFAULT_TENSOR_OP, (int)falgo_k_ -
  //     CUBLAS_GEMM_DEFAULT_TENSOR_OP, (int)balgo_b_ - CUBLAS_GEMM_DEFAULT_TENSOR_OP, (int)balgo_k_
  //     - CUBLAS_GEMM_DEFAULT_TENSOR_OP, (int)balgo_x_ - CUBLAS_GEMM_DEFAULT_TENSOR_OP);

  // Output msg
  // MESSAGE_("The fully-connected layer has finished choosing the algorithm for cublas Gemm.");
  // Clean-up
  CK_CUDA_THROW_(hipEventDestroy(start));
  CK_CUDA_THROW_(hipEventDestroy(stop));
  CK_CUDA_THROW_(hipStreamDestroy(stream));
}  // namespace HugeCTR

std::unique_ptr<DataSimulator<float>> FullyConnectedLayerHalf::get_uniform_initializer(
    const int index) {
  size_t bottom_dim = bottom_tensor_->get_dims()[1];
  size_t top_dim = top_tensor_->get_dims()[1];

  float limit = 1.0f / ((0 == index ? bottom_dim : 0) + top_dim);
  return std::unique_ptr<DataSimulator<float>>(new UnifiedDataSimulator<float>(-1 * limit, limit));
}

std::unique_ptr<DataSimulator<float>> FullyConnectedLayerHalf::get_xavier_uniform_initializer(
    const int index) {
  size_t bottom_dim = bottom_tensor_->get_dims()[1];
  size_t top_dim = top_tensor_->get_dims()[1];

  return std::unique_ptr<DataSimulator<float>>(new VarianceScalingSimulator<float>(
      1.f, data_simu::Mode_t::Fan_avg, data_simu::Distribution_t::Uniform,
      0 == index ? bottom_dim : 0, top_dim));
}

std::unique_ptr<DataSimulator<float>> FullyConnectedLayerHalf::get_xavier_norm_initializer(
    const int index) {
  size_t bottom_dim = bottom_tensor_->get_dims()[1];
  size_t top_dim = top_tensor_->get_dims()[1];

  return std::unique_ptr<DataSimulator<float>>(new VarianceScalingSimulator<float>(
      1.f, data_simu::Mode_t::Fan_avg, data_simu::Distribution_t::Norm, 0 == index ? bottom_dim : 0,
      top_dim));
}

std::unique_ptr<DataSimulator<float>> FullyConnectedLayerHalf::get_default_initializer(
    const int index) {
  size_t bottom_dim = bottom_tensor_->get_dims()[1];
  size_t top_dim = top_tensor_->get_dims()[1];

  std::unique_ptr<DataSimulator<float>> simu(nullptr);
  if (0 == index) {
    simu.reset(new VarianceScalingSimulator<float>(
        1.f, data_simu::Mode_t::Fan_avg, data_simu::Distribution_t::Norm, bottom_dim, top_dim));
  } else if (1 == index) {
    float stddev = sqrt(1.f / top_dim);
    simu.reset(new GaussianDataSimulator<float>(0, stddev, -2 * stddev, 2 * stddev));
  } else {
    CK_THROW_(Error_t::OutOfBound, "index != {0, 1}.");
  }

  return simu;
}

}  // namespace HugeCTR
