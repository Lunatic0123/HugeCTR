#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "HugeCTR/include/layers/multi_cross_layer.hpp"
#include "HugeCTR/include/utils.cuh"
#include <math.h>
#include <vector>

namespace HugeCTR {
  MultiCrossLayer::MultiCrossLayer( const std::shared_ptr<GeneralBuffer<float>>& weight_buff,
				    const std::shared_ptr<GeneralBuffer<float>>& wgrad_buff,
				    const std::shared_ptr<Tensor<float>>& in_tensor,
				    const std::shared_ptr<Tensor<float>>& out_tensor,
				    int num_layers,
				    int device_id): num_layers_(num_layers), Layer(device_id) {
    try{
      // check the in_tensor and out_tensor
      const auto& in_tensor_dim = in_tensor->get_dims();
      const auto& out_tensor_dim = out_tensor->get_dims();
      // 1. two dim?
      if (in_tensor_dim.size() != 2 || out_tensor_dim.size() != 2) {
	CK_THROW_(Error_t::WrongInput, "input or output tensor doesn't has two dimensions");
      }
      // 2. same dim?
      for(int i = 0;i<2; i++){
	if(in_tensor_dim[i] != out_tensor_dim[i]){
	  CK_THROW_(Error_t::WrongInput, "input and output tensor doesn't match");
	}
      }
      int vec_length = in_tensor_dim[0]; 
      int batchsize = in_tensor_dim[1]; 

      // check num_lyaers
      if (num_layers < 1){
	  CK_THROW_(Error_t::WrongInput, "num_layers < 1");
      }

      std::vector<int> weight_bias_dim = {1, vec_length};
      for(int i = 0; i<num_layers; i++){
	//setup weights
	weights_.emplace_back(new Tensor<float>(weight_bias_dim, weight_buff, TensorFormat_t::HW));
	//setup bias
	weights_.emplace_back(new Tensor<float>(weight_bias_dim, weight_buff, TensorFormat_t::HW));
	//setup weight gradient
	weights_.emplace_back(new Tensor<float>(weight_bias_dim, weight_buff, TensorFormat_t::HW));
	//setup bias gradient
	weights_.emplace_back(new Tensor<float>(weight_bias_dim, weight_buff, TensorFormat_t::HW));
      }
      //setup internal blobs
      std::vector<int> blob_dim = {batchsize, vec_length};
      for(int i = 0; i<num_layers-1; i++){
	blob_tensors_.emplace_back(new Tensor<float>(blob_dim, blobs_buff_, TensorFormat_t::HW));
      }
      
      for(int i = 0; i<TMP_MATS; i++){
	tmp_mat_tensors_.emplace_back(new Tensor<float>(blob_dim, blobs_buff_, TensorFormat_t::HW));
      }
      std::vector<int> tmp_vec_dim = {batchsize, 1};
      for(int i = 0; i < TMP_VECS; i++){
	tmp_vec_tensors_.emplace_back(new Tensor<float>(tmp_vec_dim, blobs_buff_, TensorFormat_t::HW));
      }
      blobs_buff_->init(device_id);
    } catch (const std::runtime_error& rt_err) {
      std::cerr << rt_err.what() << std::endl;
      throw;
    }
  }

  //kernels
  namespace {

    /**
     * Each row in `mat`dot product with vec, length of vec should be w. Then adding bias for each of the rows
     */
    __global__ void matrix_vec_mul_kernel(float* out, float* mat, int h, int w, float* vec, float bias){
      const int tid = blockDim.x*blockIdx.x+threadIdx.x;
      const int wtid = tid%WARP_SIZE; //thread id in warp
      const int wid = tid/WARP_SIZE; //warp id
      const float* mat_with_offset = mat + wid*w;
      
      float accum = 0.f;
      for(int i = wtid; i < w; i+=WARP_SIZE){
	accum += mat_with_offset[i]*vec[i];
      }
      float val = warpReduceSum(accum);
      if(tid == 0){
	out[wid] = val + bias;
      }
    }

    void matrix_vec_mul(std::shared_ptr<Tensor<float>> out, 
			std::shared_ptr<Tensor<float>> mat, 
			std::shared_ptr<Tensor<float>> vec, float bias = 0.f,
			hipStream_t stream){
      float* pout = out.get_ptr();
      float* pmat = mat.get_ptr();
      float* pvec = vec.get_ptr();

      const auto dim = out.get_dims();
      const auto idim = mat.get_dims();
      assert(dim.size() == 2 && idim.size() == 2 && dim[1] == vec.get_dims()[1] && vec.get_dims()[0] == 1);
      assert(idim[0] == dim[0] && idim[1] == dim [1]);

      const int h = dim[0];
      const int w = dim[1];

      const int BLOCK_DIM = 256;
      const int GRID_DIM = calc_grid(h*WARP_SIZE, BLOCK_DIM);

      matrix_vec_mul_kernel<<<GRID_DIM, BLOCK_DIM, 0, stream>>>(pout, pmat, h, w, pvec, bias);
    }

    /**
     * Each row in `mat` scale with the coresponding element in vec. 
     * The length of vec should be h.
     */
    __global__ void row_scaling_kenrel(float* o_mat, float* mat, int h, int w, float* vec){
      const int tid = blockDim.x*blockIdx.x+threadIdx.x;
      if(tid < h*w){
	const int row = tid/w;
	o_mat[tid] = mat[tid]*vec[row];
      }
    }
    
    void row_scaling(std::shared_ptr<Tensor<float>> o_mat,
		     std::shared_ptr<Tensor<float>> mat, 
		     std::shared_ptr<Tensor<float>>vec,
		     hipStream_t stream){

      float* pout = o_mat.get_ptr();
      float* pmat = mat.get_ptr();
      float* pvec = vec.get_ptr();

      const auto dim = o_mat.get_dims();
      const auto idim = mat.get_dims();
      assert(dim.size() == 2 && idim.size() == 2 && dim[0] == vec.get_dims()[0] && vec.get_dims()[1] == 1);
      assert(idim[0] == dim[0] && idim[1] == dim [1]);

      const int h = dim[0];
      const int w = dim[1];

      const int BLOCK_DIM = 256;
      const int GRID_DIM = calc_grid(h*w, BLOCK_DIM);

      row_scaling_kenrel<<<GRID_DIM, BLOCK_DIM, 0, stream>>>(pout, pmat, h, w, pvec);
    }

    /**
     * Each row in `mat` sum with  vec. 
     * The length of vec should be w.
     */
    __global__ void matrix_vec_add_kenrel(float* o_mat, float* mat, int h, int w, float* vec){
      const int tid = blockDim.x*blockIdx.x+threadIdx.x;
      if(tid < h*w){
	const int col = tid%w;
	o_mat[tid] = mat[tid]*vec[col];
      }
    }

    void matrix_vec_add(std::shared_ptr<Tensor<float>> o_mat, 
			std::shared_ptr<Tensor<float>> mat, 
			std::shared_ptr<Tensor<float>> vec,
			hipStream_t stream){

      float* pout = o_mat.get_ptr();
      float* pmat = mat.get_ptr();
      float* pvec = vec.get_ptr();

      const auto dim = o_mat.get_dims();
      const auto idim = mat.get_dims();
      assert(dim.size() == 2 && idim.size() == 2 && dim[1] == vec.get_dims()[1] && vec.get_dims()[0] == 1);
      assert(idim[0] == dim[0] && idim[1] == dim [1]);

      const int h = dim[0];
      const int w = dim[1];

      const int BLOCK_DIM = 256;
      const int GRID_DIM = calc_grid(h*w, BLOCK_DIM);
      
      matrix_vec_add_kenrel<<<GRID_DIM, BLOCK_DIM, 0, stream>>>(pout, pmat, h, w, pvec);
    }

    /**
     * Pointwise adding
     */

    __global__ void matrix_add_kenrel(float* o_mat, float* mat_a, int h, int w, float* mat_b){
      const int tid = blockDim.x*blockIdx.x+threadIdx.x;
      if(tid < h*w){
	o_mat[tid] = mat_a[tid]*mat_b[tid];
      }
    }

    void matrix_add(std::shared_ptr<Tensor<float>> out_mat, 
		    std::shared_ptr<Tensor<float>>  mat_a, 
		    std::shared_ptr<Tensor<float>> mat_b,
		    hipStream_t stream){
      float* pout = out_mat.get_ptr();
      float* pmat_a = mat_a.get_ptr();
      float* pmat_b = mat_b.get_ptr();

      const auto dim = out_mat.get_dims();

      const int h = dim[0];
      const int w = dim[1];

      const int BLOCK_DIM = 256;
      const int GRID_DIM = calc_grid(h*w, BLOCK_DIM);
      matrix_add_kenrel<<<GRID_DIM, BLOCK_DIM, 0, stream>>>(pout, pmat_a, h, w, pmat_b);
    }
    /**
     * compute dot product for each pair of the rows in the two matrix, 
     */
    __global__ void matrix_pair_mul_kernel(float* o_mat, float* mat_a, int h, int w, float* mat_b){
      const int tid = blockDim.x*blockIdx.x+threadIdx.x;
      const int wtid = tid%WARP_SIZE; //thread id in warp
      const int wid = tid/WARP_SIZE; //warp id
      const float* mat_a_with_offset = mat_a + wid*w;
      const float* mat_b_with_offset = mat_b + wid*w;
      float accum = 0.f;
      for(int i = wtid; i < w; i+=WARP_SIZE){
	accum += mat_a_with_offset[i]*mat_b_with_offset[i];
      }
      float val = warpReduceSum(accum);
      if(tid == 0){
	o_mat[wid] = val;
      }
    }
    void matrix_pair_mul(std::shared_ptr<Tensor<float>> out_mat, 
			 std::shared_ptr<Tensor<float>> mat_a, 
			 std::shared_ptr<Tensor<float>> mat_b,
			 hipStream_t stream){
      float* pout = out_mat.get_ptr();
      float* pmat_a = mat_a.get_ptr();
      float* pmat_b = mat_b.get_ptr();

      const auto dim = out_mat.get_dims();

      const int h = dim[0];
      const int w = dim[1];

      const int BLOCK_DIM = 256;
      const int GRID_DIM = calc_grid(h*WARP_SIZE, BLOCK_DIM);
      matrix_pair_mul_kernel<<<GRID_DIM, BLOCK_DIM, 0, stream>>>(pout, pmat_a, h, w, pmat_b);
    }
  
  
    /**
     * out product of two vectors, 
     */
    __global__ void out_product_kernel(float* out_mat, float* vec_a, int h, float* vec_b, int w){
      const int tid = blockDim.x*blockIdx.x+threadIdx.x;
      if(tid < h*w){
	const int col = tid%w;
	const int row = tid/w;
	out_mat[tid] = vec_a[row]*vec_b[col];
      }
    }
    void out_product(std::shared_ptr<Tensor<float>> out_mat, 
			    std::shared_ptr<Tensor<float>> vec_a, 
			    std::shared_ptr<Tensor<float>> vec_b,
			    hipStream_t stream){
      float* pout = out_mat.get_ptr();
      float* pvec_a = vec_a.get_ptr();
      float* pvec_b = vec_b.get_ptr();
      const auto dim = out_mat.get_dims();

      const int h = dim[0];
      const int w = dim[1];

      assert(h == vec_a.get_dims()[0] && w == vec_b.get_dims()[1] && 
	     vec_a.get_dims()[1] == 1 && vec_b.get_dims()[0] == 1);
      const int BLOCK_DIM = 256;
      const int GRID_DIM = calc_grid(h*w, BLOCK_DIM);
      out_product_kernel<<<GRID_DIM, BLOCK_DIM, 0, stream>>>(pout, pvec_a, h, pvec_b, w);
    }

  }


  void MultiCrossLayer::fprof_step_(std::shared_ptr<Tensor<float>> xL_next, //output
				    std::shared_ptr<Tensor<float>> x0, 
				    std::shared_ptr<Tensor<float>> xL,
				    std::shared_ptr<Tensor<float>> wL,
				    std::shared_ptr<Tensor<float>> bL,
				    hipStream_t stream){
    
    return;
  }

  
  void MultiCrossLayer::fprop(hipStream_t stream){
    
    return;
  }

  void MultiCrossLayer::bprop_first_step_(std::shared_ptr<Tensor<float>> dxL_pre, //output
					  std::shared_ptr<Tensor<float>> dwL, //output
					  std::shared_ptr<Tensor<float>> dbL, //output
					  std::shared_ptr<Tensor<float>> dxL,
					  std::shared_ptr<Tensor<float>> wL,
					  std::shared_ptr<Tensor<float>> bL,
					  hipStream_t stream){

    return;
  }


  void MultiCrossLayer::bprop_step_(std::shared_ptr<Tensor<float>> dxL_pre, //output
				    std::shared_ptr<Tensor<float>> dwL, //output
				    std::shared_ptr<Tensor<float>> dbL, //output
				    std::shared_ptr<Tensor<float>> x0,
				    std::shared_ptr<Tensor<float>> dxL,
				    std::shared_ptr<Tensor<float>> wL,
				    std::shared_ptr<Tensor<float>> bL,
				    hipStream_t stream){
    return;
  }


  void MultiCrossLayer::bprop(hipStream_t stream){

    return;
  }



} //namespace HugeCTR
