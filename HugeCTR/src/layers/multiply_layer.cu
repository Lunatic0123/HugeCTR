#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "HugeCTR/include/layers/multiply_layer.hpp"
#include "HugeCTR/include/layers/element_wise_function.hpp"
#include "HugeCTR/include/utils.hpp"
#include "HugeCTR/include/utils.cuh"

#include <algorithm>
#include <functional>

#ifndef NDEBUG
#include <iostream>
#endif
 
namespace HugeCTR {
 
namespace {

#define BLOCK_DIM_SIZE 32

template<typename T>
__global__ void multiply_kernel(const T * input, 
                                const T * weight, 
                                T * output, 
                                int batch_size, 
                                int slot_num,
                                int embedding_vec_size) {
  if((blockIdx.x < batch_size) && (threadIdx.x < embedding_vec_size)) {
    for(int i = 0; i < slot_num; i++) {
      output[blockIdx.x * slot_num * embedding_vec_size + i * embedding_vec_size + threadIdx.x] = 
        input[blockIdx.x * slot_num + i] * weight[i * embedding_vec_size + threadIdx.x];
    }
  }
}

template<typename T>
__global__ void multiply_transpose_fuse_kernel(int batch_size,
                                              int slot_num,
                                              int embedding_vec_size,
                                              const T * top_grad,
                                              const T * input,
                                              T * wgrad_tmp_trans) {
  int row = batch_size;
  int col = slot_num * embedding_vec_size;
  __shared__ T sh_data[BLOCK_DIM_SIZE+1][BLOCK_DIM_SIZE];

  int src_index_x = blockIdx.x * blockDim.x + threadIdx.x;
  int src_index_y = blockIdx.y * blockDim.y + threadIdx.y;
  if ((src_index_x < col) && (src_index_y < row))
  {
    int index_in = src_index_y * col + src_index_x;
    sh_data[threadIdx.x][threadIdx.y] = 
      top_grad[index_in] * input[index_in/embedding_vec_size];
  }

  __syncthreads();

  int dst_index_x = blockIdx.y*blockDim.y + threadIdx.x;
  int dst_index_y = blockIdx.x*blockDim.x + threadIdx.y;
  if ((dst_index_x < row) && (dst_index_y < col))
  {
    int index_out = dst_index_y * row + dst_index_x;
    wgrad_tmp_trans[index_out] = sh_data[threadIdx.y][threadIdx.x];
  }
}

// sum reduce computation in one block
template<typename T>
__global__ void sum_reduce_batch_kernel(int row, // row=gridDim.x
                                        int col,
                                        const T * input, 
                                        T * output) {
  float local_sum = 0.0f;
  for (int tid = threadIdx.x; tid < col; tid += blockDim.x) {
    local_sum += input[blockIdx.x * col + tid];
  }
  __syncthreads();

  local_sum = blockReduceSum(local_sum);
  if (threadIdx.x == 0) {
    output[blockIdx.x] += local_sum;
  }
}

template<typename T>
__global__ void multiply_dgrad_kernel(const T * top_grad,
                                      const T * weight,
                                      T * dgrad,
                                      int batch_size, 
                                      int slot_num,
                                      int embedding_vec_size) {
  if((blockIdx.x < batch_size) && (threadIdx.x < embedding_vec_size)) {
    for(int i = 0; i < slot_num; i++) {
      T local_sum = 
      top_grad[blockIdx.x * slot_num * embedding_vec_size + i * embedding_vec_size + threadIdx.x]
        * weight[i * embedding_vec_size + threadIdx.x];

      local_sum = blockReduceSum(local_sum);
      if(threadIdx.x == 0) {
        dgrad[blockIdx.x * slot_num + i] = local_sum;
      }
    }
  }
}

template<typename T>
void multiply_wgrad(const T * top_grad,
                    const T * input,
                    T * wgrad,
                    T * wgrad_tmp_trans,
                    int batch_size,
                    int slot_num,
                    int embedding_vec_size,
                    hipStream_t stream) {

  dim3 blockSize1(BLOCK_DIM_SIZE, BLOCK_DIM_SIZE, 1);
  dim3 gridSize1((slot_num*embedding_vec_size+blockSize1.x-1)/blockSize1.x,
                 (batch_size+blockSize1.y-1)/blockSize1.y, 1);
  multiply_transpose_fuse_kernel<<<gridSize1, blockSize1, 0, stream>>>(batch_size, 
                                                                      slot_num,
                                                                      embedding_vec_size,
                                                                      top_grad,
                                                                      input,
                                                                      wgrad_tmp_trans);
             
  dim3 blockSize2(256, 1, 1);
  dim3 gridSize2(slot_num*embedding_vec_size, 1, 1);
  sum_reduce_batch_kernel<<<gridSize2, blockSize2, 0, stream>>>(slot_num*embedding_vec_size, 
                                                                batch_size, 
                                                                wgrad_tmp_trans, 
                                                                wgrad);
}

template<typename T> 
void multiply_dgrad(const T * top_grad,
                    const T * weight,
                    T * dgrad,
                    int batch_size,
                    int slot_num,
                    int embedding_vec_size,
                    hipStream_t stream) {

  dim3 blockSize(embedding_vec_size, 1, 1);
  dim3 gridSize(batch_size, 1, 1);
  multiply_dgrad_kernel<<<gridSize, blockSize, 0, stream>>>(top_grad, weight, dgrad, 
                                                            batch_size, slot_num,
                                                            embedding_vec_size);
}

} // end of namespace

MultiplyLayer::MultiplyLayer(const std::shared_ptr<GeneralBuffer<float>>& weight_buff,
                            const std::shared_ptr<GeneralBuffer<float>>& wgrad_buff,
                            const std::shared_ptr<Tensor<float>>& in_tensor,
                            const std::shared_ptr<Tensor<float>>& out_tensor, 
                            int device_id)
     : Layer(device_id) {
  try {
    CudaDeviceContext context(get_device_id());

    auto in_dims = in_tensor->get_dims();
    if(in_dims.size() != 3) {
      CK_THROW_(Error_t::WrongInput, "Only 3D tensors can be multiplied");
    }
    if(in_tensor->get_format() != TensorFormat_t::HSW) {
      CK_THROW_(Error_t::WrongInput, "Only TensorFormat_t::HSW is allowed for multiply layer");
    }
    auto out_dims = out_tensor->get_dims();
    if(out_dims.size() != 3) {
      CK_THROW_(Error_t::WrongInput, "only 3D tensors can be set as the result of multiply layer");
    }
    if(out_tensor->get_format() != TensorFormat_t::HSW) {
      CK_THROW_(Error_t::WrongInput, "Only TensorFormat_t::HSW is allowed for multiply layer");
    }
    if(out_dims[2] > 1024) { // embedding_vec_size 
      CK_THROW_(Error_t::WrongInput, "the out_dims[2] can not be more than 1024 in multiply layer");
    }

    in_tensors_.emplace_back(in_tensor);
    out_tensors_.emplace_back(out_tensor);

    std::vector<int> w_dim = {out_dims[1], out_dims[2]};  // {slot_num. embedding_vec_size}
    TensorFormat_t w_format = TensorFormat_t::HW;
    weights_.emplace_back(new Tensor<float>(w_dim, weight_buff, w_format));
    wgrad_.emplace_back(new Tensor<float>(w_dim, wgrad_buff, w_format));

    internal_buff_.reset(new GeneralBuffer<float>());
    wgrad_tmp_trans_.reset(new Tensor<float>(out_dims, internal_buff_, TensorFormat_t::HSW));
    internal_buff_->init(get_device_id());

  } catch (const std::runtime_error& rt_err) {
    std::cerr << rt_err.what() << std::endl;
    throw;
  }
}
 
void MultiplyLayer::fprop(hipStream_t stream) {
  CudaDeviceContext context(get_device_id());

  float* input = in_tensors_[0]->get_ptr();
  float * weight = weights_[0]->get_ptr();
  float* output = out_tensors_[0]->get_ptr();
  int batch_size = out_tensors_[0]->get_dims()[0];
  int slot_num = out_tensors_[0]->get_dims()[1];
  int embedding_vec_size = out_tensors_[0]->get_dims()[2];

  dim3 blockSize(embedding_vec_size, 1, 1);
  dim3 gridSize(batch_size, 1, 1);
  multiply_kernel<<<gridSize, blockSize, 0, stream>>>(input, weight, output, 
                                                      batch_size, slot_num, 
                                                      embedding_vec_size);
}
 
void MultiplyLayer::bprop(hipStream_t stream) {
  CudaDeviceContext context(get_device_id());

  float* weight = weights_[0]->get_ptr();
  float* wgrad = wgrad_[0]->get_ptr();
  float* wgrad_tmp_trans = wgrad_tmp_trans_->get_ptr();
  float* input = in_tensors_[0]->get_ptr();
  float* output = out_tensors_[0]->get_ptr();
  int batch_size = out_tensors_[0]->get_dims()[0];
  int slot_num = out_tensors_[0]->get_dims()[1];
  int embedding_vec_size = out_tensors_[0]->get_dims()[2];

  hipMemsetAsync(wgrad, 0, wgrad_[0]->get_size(), stream);

  multiply_wgrad(output, input, wgrad, wgrad_tmp_trans, batch_size, slot_num, embedding_vec_size, stream);

  // CAUSION: dgrad computation will modify the "input", so it must be put after wgrad computation
  multiply_dgrad(output, weight, input, batch_size, slot_num, embedding_vec_size, stream);
}
 
}  // namespace HugeCTR
 