
#include <hip/hip_runtime.h>
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

namespace HugeCTR {

// Kernels to combine the value buffer
__global__ void merge_emb_vec(
    float* d_output_emb_vec,
    const float* d_missing_emb_vec,
    const uint64_t* d_missing_index,
    const size_t len,
    const size_t emb_vec_size
    )
{
    const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if ( idx < (len * emb_vec_size) )
    {
      size_t src_emb_vec = idx / emb_vec_size;
      size_t dst_emb_vec = d_missing_index[src_emb_vec];
      size_t dst_float = idx % emb_vec_size;
      d_output_emb_vec[dst_emb_vec * emb_vec_size + dst_float] = d_missing_emb_vec[src_emb_vec * emb_vec_size + dst_float];
    }
}

void merge_emb_vec_async(float* d_vals_merge_dst_ptr, 
                         const float* d_vals_retrieved_ptr, 
                         const uint64_t* d_missing_index_ptr,
                         const size_t missing_len, 
                         const size_t emb_vec_size, 
                         const size_t BLOCK_SIZE, 
                         hipStream_t stream){
  size_t missing_len_in_float = missing_len * emb_vec_size;
  merge_emb_vec<<<((missing_len_in_float - 1) / BLOCK_SIZE) + 1, BLOCK_SIZE, 0, stream>>>(d_vals_merge_dst_ptr, 
                                                                                          d_vals_retrieved_ptr, 
                                                                                          d_missing_index_ptr, 
                                                                                          missing_len,
                                                                                          emb_vec_size);
}

}  // namespace HugeCTR
