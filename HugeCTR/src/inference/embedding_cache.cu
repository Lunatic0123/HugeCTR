#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <inference/embedding_cache.hpp>

namespace HugeCTR {
// Temp interface, should be delete later
nlohmann::json read_json_file(const std::string& filename);

// Kernels to combine the value buffer
__global__ void merge_emb_vec(
    float* d_output_emb_vec,
    const float* d_missing_emb_vec,
    const uint64_t* d_missing_index,
    const size_t len,
    const size_t emb_vec_size
    )
{
    const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if ( idx < (len * emb_vec_size) )
    {
      size_t src_emb_vec = idx / emb_vec_size;
      size_t dst_emb_vec = d_missing_index[src_emb_vec];
      size_t dst_float = idx % emb_vec_size;
      d_output_emb_vec[dst_emb_vec * emb_vec_size + dst_float] = d_missing_emb_vec[src_emb_vec * emb_vec_size + dst_float];
    }
}

template <typename TypeHashKey>
embedding_cache<TypeHashKey>::embedding_cache(HugectrUtility<TypeHashKey>* parameter_server,
                                              int cuda_dev_id,
                                              bool use_gpu_embedding_cache,
                                              float cache_size_percentage,
                                              const std::string& model_config_path,
                                              const std::string& model_name){
  // Store the configuration
  parameter_server_ = parameter_server;
  cache_config_.use_gpu_embedding_cache_ = use_gpu_embedding_cache;
  cache_config_.model_name_ = model_name;
  if(cache_config_.use_gpu_embedding_cache_){
    cache_config_.cuda_dev_id_ = cuda_dev_id;
    cache_config_.cache_size_percentage_ = cache_size_percentage;
  }

  // Open model config file and input model json config
  nlohmann::json model_config(read_json_file(model_config_path));

  // Read inference config
  const nlohmann::json& j_inference = get_json(model_config, "inference");
  const size_t max_batchsize = get_value_from_json<size_t>(j_inference, "max_batchsize");
  const nlohmann::json& j_emb_table_file = get_json(j_inference, "sparse_model_file");
  std::vector<std::string> emb_file_path;
  if (j_emb_table_file.is_array()){
    cache_config_.num_emb_table_ = j_emb_table_file.size();
    for(unsigned int i = 0; i < j_emb_table_file.size(); i++){
      emb_file_path.emplace_back(j_emb_table_file[i].get<std::string>());
    }
  }
  else{
    cache_config_.num_emb_table_ = 1;
    emb_file_path.emplace_back(j_emb_table_file.get<std::string>());
  }

  const nlohmann::json& j_layers = get_json(model_config, "layers");
  // Read data layer config
  const nlohmann::json& j_data_layer = j_layers[0];
  std::string data_layer_type = get_value_from_json<std::string>(j_data_layer, "type");
  if(data_layer_type.compare("Data") != 0){
    CK_THROW_(Error_t::WrongInput, "Wrong json format: The first layer is not Data layer:" + data_layer_type);
  }
  const nlohmann::json& j_data_layer_sparse_layer = get_json(j_data_layer, "sparse");
  if(!j_data_layer_sparse_layer.is_array()){
    CK_THROW_(Error_t::WrongInput, "Wrong json format: The sparse layer in data layer is not an array.");
  }
  if(j_data_layer_sparse_layer.size() != cache_config_.num_emb_table_){
    CK_THROW_(Error_t::WrongInput, "Wrong json format: The number of embedding table is not consistent.");
  }
  std::vector<size_t> max_feature_num_per_sample;
  for(unsigned int i = 0; i < j_data_layer_sparse_layer.size(); i++){
    max_feature_num_per_sample.emplace_back(get_value_from_json<size_t>(j_data_layer_sparse_layer[i], "max_feature_num_per_sample"));
  }

  // Read embedding layer config
  std::vector<bool> distributed_emb;
  // Search for all embedding layers
  for(unsigned int i = 1; i < j_layers.size(); i++){
    const nlohmann::json& j_single_layer = j_layers[i];
    std::string embedding_type = get_value_from_json<std::string>(j_single_layer, "type");
    if(embedding_type.compare("DistributedSlotSparseEmbeddingHash") == 0){
      distributed_emb.emplace_back(true);
      const nlohmann::json& embedding_hparam = get_json(j_single_layer, "sparse_embedding_hparam");
      cache_config_.embedding_vec_size_.emplace_back(get_value_from_json<size_t>(embedding_hparam, "embedding_vec_size"));
    }
    else if(embedding_type.compare("LocalizedSlotSparseEmbeddingHash") == 0 || embedding_type.compare("LocalizedSlotSparseEmbeddingOneHot") == 0){
      distributed_emb.emplace_back(false);
      const nlohmann::json& embedding_hparam = get_json(j_single_layer, "sparse_embedding_hparam");
      cache_config_.embedding_vec_size_.emplace_back(get_value_from_json<size_t>(embedding_hparam, "embedding_vec_size"));
    }
    else{
      break;
    }
  }

  if(distributed_emb.size() != cache_config_.num_emb_table_){
    CK_THROW_(Error_t::WrongInput, "Wrong json format: The number of embedding table is not consistent.");
  }

  // Calculate max_query_len_per_emb_table
  for(unsigned int i = 0; i < cache_config_.num_emb_table_; i++){
    cache_config_.max_query_len_per_emb_table_.emplace_back(max_batchsize * max_feature_num_per_sample[i]);
  }

  // Query the size of all embedding tables and calculate the size of each embedding cache
  if(cache_config_.use_gpu_embedding_cache_){
    for(unsigned int i = 0; i < cache_config_.num_emb_table_; i++){
      std::ifstream emb_file(emb_file_path[i]);
      // Check if file is opened successfully
      if (!emb_file.is_open()) {
        CK_THROW_(Error_t::WrongInput, "Error: embeddings file cannot open for reading");
      }
      emb_file.seekg(0, emb_file.end);
      size_t file_size = emb_file.tellg();
      emb_file.seekg(0, emb_file.beg);

      // File format is different for distributed and localized embeddings
      if(distributed_emb[i]){
        size_t row_size = sizeof(TypeHashKey) + sizeof(float) * cache_config_.embedding_vec_size_[i];
        size_t row_num = file_size / row_size;
        if (file_size % row_size != 0){
          CK_THROW_(Error_t::WrongInput, "Error: embeddings file size is not correct");
        }
        size_t num_feature_in_cache = (size_t)((double)(cache_config_.cache_size_percentage_) * (double)row_num);
        cache_config_.num_set_in_cache_.emplace_back(num_feature_in_cache / (SLAB_SIZE * SET_ASSOCIATIVITY));
      }
      else{
        size_t row_size = sizeof(TypeHashKey) + sizeof(size_t) + sizeof(float) * cache_config_.embedding_vec_size_[i];
        size_t row_num = file_size / row_size;
        if (file_size % row_size != 0){
          CK_THROW_(Error_t::WrongInput, "Error: embeddings file size is not correct");
        }
        size_t num_feature_in_cache = (size_t)((double)(cache_config_.cache_size_percentage_) * (double)row_num);
        cache_config_.num_set_in_cache_.emplace_back(num_feature_in_cache / (SLAB_SIZE * SET_ASSOCIATIVITY));
      }
      emb_file.close();
    }
  }

  // Construct gpu embedding cache, 1 per embedding table
  if(cache_config_.use_gpu_embedding_cache_){

    // Device Restorer
    CudaDeviceContext dev_restorer;

    // Set CUDA device before creating gpu embedding cache
    CK_CUDA_THROW_(hipSetDevice(cache_config_.cuda_dev_id_));

    for(unsigned int i = 0; i < cache_config_.num_emb_table_; i++){
      gpu_emb_caches_.emplace_back(new cache_(cache_config_.num_set_in_cache_[i], cache_config_.embedding_vec_size_[i]));
    }

  }
  
}

template <typename TypeHashKey>
embedding_cache<TypeHashKey>::~embedding_cache(){
  // Destruct gpu embedding cache
  if(cache_config_.use_gpu_embedding_cache_){
    // Device Restorer
    CudaDeviceContext dev_restorer;
    // Set CUDA device before destructing gpu embedding cache
    CK_CUDA_THROW_(hipSetDevice(cache_config_.cuda_dev_id_));
    for(unsigned int i = 0; i < cache_config_.num_emb_table_; i++){
      delete gpu_emb_caches_[i];
    }
  }
}

template <typename TypeHashKey> 
void embedding_cache<TypeHashKey>::look_up(const void* h_embeddingcolumns,
                                           const std::vector<size_t>& h_embedding_offset,
                                           float* d_shuffled_embeddingoutputvector,
                                           embedding_cache_workspace& workspace_handler,
                                           const std::vector<hipStream_t>& streams){
  // Shuffle the input embeddingcolumns
  size_t num_sample = (h_embedding_offset.size() - 1) / cache_config_.num_emb_table_;
  size_t acc_offset = 0;
  for(unsigned int i = 0; i < cache_config_.num_emb_table_; i++){
    workspace_handler.h_shuffled_embedding_offset_[i] = acc_offset;
    for(unsigned int j = 0; j < num_sample; j++){
      TypeHashKey* dst_ptr = (TypeHashKey*)(workspace_handler.h_shuffled_embeddingcolumns_) + acc_offset;
      TypeHashKey* src_prt = (TypeHashKey*)(h_embeddingcolumns) + h_embedding_offset[j * cache_config_.num_emb_table_ + i];
      size_t cpy_len = h_embedding_offset[j * cache_config_.num_emb_table_ + i + 1] - h_embedding_offset[j * cache_config_.num_emb_table_ + i];
      size_t cpy_len_in_byte = cpy_len * sizeof(TypeHashKey);
      memcpy(dst_ptr, src_prt, cpy_len_in_byte);
      acc_offset += cpy_len;
    }
  }
  workspace_handler.h_shuffled_embedding_offset_[cache_config_.num_emb_table_] = acc_offset;
  if(workspace_handler.h_shuffled_embedding_offset_[cache_config_.num_emb_table_] != h_embedding_offset[num_sample * cache_config_.num_emb_table_]){
    CK_THROW_(Error_t::WrongInput, "Error: embeddingcolumns buffer size is not consist before and after shuffle.");
  }

  // If GPU embedding cache is enabled
  if(cache_config_.use_gpu_embedding_cache_){

    // Device Restorer
    CudaDeviceContext dev_restorer;
    // Set CUDA device before doing look up
    CK_CUDA_THROW_(hipSetDevice(cache_config_.cuda_dev_id_));

    // Copy the shuffled embeddingcolumns buffer to device
    CK_CUDA_THROW_(hipMemcpyAsync(workspace_handler.d_shuffled_embeddingcolumns_, 
                                   workspace_handler.h_shuffled_embeddingcolumns_, 
                                   workspace_handler.h_shuffled_embedding_offset_[cache_config_.num_emb_table_] * sizeof(TypeHashKey), 
                                   hipMemcpyHostToDevice, 
                                   streams[0]));
    CK_CUDA_THROW_(hipStreamSynchronize(streams[0]));

    // Query the embeddingcolumns from GPU embedding cache & copy the missing length back
    size_t acc_emb_vec_offset = 0;
    for(unsigned int i = 0; i < cache_config_.num_emb_table_; i++){
      TypeHashKey* d_query_key_ptr = (TypeHashKey*)(workspace_handler.d_shuffled_embeddingcolumns_) + workspace_handler.h_shuffled_embedding_offset_[i];
      size_t query_length = workspace_handler.h_shuffled_embedding_offset_[i + 1] - workspace_handler.h_shuffled_embedding_offset_[i];
      float* d_vals_retrieved_ptr = d_shuffled_embeddingoutputvector + acc_emb_vec_offset;
      uint64_t* d_missing_index_ptr = workspace_handler.d_missing_index_ + workspace_handler.h_shuffled_embedding_offset_[i];
      TypeHashKey* d_missing_key_ptr = (TypeHashKey*)(workspace_handler.d_missing_embeddingcolumns_) + workspace_handler.h_shuffled_embedding_offset_[i];

      gpu_emb_caches_[i] -> Query(d_query_key_ptr, 
                                  query_length, 
                                  d_vals_retrieved_ptr, 
                                  d_missing_index_ptr, 
                                  d_missing_key_ptr, 
                                  workspace_handler.d_missing_length_ + i, 
                                  streams[i]);
      
      CK_CUDA_THROW_(hipMemcpyAsync(workspace_handler.h_missing_length_ + i, workspace_handler.d_missing_length_ + i, sizeof(size_t), hipMemcpyDeviceToHost, streams[i]));
      acc_emb_vec_offset += query_length * cache_config_.embedding_vec_size_[i];
    }

    // Copy the missing embeddingcolumns to host
    for(unsigned int i = 0; i < cache_config_.num_emb_table_; i++){
      TypeHashKey* d_missing_key_ptr = (TypeHashKey*)(workspace_handler.d_missing_embeddingcolumns_) + workspace_handler.h_shuffled_embedding_offset_[i];
      TypeHashKey* h_missing_key_ptr = (TypeHashKey*)(workspace_handler.h_missing_embeddingcolumns_) + workspace_handler.h_shuffled_embedding_offset_[i];
      CK_CUDA_THROW_(hipStreamSynchronize(streams[i]));
      CK_CUDA_THROW_(hipMemcpyAsync(h_missing_key_ptr, d_missing_key_ptr, workspace_handler.h_missing_length_[i] * sizeof(TypeHashKey), hipMemcpyDeviceToHost, streams[i]));
    }
    
    // Query the missing embeddingcolumns from Parameter Server
    acc_emb_vec_offset = 0;
    for(unsigned int i = 0; i < cache_config_.num_emb_table_; i++){
      TypeHashKey* h_missing_key_ptr = (TypeHashKey*)(workspace_handler.h_missing_embeddingcolumns_) + workspace_handler.h_shuffled_embedding_offset_[i];
      size_t query_length = workspace_handler.h_shuffled_embedding_offset_[i + 1] - workspace_handler.h_shuffled_embedding_offset_[i];
      float* h_vals_retrieved_ptr = workspace_handler.h_missing_emb_vec_ + acc_emb_vec_offset;
      CK_CUDA_THROW_(hipStreamSynchronize(streams[i]));
      parameter_server_ -> look_up(h_missing_key_ptr, workspace_handler.h_missing_length_[i], h_vals_retrieved_ptr, cache_config_.model_name_, i);
      acc_emb_vec_offset += query_length * cache_config_.embedding_vec_size_[i];
    }

    //Copy missing emb_vec to device 
    acc_emb_vec_offset = 0;
    for(unsigned int i = 0; i < cache_config_.num_emb_table_; i++){
      float* h_vals_retrieved_ptr = workspace_handler.h_missing_emb_vec_ + acc_emb_vec_offset;
      float* d_vals_retrieved_ptr = workspace_handler.d_missing_emb_vec_ + acc_emb_vec_offset;
      size_t missing_len_in_float = workspace_handler.h_missing_length_[i] * cache_config_.embedding_vec_size_[i];
      size_t missing_len_in_byte = missing_len_in_float * sizeof(float); 
      size_t query_length = workspace_handler.h_shuffled_embedding_offset_[i + 1] - workspace_handler.h_shuffled_embedding_offset_[i];
      acc_emb_vec_offset += query_length * cache_config_.embedding_vec_size_[i];
      CK_CUDA_THROW_(hipMemcpyAsync(d_vals_retrieved_ptr, h_vals_retrieved_ptr, missing_len_in_byte, hipMemcpyHostToDevice, streams[i]));
    }

    //Merge missing emb_vec into output
    acc_emb_vec_offset = 0;
    for(unsigned int i = 0; i < cache_config_.num_emb_table_; i++){
      float* d_vals_retrieved_ptr = workspace_handler.d_missing_emb_vec_ + acc_emb_vec_offset;
      float* d_vals_merge_dst_ptr = d_shuffled_embeddingoutputvector + acc_emb_vec_offset;
      uint64_t* d_missing_index_ptr = workspace_handler.d_missing_index_ + workspace_handler.h_shuffled_embedding_offset_[i];
      size_t missing_len_in_float = workspace_handler.h_missing_length_[i] * cache_config_.embedding_vec_size_[i];
      size_t query_length = workspace_handler.h_shuffled_embedding_offset_[i + 1] - workspace_handler.h_shuffled_embedding_offset_[i];
      acc_emb_vec_offset += query_length * cache_config_.embedding_vec_size_[i];
      //Wait for memory copy to complete
      CK_CUDA_THROW_(hipStreamSynchronize(streams[i]));
      merge_emb_vec<<<((missing_len_in_float - 1) / BLOCK_SIZE_) + 1, BLOCK_SIZE_, 0, streams[i]>>>(d_vals_merge_dst_ptr, 
                                                                                                    d_vals_retrieved_ptr, 
                                                                                                    d_missing_index_ptr, 
                                                                                                    workspace_handler.h_missing_length_[i],
                                                                                                    cache_config_.embedding_vec_size_[i]);
    }
  }
  else{
    //Query the shuffled embeddingcolumns from Parameter Server & copy to device output buffer
    size_t acc_emb_vec_offset = 0;
    for(unsigned int i = 0; i < cache_config_.num_emb_table_; i++){
      TypeHashKey* h_query_key_ptr = (TypeHashKey*)(workspace_handler.h_shuffled_embeddingcolumns_) + workspace_handler.h_shuffled_embedding_offset_[i];
      size_t query_length = workspace_handler.h_shuffled_embedding_offset_[i + 1] - workspace_handler.h_shuffled_embedding_offset_[i];
      size_t query_length_in_float = query_length * cache_config_.embedding_vec_size_[i];
      size_t query_length_in_byte = query_length_in_float * sizeof(float);
      float* h_vals_retrieved_ptr = workspace_handler.h_missing_emb_vec_ + acc_emb_vec_offset;
      float* d_vals_retrieved_ptr = d_shuffled_embeddingoutputvector + acc_emb_vec_offset;
      acc_emb_vec_offset += query_length_in_float;
      parameter_server_ -> look_up(h_query_key_ptr, query_length, h_vals_retrieved_ptr, cache_config_.model_name_, i);
      CK_CUDA_THROW_(hipMemcpyAsync(d_vals_retrieved_ptr, h_vals_retrieved_ptr, query_length_in_byte, hipMemcpyHostToDevice, streams[i]));
    }
  }

}

template <typename TypeHashKey>
void embedding_cache<TypeHashKey>::update(embedding_cache_workspace& workspace_handler, 
                                          const std::vector<hipStream_t>& streams){
  // If GPU embedding cache is enabled
  if(cache_config_.use_gpu_embedding_cache_){
    // Device Restorer
    CudaDeviceContext dev_restorer;
    // Set CUDA device before doing update
    CK_CUDA_THROW_(hipSetDevice(cache_config_.cuda_dev_id_));
    size_t acc_emb_vec_offset = 0;
    for(unsigned int i = 0; i < cache_config_.num_emb_table_; i++){
      TypeHashKey* d_missing_key_ptr = (TypeHashKey*)(workspace_handler.d_missing_embeddingcolumns_) + workspace_handler.h_shuffled_embedding_offset_[i];
      float* d_vals_retrieved_ptr = workspace_handler.d_missing_emb_vec_ + acc_emb_vec_offset;
      size_t query_length = workspace_handler.h_shuffled_embedding_offset_[i + 1] - workspace_handler.h_shuffled_embedding_offset_[i];
      acc_emb_vec_offset += query_length * cache_config_.embedding_vec_size_[i];
      gpu_emb_caches_[i] -> Replace(d_missing_key_ptr, 
                                    workspace_handler.h_missing_length_[i], 
                                    d_vals_retrieved_ptr, 
                                    streams[i]);
    }
  }
}

template <typename TypeHashKey>
void embedding_cache<TypeHashKey>::create_workspace(embedding_cache_workspace& workspace_handler){
  size_t max_query_len_per_batch = 0;
  size_t max_emb_vec_len_per_batch_in_float = 0;
  for(unsigned int i = 0; i < cache_config_.num_emb_table_; i++){
    max_query_len_per_batch += cache_config_.max_query_len_per_emb_table_[i];
    max_emb_vec_len_per_batch_in_float += (cache_config_.max_query_len_per_emb_table_[i] * cache_config_.embedding_vec_size_[i]);
  }
  // Allocate common buffer
  CK_CUDA_THROW_(hipHostAlloc((void**)&workspace_handler.h_shuffled_embeddingcolumns_, 
                               max_query_len_per_batch * sizeof(TypeHashKey), 
                               hipHostMallocPortable));
  CK_CUDA_THROW_(hipHostAlloc((void**)&workspace_handler.h_shuffled_embedding_offset_, 
                               (cache_config_.num_emb_table_ + 1) * sizeof(size_t), 
                               hipHostMallocPortable));
  CK_CUDA_THROW_(hipHostAlloc((void**)&workspace_handler.h_missing_emb_vec_, 
                               max_emb_vec_len_per_batch_in_float * sizeof(float), 
                               hipHostMallocPortable));

  // If GPU embedding cache is enabled
  if(cache_config_.use_gpu_embedding_cache_){
    // Device Restorer
    CudaDeviceContext dev_restorer;
    // Set CUDA device before creating workspace buffer
    CK_CUDA_THROW_(hipSetDevice(cache_config_.cuda_dev_id_));

    CK_CUDA_THROW_(hipMalloc((void**)&workspace_handler.d_shuffled_embeddingcolumns_, 
                              max_query_len_per_batch * sizeof(TypeHashKey)));
    CK_CUDA_THROW_(hipMalloc((void**)&workspace_handler.d_missing_embeddingcolumns_, 
                              max_query_len_per_batch * sizeof(TypeHashKey)));
    CK_CUDA_THROW_(hipHostAlloc((void**)&workspace_handler.h_missing_embeddingcolumns_, 
                                 max_query_len_per_batch * sizeof(TypeHashKey), 
                                 hipHostMallocPortable));
    CK_CUDA_THROW_(hipMalloc((void**)&workspace_handler.d_missing_length_, 
                              cache_config_.num_emb_table_ * sizeof(size_t)));
    CK_CUDA_THROW_(hipHostAlloc((void**)&workspace_handler.h_missing_length_, 
                                 cache_config_.num_emb_table_ * sizeof(size_t), 
                                 hipHostMallocPortable));
    CK_CUDA_THROW_(hipMalloc((void**)&workspace_handler.d_missing_index_, 
                              max_query_len_per_batch * sizeof(uint64_t)));
    CK_CUDA_THROW_(hipMalloc((void**)&workspace_handler.d_missing_emb_vec_, 
                              max_emb_vec_len_per_batch_in_float * sizeof(float)));
  }
}

template <typename TypeHashKey>
void embedding_cache<TypeHashKey>::destroy_workspace(embedding_cache_workspace& workspace_handler){
  // Free common buffer
  CK_CUDA_THROW_(hipHostFree(workspace_handler.h_shuffled_embeddingcolumns_));
  CK_CUDA_THROW_(hipHostFree(workspace_handler.h_shuffled_embedding_offset_));
  CK_CUDA_THROW_(hipHostFree(workspace_handler.h_missing_emb_vec_));
  // If GPU embedding cache is enabled
  if(cache_config_.use_gpu_embedding_cache_){
    // Device Restorer
    CudaDeviceContext dev_restorer;
    // Set CUDA device before free workspace buffer
    CK_CUDA_THROW_(hipSetDevice(cache_config_.cuda_dev_id_));
    
    CK_CUDA_THROW_(hipFree(workspace_handler.d_shuffled_embeddingcolumns_));
    CK_CUDA_THROW_(hipFree(workspace_handler.d_missing_embeddingcolumns_));
    CK_CUDA_THROW_(hipHostFree(workspace_handler.h_missing_embeddingcolumns_));
    CK_CUDA_THROW_(hipFree(workspace_handler.d_missing_length_));
    CK_CUDA_THROW_(hipHostFree(workspace_handler.h_missing_length_));
    CK_CUDA_THROW_(hipFree(workspace_handler.d_missing_index_));
    CK_CUDA_THROW_(hipFree(workspace_handler.d_missing_emb_vec_));
  }
}

template class embedding_cache<unsigned int>;
template class embedding_cache<long long>;
}  // namespace HugeCTR
