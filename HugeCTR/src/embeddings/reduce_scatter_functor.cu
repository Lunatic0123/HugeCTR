#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "HugeCTR/include/embeddings/sparse_embedding_functors.hpp"

namespace HugeCTR {

template <typename TypeEmbeddingComp>
void SparseEmbeddingFunctors::reduce_scatter(size_t recv_count,
                                             const Tensors2<TypeEmbeddingComp> &send_tensors,
                                             Tensors2<TypeEmbeddingComp> &recv_tensors,
                                             const GPUResourceGroup &device_resources) {
  size_t local_gpu_count = device_resources.size();
  size_t total_gpu_count = device_resources.get_total_gpu_count();

  // need to know the type of TypeHashKey here
  ncclDataType_t type;
  switch (sizeof(TypeEmbeddingComp)) {
    case 2:
      type = ncclHalf;
      break;
    case 4:
      type = ncclFloat;
      break;
    default:
      CK_THROW_(Error_t::WrongInput, "Error: TypeHashKey not support by now");
  }

  // for multi GPUs, use NCCL to do Reduce-Scatter(supporting multi-node GPU servers)
  if (total_gpu_count > 1) {
    CK_NCCL_THROW_(ncclGroupStart());
    for (size_t id = 0; id < local_gpu_count; id++) {
      CK_NCCL_THROW_(ncclReduceScatter(send_tensors[id].get_ptr(),  // send buf
                                       recv_tensors[id].get_ptr(),  // recv buff
                                       recv_count, type, ncclSum, device_resources[id].get_nccl(),
                                       device_resources[id].get_stream()));
    }
    CK_NCCL_THROW_(ncclGroupEnd());
  }
  // for single GPU, just do memcpyD2D
  else {  // total_gpu_count == 1
    CudaDeviceContext context(device_resources[0].get_device_id());
    CK_CUDA_THROW_(hipMemcpyAsync(recv_tensors[0].get_ptr(), send_tensors[0].get_ptr(),
                                   recv_count * sizeof(TypeEmbeddingComp), hipMemcpyDeviceToDevice,
                                   device_resources[0].get_stream()));
  }

  return;
}

template void SparseEmbeddingFunctors::reduce_scatter<float>(
    size_t recv_count, const Tensors2<float> &send_tensors, Tensors2<float> &recv_tensors,
    const GPUResourceGroup &device_resources);

template void SparseEmbeddingFunctors::reduce_scatter<__half>(
    size_t recv_count, const Tensors2<__half> &send_tensors, Tensors2<__half> &recv_tensors,
    const GPUResourceGroup &device_resources);

}  // namespace HugeCTR