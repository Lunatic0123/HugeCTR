#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "HugeCTR/include/embeddings/localized_slot_sparse_embedding_one_hot.hpp"

namespace HugeCTR {

namespace {

template <typename value_type>
__global__ void upload_value_tensor_kernel(value_type *value_buf, size_t *index_buf,
                                           value_type *dst_tensor, int emb_vec_size, size_t len) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < len) {
    size_t src_offset = gid * emb_vec_size;
    size_t dst_offset = index_buf[gid] * emb_vec_size;
    for (int i = 0; i < emb_vec_size; i++) {
      dst_tensor[dst_offset + i] = value_buf[src_offset + i];
    }
  }
}

}  // namespace

template <typename TypeHashKey, typename TypeEmbeddingComp>
LocalizedSlotSparseEmbeddingOneHot<TypeHashKey, TypeEmbeddingComp>::
    LocalizedSlotSparseEmbeddingOneHot(
        const Tensors2<TypeHashKey> &train_row_offsets_tensors,
        const Tensors2<TypeHashKey> &train_value_tensors,
        const std::vector<std::shared_ptr<size_t>> &train_nnz_array,
        const Tensors2<TypeHashKey> &evaluate_row_offsets_tensors,
        const Tensors2<TypeHashKey> &evaluate_value_tensors,
        const std::vector<std::shared_ptr<size_t>> &evaluate_nnz_array,
        const SparseEmbeddingHashParams<TypeEmbeddingComp> &embedding_params,
        const std::string plan_file, const std::shared_ptr<ResourceManager> &resource_manager)
    : Base(train_row_offsets_tensors, train_value_tensors, train_nnz_array,
           evaluate_row_offsets_tensors, evaluate_value_tensors, evaluate_nnz_array,
           Embedding_t::LocalizedSlotSparseEmbeddingOneHot, embedding_params, resource_manager),
      slot_size_array_(embedding_params.slot_size_array) {
  try {
    max_vocabulary_size_ = 0;
    for (size_t slot_size : slot_size_array_) {
      max_vocabulary_size_ += slot_size;
    }

    max_vocabulary_size_per_gpu_ =
        cal_max_voc_size_per_gpu(slot_size_array_, Base::get_resource_manager());

    MESSAGE_("max_vocabulary_size_per_gpu_=" + std::to_string(max_vocabulary_size_per_gpu_));

    CudaDeviceContext context;
    for (size_t id = 0; id < Base::get_resource_manager().get_local_gpu_count(); id++) {
      context.set_device(Base::get_local_gpu(id).get_device_id());

      size_t gid = Base::get_local_gpu(id).get_global_id();
      size_t slot_num_per_gpu =
          Base::get_slot_num() / Base::get_resource_manager().get_global_gpu_count() +
          ((gid < Base::get_slot_num() % Base::get_resource_manager().get_global_gpu_count()) ? 1
                                                                                              : 0);
      slot_num_per_gpu_.push_back(slot_num_per_gpu);

      // new GeneralBuffer objects
      const std::shared_ptr<GeneralBuffer2<CudaAllocator>> &buf = Base::get_buffer(id);

      // new hash table value vectors
      {
        const std::shared_ptr<BufferBlock2<float>> &block = buf->create_block<float>();
        Tensors2<float> tensors;
        for (size_t i = 0; i < slot_size_array_.size(); i++) {
          if ((i % Base::get_resource_manager().get_global_gpu_count()) == gid) {
            Tensor2<float> tensor;
            block->reserve({slot_size_array_[i], Base::get_embedding_vec_size()}, &tensor);
            tensors.push_back(tensor);
          }
        }
        value_table_tensors_.push_back(tensors);
        hash_table_value_tensors_.push_back(block->as_tensor());
      }

      // new hash table value_index that get() from HashTable
      {
        Tensor2<size_t> tensor;
        buf->reserve({1, Base::get_universal_batch_size() * Base::get_max_feature_num()}, &tensor);
        hash_value_index_tensors_.push_back(tensor);
      }

      // new embedding features reduced by hash table values(results of forward)
      {
        Tensor2<TypeEmbeddingComp> tensor;
        buf->reserve(
            {Base::get_universal_batch_size() * slot_num_per_gpu, Base::get_embedding_vec_size()},
            &tensor);
        embedding_feature_tensors_.push_back(tensor);
      }

      // new wgrad used by backward
      {
        Tensor2<TypeEmbeddingComp> tensor;
        buf->reserve(
            {Base::get_batch_size(true) * slot_num_per_gpu, Base::get_embedding_vec_size()},
            &tensor);
        wgrad_tensors_.push_back(tensor);
      }

      // new optimizer params used by update_params
      switch (Base::get_optimizer()) {
        case Optimizer_t::SGD:
          break;

        default:
          throw std::runtime_error(
              std::string("[HCDEBUG][ERROR] Runtime error: Invalid optimizer type\n"));
      }

      // the tenosrs for storing slot ids
      // TODO: init to -1 ?
      {
        Tensor2<size_t> tensor;
        buf->reserve({max_vocabulary_size_per_gpu_, 1}, &tensor);
        hash_table_slot_id_tensors_.push_back(tensor);
      }

      // temp tensors for all2all
      {
        Tensor2<TypeEmbeddingComp> tensor;
        buf->reserve({Base::get_universal_batch_size_per_gpu() * Base::get_slot_num(),
                      Base::get_embedding_vec_size()},
                     &tensor);
        all2all_tensors_.push_back(tensor);
      }
      {
        Tensor2<TypeEmbeddingComp> tensor;
        buf->reserve({Base::get_universal_batch_size() * Base::get_slot_num(),
                      Base::get_embedding_vec_size()},
                     &tensor);
        utest_forward_temp_tensors_.push_back(tensor);
      }
      {
        Tensor2<TypeEmbeddingComp> tensor;
        buf->reserve({Base::get_batch_size_per_gpu(true) * Base::get_slot_num(),
                      Base::get_embedding_vec_size()},
                     &tensor);
        utest_all2all_tensors_.push_back(tensor);
      }
      {
        Tensor2<TypeEmbeddingComp> tensor;
        buf->reserve({Base::get_batch_size_per_gpu(true) * Base::get_slot_num(),
                      Base::get_embedding_vec_size()},
                     &tensor);
        utest_reorder_tensors_.push_back(tensor);
      }
      {
        Tensor2<TypeEmbeddingComp> tensor;
        buf->reserve(
            {Base::get_batch_size(true) * Base::get_slot_num(), Base::get_embedding_vec_size()},
            &tensor);
        utest_backward_temp_tensors_.push_back(tensor);
      }
      {
        Tensor2<uint32_t> tensor;
        buf->reserve({1, slot_num_per_gpu}, &tensor);
        mapping_offsets_per_gpu_tensors_.push_back(tensor);
      }

// init GenenralBuffers to do real allocation
#ifndef NDEBUG
      std::cout << " max_feature_num_:" << Base::get_max_feature_num() << std::endl;
#endif

      const OptParams<TypeEmbeddingComp> &source_opt_param = Base::get_opt_params();
      OptParams<TypeEmbeddingComp> &target_opt_param = Base::get_opt_params(id);

      switch (Base::get_optimizer()) {
        case Optimizer_t::SGD:
          target_opt_param.hyperparams.sgd.atomic_update =
              source_opt_param.hyperparams.sgd.atomic_update;

          break;

        default:
          throw std::runtime_error(
              std::string("[HCDEBUG][ERROR] Runtime error: Invalid optimizer type\n"));
      }

    }  // end of for(int id = 0; id < Base::get_local_gpu_count(); id++)

#pragma omp parallel num_threads(Base::get_resource_manager().get_local_gpu_count())
    {
      size_t id = omp_get_thread_num();
      CudaDeviceContext context(Base::get_local_gpu(id).get_device_id());
      Base::get_buffer(id)->allocate();
      CK_CUDA_THROW_(hipStreamSynchronize(Base::get_local_gpu(id).get_stream()));
    }

    // get the mapping table between local value_index and input value_index
    for (size_t id = 0; id < Base::get_resource_manager().get_local_gpu_count(); id++) {
      context.set_device(Base::get_local_gpu(id).get_device_id());
      uint32_t slot_sizes_prefix_sum = 0;
      uint32_t slot_sizes_prefix_sum_local = 0;
      int slot_num = 0;
      for (size_t i = 0; i < slot_size_array_.size(); i++) {
        size_t global_id = Base::get_local_gpu(id).get_global_id();
        size_t slot_size = slot_size_array_[i];
        if (i % Base::get_resource_manager().get_global_gpu_count() == global_id) {
          uint32_t mapping_offset = slot_sizes_prefix_sum - slot_sizes_prefix_sum_local;
          CK_CUDA_THROW_(hipMemcpy(&((mapping_offsets_per_gpu_tensors_[id].get_ptr())[slot_num]),
                                    &mapping_offset, sizeof(uint32_t), hipMemcpyHostToDevice));
          slot_sizes_prefix_sum_local += slot_size;
          slot_num++;
        }
        slot_sizes_prefix_sum += slot_size;
      }
    }

    // Check whether the P2P access can be enabled
    if (Base::get_resource_manager().get_local_gpu_count() > 1 &&
        !Base::get_resource_manager().all_p2p_enabled()) {
      throw std::runtime_error(
          std::string("[HCDEBUG][ERROR] Runtime error: Localized_slot_sparse_embedding_one_hot "
                      "cannot be used on machine without GPU peer2peer access support. \n"));
    }
#ifdef ENABLE_MPI
    throw std::runtime_error(
        std::string("[HCDEBUG][ERROR] Runtime error: Localized_slot_sparse_embedding_one_hot "
                    "cannot support multi-node currently. \n"));
#endif

    std::shared_ptr<GeneralBuffer2<CudaManagedAllocator>> unified_buf =
        GeneralBuffer2<CudaManagedAllocator>::create();
    unified_buf->reserve({Base::get_resource_manager().get_local_gpu_count()},
                         &train_embedding_features_);
    unified_buf->reserve({Base::get_resource_manager().get_local_gpu_count()},
                         &evaluate_embedding_features_);
    unified_buf->allocate();

    for (size_t id = 0; id < Base::get_resource_manager().get_local_gpu_count(); id++) {
      train_embedding_features_.get_ptr()[id] = Base::get_output_tensors(true)[id].get_ptr();
      evaluate_embedding_features_.get_ptr()[id] = Base::get_output_tensors(false)[id].get_ptr();
    }

  } catch (const std::runtime_error &rt_err) {
    std::cerr << rt_err.what() << std::endl;
    throw;
  }

  return;
}

template <typename TypeHashKey, typename TypeEmbeddingComp>
void LocalizedSlotSparseEmbeddingOneHot<TypeHashKey, TypeEmbeddingComp>::load_parameters(
    std::ifstream &stream) {
  // check if file is opened successfully
  if (!stream.is_open()) {
    CK_THROW_(Error_t::WrongInput, "Error: file not open for reading");
  }

  stream.seekg(0, stream.end);
  size_t file_size = stream.tellg();
  stream.seekg(0, stream.beg);

  size_t row_size =
      sizeof(TypeHashKey) + sizeof(size_t) + sizeof(float) * Base::get_embedding_vec_size();
  size_t row_num = file_size / row_size;

  if (file_size % row_size != 0) {
    CK_THROW_(Error_t::WrongInput, "Error: file size is not correct");
  }

  std::shared_ptr<GeneralBuffer2<CudaHostAllocator>> blobs_buff =
      GeneralBuffer2<CudaHostAllocator>::create();

  Tensor2<TypeHashKey> keys;
  blobs_buff->reserve({row_num}, &keys);

  Tensor2<size_t> slot_id;
  blobs_buff->reserve({row_num}, &slot_id);

  Tensor2<float> embeddings;
  blobs_buff->reserve({row_num, Base::get_embedding_vec_size()}, &embeddings);

  blobs_buff->allocate();

  TypeHashKey *key_ptr = keys.get_ptr();
  size_t *slot_id_ptr = slot_id.get_ptr();
  float *embedding_ptr = embeddings.get_ptr();

  for (size_t i = 0; i < row_num; i++) {
    stream.read(reinterpret_cast<char *>(key_ptr + i), sizeof(TypeHashKey));
    stream.read(reinterpret_cast<char *>(slot_id_ptr + i), sizeof(size_t));
    stream.read(reinterpret_cast<char *>(embedding_ptr + i * Base::get_embedding_vec_size()),
                sizeof(float) * Base::get_embedding_vec_size());
  }

  load_parameters(keys, slot_id, embeddings, row_num, Base::get_embedding_vec_size(),
                  hash_table_value_tensors_, slot_size_array_, mapping_offsets_per_gpu_tensors_);

  return;
}

template <typename TypeHashKey, typename TypeEmbeddingComp>
void LocalizedSlotSparseEmbeddingOneHot<TypeHashKey, TypeEmbeddingComp>::load_parameters(
    BufferBag &buf_bag, size_t num) {
  const TensorBag2 &keys_bag = buf_bag.keys;
  const TensorBag2 &slot_id_bag = buf_bag.slot_id;
  const Tensor2<float> &embeddings = buf_bag.embedding;
  Tensor2<TypeHashKey> keys = Tensor2<TypeHashKey>::stretch_from(keys_bag);
  Tensor2<size_t> slot_id = Tensor2<size_t>::stretch_from(slot_id_bag);

  load_parameters(keys, slot_id, embeddings, num, Base::get_embedding_vec_size(),
                  hash_table_value_tensors_, slot_size_array_, mapping_offsets_per_gpu_tensors_);
}

template <typename TypeHashKey, typename TypeEmbeddingComp>
void LocalizedSlotSparseEmbeddingOneHot<TypeHashKey, TypeEmbeddingComp>::load_parameters(
    const Tensor2<TypeHashKey> &keys, const Tensor2<size_t> &slot_id,
    const Tensor2<float> &embeddings, size_t num, size_t embedding_vec_size,
    Tensors2<float> &hash_table_value_tensors, const std::vector<size_t> &slot_sizes,
    const Tensors2<uint32_t> &mapping_offsets_per_gpu_tensors) {
  CudaDeviceContext context;
  if (keys.get_dimensions()[0] < num || embeddings.get_dimensions()[0] < num) {
    CK_THROW_(Error_t::WrongInput, "The rows of keys and embeddings are not consistent.");
  }

  const TypeHashKey *key_ptr = keys.get_ptr();
  const size_t *slot_id_ptr = slot_id.get_ptr();
  const float *embedding_ptr = embeddings.get_ptr();

  // define size
  size_t local_gpu_count = Base::get_resource_manager().get_local_gpu_count();
  size_t chunk_size = 1000;
  size_t tile_size = 1;  // must be 1, because we need to cal (key&local_gpu_count) to decide
                         // gpu_id for each <key,value>
  size_t hash_table_value_tile_size = tile_size * embedding_vec_size;
  size_t hash_table_value_tile_size_in_B = hash_table_value_tile_size * sizeof(float);
  size_t hash_table_value_chunk_size = hash_table_value_tile_size * chunk_size;
  size_t hash_table_value_chunk_size_in_B = hash_table_value_chunk_size * sizeof(float);
  size_t total_gpu_count = Base::get_resource_manager().get_global_gpu_count();

  // CAUSION: can not decide how many values for each GPU, so need to allocate enough memory for
  // each GPU allocate CPU/GPU memory for value/index chunk
  std::unique_ptr<float *[]> h_hash_table_value_chunk_per_gpu(new float *[local_gpu_count]);
  for (size_t id = 0; id < local_gpu_count; id++) {
    CK_CUDA_THROW_(
        hipHostMalloc(&h_hash_table_value_chunk_per_gpu[id], hash_table_value_chunk_size_in_B));
  }
  std::unique_ptr<float *[]> d_hash_table_value_chunk_per_gpu(new float *[local_gpu_count]);
  for (size_t id = 0; id < local_gpu_count; id++) {
    context.set_device(Base::get_local_gpu(id).get_device_id());
    CK_CUDA_THROW_(
        hipMalloc(&d_hash_table_value_chunk_per_gpu[id], hash_table_value_chunk_size_in_B));
  }
  std::unique_ptr<size_t *[]> h_hash_table_index_chunk_per_gpu(new size_t *[local_gpu_count]);
  for (size_t id = 0; id < local_gpu_count; id++) {
    CK_CUDA_THROW_(
        hipHostMalloc(&h_hash_table_index_chunk_per_gpu[id], chunk_size * sizeof(size_t)));
  }
  std::unique_ptr<size_t *[]> d_hash_table_index_chunk_per_gpu(new size_t *[local_gpu_count]);
  for (size_t id = 0; id < local_gpu_count; id++) {
    context.set_device(Base::get_local_gpu(id).get_device_id());
    CK_CUDA_THROW_(hipMalloc(&d_hash_table_index_chunk_per_gpu[id], chunk_size * sizeof(size_t)));
  }

  std::unique_ptr<size_t[]> tile_counter_in_chunk_per_gpu(new size_t[local_gpu_count]);
  memset(tile_counter_in_chunk_per_gpu.get(), 0, sizeof(size_t) * local_gpu_count);

  // The vector that store the relationship between slot_id and slot order on the specific GPU
  std::vector<size_t> local_slot_id(slot_sizes.size());
  std::vector<size_t> local_slot_num(local_gpu_count, 0);
  for (size_t i = 0; i < slot_sizes.size(); i++) {
    size_t gid = i % total_gpu_count;  // global GPU ID
    size_t id = Base::get_resource_manager().get_gpu_local_id_from_global_id(
        gid);  // local GPU ID (not gpudevice id)
    int dst_rank = Base::get_resource_manager().get_process_id_from_gpu_global_id(gid);  // node id
    if (Base::get_resource_manager().get_process_id() == dst_rank) {
      local_slot_id[i] = local_slot_num[id];
      local_slot_num[id]++;
    }
  }

  // Host buffer to keep mapping_offset
  std::vector<uint32_t *> h_mapping_offsets_per_gpu_tensors(local_gpu_count);
  for (size_t id = 0; id < local_gpu_count; id++) {
    context.set_device(Base::get_local_gpu(id).get_device_id());
    CK_CUDA_THROW_(hipHostMalloc(&h_mapping_offsets_per_gpu_tensors[id],
                                  local_slot_num[id] * sizeof(uint32_t)));
    // Copy the mapping offset from GPU to Host
    hipMemcpyAsync(h_mapping_offsets_per_gpu_tensors[id],
                    mapping_offsets_per_gpu_tensors[id].get_ptr(),
                    local_slot_num[id] * sizeof(uint32_t), hipMemcpyDeviceToHost,
                    Base::get_local_gpu(id).get_stream());
  }

  // sync wait
  functors_.sync_all_gpus(Base::get_resource_manager());

  // do upload
  size_t loop_num = num / chunk_size;
  MESSAGE_("Start to upload embedding table file to GPUs, total loop_num: " +
           std::to_string(loop_num));
  for (size_t i = 0; i < loop_num; i++) {
    float *value_dst_buf;
    size_t *tensor_index_dst_buf;
    for (size_t k = 0; k < chunk_size; k++) {  // process a tile in each loop
      size_t slot_id = slot_id_ptr[i * chunk_size + k];
      size_t gid = slot_id % total_gpu_count;  // global GPU ID
      size_t id = Base::get_resource_manager().get_gpu_local_id_from_global_id(
          gid);  // local GPU ID (not gpudevice id)
      int dst_rank =
          Base::get_resource_manager().get_process_id_from_gpu_global_id(gid);  // node id

      if (Base::get_resource_manager().get_process_id() == dst_rank) {
        TypeHashKey tile_key = key_ptr[i * chunk_size + k];
        size_t tensor_index =
            tile_key - (h_mapping_offsets_per_gpu_tensors[id][local_slot_id[slot_id]]);

        // memcpy hash_table_value to corresponding GPU
        value_dst_buf = h_hash_table_value_chunk_per_gpu[id] +
                        tile_counter_in_chunk_per_gpu[id] * hash_table_value_tile_size;
        memcpy(value_dst_buf, embedding_ptr + (i * chunk_size + k) * embedding_vec_size,
               hash_table_value_tile_size_in_B);

        tensor_index_dst_buf =
            h_hash_table_index_chunk_per_gpu[id] + tile_counter_in_chunk_per_gpu[id];
        *tensor_index_dst_buf = tensor_index;
        tile_counter_in_chunk_per_gpu[id] += 1;
      } else {
        continue;
      }
    }  // end of for(int k = 0; k < (chunk_size * local_gpu_count); k++)

    // memcpy hash_table_slot_id and hash_table_value from CPU to GPU
    for (size_t id = 0; id < local_gpu_count; id++) {
      if (tile_counter_in_chunk_per_gpu[id] == 0) {
        continue;
      }

      context.set_device(Base::get_local_gpu(id).get_device_id());

      // Copy value buffer and tensor_index buffer to GPU
      size_t value_chunk_size = tile_counter_in_chunk_per_gpu[id] * hash_table_value_tile_size;
      float *src_buf_value = h_hash_table_value_chunk_per_gpu[id];
      float *dst_buf_value = d_hash_table_value_chunk_per_gpu[id];
      CK_CUDA_THROW_(hipMemcpyAsync(dst_buf_value, src_buf_value, value_chunk_size * sizeof(float),
                                     hipMemcpyHostToDevice, Base::get_local_gpu(id).get_stream()));
      size_t *src_buf_index = h_hash_table_index_chunk_per_gpu[id];
      size_t *dst_buf_index = d_hash_table_index_chunk_per_gpu[id];
      value_chunk_size = tile_counter_in_chunk_per_gpu[id];
      CK_CUDA_THROW_(hipMemcpyAsync(dst_buf_index, src_buf_index,
                                     value_chunk_size * sizeof(size_t), hipMemcpyHostToDevice,
                                     Base::get_local_gpu(id).get_stream()));

      // Call kernel to insert the value into embedding value tensor
      const size_t grid_size = (tile_counter_in_chunk_per_gpu[id] - 1) / 256 + 1;
      upload_value_tensor_kernel<<<grid_size, 256, 0, Base::get_local_gpu(id).get_stream()>>>(
          d_hash_table_value_chunk_per_gpu[id], d_hash_table_index_chunk_per_gpu[id],
          hash_table_value_tensors[id].get_ptr(), hash_table_value_tile_size,
          tile_counter_in_chunk_per_gpu[id]);
    }

    functors_.sync_all_gpus(Base::get_resource_manager());

    // set counter value
    for (size_t id = 0; id < local_gpu_count; id++) {
      tile_counter_in_chunk_per_gpu[id] = 0;  // reset chunk counter to zero
    }

    /*       std::cout << "\rUploading " << std::fixed << std::setprecision(2)
                    << (float)(i) / loop_num * 100.0f << "%, loop " << i << " of " << loop_num
                    << std::flush; */
  }  // end of for(int i = 0; i < loop_num; i++)

  // std::cout << std::endl;

  // process the remaining data(less than a chunk)
  const size_t remain_loop_num = num - loop_num * chunk_size;
  float *value_dst_buf;
  size_t *tensor_index_dst_buf;
  for (size_t i = 0; i < remain_loop_num; i++) {  // process one tile in each loop

    size_t slot_id = slot_id_ptr[loop_num * chunk_size + i];
    size_t gid = slot_id % total_gpu_count;  // global GPU ID
    size_t id = Base::get_resource_manager().get_gpu_local_id_from_global_id(
        gid);  // local GPU ID (not gpudevice id)
    int dst_rank = Base::get_resource_manager().get_process_id_from_gpu_global_id(gid);  // node id

    if (Base::get_resource_manager().get_process_id() == dst_rank) {
      TypeHashKey tile_key = key_ptr[loop_num * chunk_size + i];
      size_t tensor_index =
          tile_key - (h_mapping_offsets_per_gpu_tensors[id][local_slot_id[slot_id]]);

      // memcpy hash_table_value to corresponding GPU
      value_dst_buf = h_hash_table_value_chunk_per_gpu[id] +
                      tile_counter_in_chunk_per_gpu[id] * hash_table_value_tile_size;
      memcpy(value_dst_buf, embedding_ptr + (loop_num * chunk_size + i) * embedding_vec_size,
             hash_table_value_tile_size_in_B);

      tensor_index_dst_buf =
          h_hash_table_index_chunk_per_gpu[id] + tile_counter_in_chunk_per_gpu[id];
      *tensor_index_dst_buf = tensor_index;
      tile_counter_in_chunk_per_gpu[id] += 1;

    } else {
      continue;
    }
  }

  // memcpy hash_table_slot_id and hash_table_value from CPU to GPU and insert into embedding
  // table
  for (size_t id = 0; id < local_gpu_count; id++) {
    if (tile_counter_in_chunk_per_gpu[id] == 0) {
      continue;
    }

    context.set_device(Base::get_local_gpu(id).get_device_id());

    // Copy value buffer and tensor_index buffer to GPU
    size_t value_chunk_size = tile_counter_in_chunk_per_gpu[id] * hash_table_value_tile_size;
    float *src_buf_value = h_hash_table_value_chunk_per_gpu[id];
    float *dst_buf_value = d_hash_table_value_chunk_per_gpu[id];
    CK_CUDA_THROW_(hipMemcpyAsync(dst_buf_value, src_buf_value, value_chunk_size * sizeof(float),
                                   hipMemcpyHostToDevice, Base::get_local_gpu(id).get_stream()));
    size_t *src_buf_index = h_hash_table_index_chunk_per_gpu[id];
    size_t *dst_buf_index = d_hash_table_index_chunk_per_gpu[id];
    value_chunk_size = tile_counter_in_chunk_per_gpu[id];
    CK_CUDA_THROW_(hipMemcpyAsync(dst_buf_index, src_buf_index, value_chunk_size * sizeof(size_t),
                                   hipMemcpyHostToDevice, Base::get_local_gpu(id).get_stream()));

    // Call kernel to insert the value into embedding value tensor
    const size_t grid_size = (tile_counter_in_chunk_per_gpu[id] - 1) / 256 + 1;
    upload_value_tensor_kernel<<<grid_size, 256, 0, Base::get_local_gpu(id).get_stream()>>>(
        d_hash_table_value_chunk_per_gpu[id], d_hash_table_index_chunk_per_gpu[id],
        hash_table_value_tensors[id].get_ptr(), hash_table_value_tile_size,
        tile_counter_in_chunk_per_gpu[id]);
  }

  // sync wait
  functors_.sync_all_gpus(Base::get_resource_manager());

  MESSAGE_("Done");

  // release resources
  for (size_t id = 0; id < local_gpu_count; id++) {
    context.set_device(Base::get_local_gpu(id).get_device_id());
    CK_CUDA_THROW_(hipFree(d_hash_table_value_chunk_per_gpu[id]));
    CK_CUDA_THROW_(hipFree(d_hash_table_index_chunk_per_gpu[id]));
  }
  for (size_t id = 0; id < local_gpu_count; id++) {
    CK_CUDA_THROW_(hipHostFree(h_hash_table_value_chunk_per_gpu[id]));
    CK_CUDA_THROW_(hipHostFree(h_hash_table_index_chunk_per_gpu[id]));
    CK_CUDA_THROW_(hipHostFree(h_mapping_offsets_per_gpu_tensors[id]));
  }
}

template <typename TypeHashKey, typename TypeEmbeddingComp>
void LocalizedSlotSparseEmbeddingOneHot<TypeHashKey, TypeEmbeddingComp>::dump_parameters(
    std::ofstream &stream) const {
  // check if the file is opened successfully
  if (!stream.is_open()) {
    CK_THROW_(Error_t::WrongInput, "Error: file not open for writing");
    return;
  }

  dump_parameters(stream, Base::get_embedding_vec_size(), hash_table_value_tensors_,
                  slot_size_array_);

  return;
}

template <typename TypeHashKey, typename TypeEmbeddingComp>
void LocalizedSlotSparseEmbeddingOneHot<TypeHashKey, TypeEmbeddingComp>::dump_parameters(
    BufferBag &buf_bag, size_t *num) const {
  TensorBag2 keys_bag = buf_bag.keys;
  TensorBag2 slot_id_bag = buf_bag.slot_id;
  Tensor2<float> &embeddings = buf_bag.embedding;
  Tensor2<TypeHashKey> keys = Tensor2<TypeHashKey>::stretch_from(keys_bag);
  Tensor2<size_t> slot_id = Tensor2<size_t>::stretch_from(slot_id_bag);

  dump_parameters(keys, slot_id, embeddings, num, Base::get_embedding_vec_size(),
                  hash_table_value_tensors_, slot_size_array_);
}

template <typename TypeHashKey, typename TypeEmbeddingComp>
void LocalizedSlotSparseEmbeddingOneHot<TypeHashKey, TypeEmbeddingComp>::dump_parameters(
    std::ofstream &weight_stream, size_t embedding_vec_size,
    const Tensors2<float> &hash_table_value_tensors, const std::vector<size_t> &slot_sizes) const {
  size_t local_gpu_count = Base::get_resource_manager().get_local_gpu_count();

  // memory allocation
  std::unique_ptr<size_t[]> count(new size_t[local_gpu_count]);
  size_t max_count = 0;
  size_t total_count = 0;

  CudaDeviceContext context;
  for (size_t id = 0; id < local_gpu_count; id++) {
    context.set_device(Base::get_local_gpu(id).get_device_id());
    count[id] = 0;
    for (size_t i = 0; i < slot_sizes.size(); i++) {
      size_t global_id = Base::get_local_gpu(id).get_global_id();
      if ((i % Base::get_resource_manager().get_global_gpu_count()) == global_id) {
        count[id] += slot_sizes[i];
      }
    }
    max_count = max(max_count, count[id]);
    total_count += count[id];
  }

#ifdef ENABLE_MPI
  CK_MPI_THROW_(
      MPI_Allreduce(MPI_IN_PLACE, &max_count, sizeof(size_t), MPI_CHAR, MPI_MAX, MPI_COMM_WORLD));
#endif

  /*if (total_count > (size_t)vocabulary_size) {
    CK_THROW_(Error_t::WrongInput,
              "Error: required download size is larger than hash table vocabulary_size");
  }*/

  std::unique_ptr<TypeHashKey *[]> h_hash_table_key(new TypeHashKey *[local_gpu_count]);
  std::unique_ptr<TypeHashKey *[]> d_hash_table_key(new TypeHashKey *[local_gpu_count]);
  std::unique_ptr<size_t *[]> h_hash_table_slot_id(new size_t *[local_gpu_count]);
  std::unique_ptr<size_t *[]> d_hash_table_slot_id(new size_t *[local_gpu_count]);
  std::unique_ptr<float *[]> h_hash_table_value(new float *[local_gpu_count]);
  for (size_t id = 0; id < local_gpu_count; id++) {
    if (count[id] == 0) {
      continue;
    }

    context.set_device(Base::get_local_gpu(id).get_device_id());

    hipHostMalloc(&h_hash_table_key[id], count[id] * sizeof(TypeHashKey));
    hipMalloc(&d_hash_table_key[id], count[id] * sizeof(TypeHashKey));
    hipHostMalloc(&h_hash_table_slot_id[id], count[id] * sizeof(size_t));
    hipMalloc(&d_hash_table_slot_id[id], count[id] * sizeof(size_t));
    hipHostMalloc(&h_hash_table_value[id], count[id] * embedding_vec_size * sizeof(float));
  }

  // Generate key and slot_id tensor, dump value tensor on GPU
  for (size_t id = 0; id < local_gpu_count; id++) {
    if (count[id] == 0) {
      continue;
    }

    MESSAGE_("Rank" + std::to_string(Base::get_resource_manager().get_process_id()) +
             ": Dump embedding table from GPU" + std::to_string(id));

    context.set_device(Base::get_local_gpu(id).get_device_id());

    // Loop for each slot
    size_t buffer_offset = 0;
    for (size_t i = 0; i < slot_sizes.size(); i++) {
      size_t global_id = Base::get_local_gpu(id).get_global_id();
      if ((i % Base::get_resource_manager().get_global_gpu_count()) == global_id) {
        // Generate key buffer
        size_t key_offset = 0;
        for (size_t j = 0; j < i; j++) {
          key_offset += slot_sizes[j];
        }
        functors_.memset_liner(d_hash_table_key[id] + buffer_offset, (TypeHashKey)key_offset,
                               (TypeHashKey)1, slot_sizes[i], Base::get_local_gpu(id).get_stream());

        // Generate slot_id
        functors_.memset_const(d_hash_table_slot_id[id] + buffer_offset, i, slot_sizes[i],
                               Base::get_local_gpu(id).get_stream());

        buffer_offset += slot_sizes[i];
      }
    }
    // Copy key buffer to host
    CK_CUDA_THROW_(hipMemcpyAsync(h_hash_table_key[id], d_hash_table_key[id],
                                   count[id] * sizeof(TypeHashKey), hipMemcpyDeviceToHost,
                                   Base::get_local_gpu(id).get_stream()));
    // Copy value buffer to host
    CK_CUDA_THROW_(hipMemcpyAsync(h_hash_table_value[id], hash_table_value_tensors[id].get_ptr(),
                                   count[id] * embedding_vec_size * sizeof(float),
                                   hipMemcpyDeviceToHost, Base::get_local_gpu(id).get_stream()));
    // Copy slot_id to host
    CK_CUDA_THROW_(hipMemcpyAsync(h_hash_table_slot_id[id], d_hash_table_slot_id[id],
                                   count[id] * sizeof(size_t), hipMemcpyDeviceToHost,
                                   Base::get_local_gpu(id).get_stream()));
  }

  // sync wait
  functors_.sync_all_gpus(Base::get_resource_manager());

#ifdef ENABLE_MPI
  const int base_tag = 0xed;
#endif
  // TODO: could be optimized ???
  // one pair in the file includes <key,slot_id,value>
  size_t pair_size_in_B = sizeof(TypeHashKey) + sizeof(size_t) + sizeof(float) * embedding_vec_size;
  size_t max_size_in_B = max_count * pair_size_in_B;
  std::unique_ptr<char[]> file_buf(new char[max_size_in_B]);
  size_t key_size = sizeof(TypeHashKey);
  size_t slot_id_size = sizeof(size_t);
  size_t value_size = sizeof(float) * embedding_vec_size;
  for (size_t id = 0; id < local_gpu_count; id++) {
    size_t size_in_B = count[id] * pair_size_in_B;
    size_t offset = 0;
    for (unsigned int k = 0; k < count[id]; k++) {
      /*         std::cout << "\rRank" << my_rank << ": Seperate keys, slot_ids and values on GPU"
         << id
                        << ", finish " << k << " of total count " << count[id] << ", "
                        << (float)k / count[id] * 100.0f << "%" << std::flush; */

      memcpy(file_buf.get() + offset, h_hash_table_key[id] + k, key_size);
      offset += key_size;
      memcpy(file_buf.get() + offset, h_hash_table_slot_id[id] + k, slot_id_size);
      offset += slot_id_size;
      memcpy(file_buf.get() + offset, h_hash_table_value[id] + k * embedding_vec_size, value_size);
      offset += value_size;
    }
    if (Base::get_resource_manager().is_master_process()) {
      MESSAGE_("Rank" + std::to_string(Base::get_resource_manager().get_process_id()) +
               ": Write hash table <key,value> pairs to file");
      weight_stream.write(file_buf.get(), size_in_B);
    }
#ifdef ENABLE_MPI
    else {
      MESSAGE_("Rank" + std::to_string(Base::get_resource_manager().get_process_id()) +
               ": Send hash table <key,value> pairs on GPU" + std::to_string(id) +
               " to master node  ");
      int tag = (id << 8) | base_tag;
      CK_MPI_THROW_(MPI_Send(file_buf.get(), size_in_B, MPI_CHAR,
                             Base::get_resource_manager().get_master_process_id(), tag,
                             MPI_COMM_WORLD));
    }
#endif
  }

#ifdef ENABLE_MPI
  if (Base::get_resource_manager().is_master_process()) {
    for (int r = 1; r < Base::get_resource_manager().get_num_process(); r++) {
      for (size_t id = 0; id < local_gpu_count; id++) {
        MESSAGE_("Rank" + std::to_string(Base::get_resource_manager().get_process_id()) +
                 ": Recv hash table <key,value> pairs from rank" + std::to_string(r) + " on GPU" +
                 std::to_string(id) + ", and write to file ");
        int tag = (id << 8) | base_tag;
        MPI_Status status;
        CK_MPI_THROW_(MPI_Probe(r, tag, MPI_COMM_WORLD, &status));
        int size_in_B;
        CK_MPI_THROW_(MPI_Get_count(&status, MPI_CHAR, &size_in_B));
        CK_MPI_THROW_(MPI_Recv(file_buf.get(), size_in_B, MPI_CHAR, r, tag, MPI_COMM_WORLD,
                               MPI_STATUS_IGNORE));
        weight_stream.write(file_buf.get(), size_in_B);
      }
    }
  }
#endif

  MESSAGE_("Done");

  for (size_t id = 0; id < local_gpu_count; id++) {
    if (count[id] == 0) {
      continue;
    }

    context.set_device(Base::get_local_gpu(id).get_device_id());

    CK_CUDA_THROW_(hipHostFree(h_hash_table_key[id]));
    CK_CUDA_THROW_(hipFree(d_hash_table_key[id]));
    CK_CUDA_THROW_(hipHostFree(h_hash_table_slot_id[id]));
    CK_CUDA_THROW_(hipFree(d_hash_table_slot_id[id]));
    CK_CUDA_THROW_(hipHostFree(h_hash_table_value[id]));
  }

  return;
}

template <typename TypeHashKey, typename TypeEmbeddingComp>
void LocalizedSlotSparseEmbeddingOneHot<TypeHashKey, TypeEmbeddingComp>::dump_parameters(
    Tensor2<TypeHashKey> &keys, Tensor2<size_t> &slot_id, Tensor2<float> &embeddings, size_t *num,
    size_t embedding_vec_size, const Tensors2<float> &hash_table_value_tensors,
    const std::vector<size_t> &slot_sizes) const {
  TypeHashKey *key_ptr = keys.get_ptr();
  size_t *slot_id_ptr = slot_id.get_ptr();
  float *embedding_ptr = embeddings.get_ptr();

  size_t local_gpu_count = Base::get_resource_manager().get_local_gpu_count();

  // memory allocation
  std::unique_ptr<size_t[]> count(new size_t[local_gpu_count]);
  size_t max_count = 0;
  size_t total_count = 0;

  CudaDeviceContext context;
  for (size_t id = 0; id < local_gpu_count; id++) {
    context.set_device(Base::get_local_gpu(id).get_device_id());
    count[id] = 0;
    for (size_t i = 0; i < slot_sizes.size(); i++) {
      size_t global_id = Base::get_local_gpu(id).get_global_id();
      if ((i % Base::get_resource_manager().get_global_gpu_count()) == global_id) {
        count[id] += slot_sizes[i];
      }
    }
    max_count = max(max_count, count[id]);
    total_count += count[id];
  }

#ifdef ENABLE_MPI
  CK_MPI_THROW_(
      MPI_Allreduce(MPI_IN_PLACE, &max_count, sizeof(size_t), MPI_CHAR, MPI_MAX, MPI_COMM_WORLD));
#endif

  /*if (total_count > (size_t)vocabulary_size) {
    CK_THROW_(Error_t::WrongInput,
              "Error: required download size is larger than hash table vocabulary_size");
  }*/

  std::unique_ptr<TypeHashKey *[]> h_hash_table_key(new TypeHashKey *[local_gpu_count]);
  std::unique_ptr<TypeHashKey *[]> d_hash_table_key(new TypeHashKey *[local_gpu_count]);
  std::unique_ptr<size_t *[]> h_hash_table_slot_id(new size_t *[local_gpu_count]);
  std::unique_ptr<size_t *[]> d_hash_table_slot_id(new size_t *[local_gpu_count]);
  std::unique_ptr<float *[]> h_hash_table_value(new float *[local_gpu_count]);
  for (size_t id = 0; id < local_gpu_count; id++) {
    if (count[id] == 0) {
      continue;
    }

    context.set_device(Base::get_local_gpu(id).get_device_id());

    hipHostMalloc(&h_hash_table_key[id], count[id] * sizeof(TypeHashKey));
    hipMalloc(&d_hash_table_key[id], count[id] * sizeof(TypeHashKey));
    hipHostMalloc(&h_hash_table_slot_id[id], count[id] * sizeof(size_t));
    hipMalloc(&d_hash_table_slot_id[id], count[id] * sizeof(size_t));
    hipHostMalloc(&h_hash_table_value[id], count[id] * embedding_vec_size * sizeof(float));
  }

  // Generate key and slot_id tensor, dump value tensor on GPU
  for (size_t id = 0; id < local_gpu_count; id++) {
    if (count[id] == 0) {
      continue;
    }

    MESSAGE_("Rank" + std::to_string(Base::get_resource_manager().get_process_id()) +
             ": Dump embedding table from GPU" + std::to_string(id));

    context.set_device(Base::get_local_gpu(id).get_device_id());

    // Loop for each slot
    size_t buffer_offset = 0;
    for (size_t i = 0; i < slot_sizes.size(); i++) {
      size_t global_id = Base::get_local_gpu(id).get_global_id();
      if ((i % Base::get_resource_manager().get_global_gpu_count()) == global_id) {
        // Generate key buffer
        size_t key_offset = 0;
        for (size_t j = 0; j < i; j++) {
          key_offset += slot_sizes[j];
        }
        functors_.memset_liner(d_hash_table_key[id] + buffer_offset, (TypeHashKey)key_offset,
                               (TypeHashKey)1, slot_sizes[i], Base::get_local_gpu(id).get_stream());

        // Generate slot_id
        functors_.memset_const(d_hash_table_slot_id[id] + buffer_offset, i, slot_sizes[i],
                               Base::get_local_gpu(id).get_stream());

        buffer_offset += slot_sizes[i];
      }
    }
    // Copy key buffer to host
    CK_CUDA_THROW_(hipMemcpyAsync(h_hash_table_key[id], d_hash_table_key[id],
                                   count[id] * sizeof(TypeHashKey), hipMemcpyDeviceToHost,
                                   Base::get_local_gpu(id).get_stream()));
    // Copy value buffer to host
    CK_CUDA_THROW_(hipMemcpyAsync(h_hash_table_value[id], hash_table_value_tensors[id].get_ptr(),
                                   count[id] * embedding_vec_size * sizeof(float),
                                   hipMemcpyDeviceToHost, Base::get_local_gpu(id).get_stream()));
    // Copy slot_id to host
    CK_CUDA_THROW_(hipMemcpyAsync(h_hash_table_slot_id[id], d_hash_table_slot_id[id],
                                   count[id] * sizeof(size_t), hipMemcpyDeviceToHost,
                                   Base::get_local_gpu(id).get_stream()));
  }

  // sync wait
  functors_.sync_all_gpus(Base::get_resource_manager());

  // TODO: could be optimized ???
  // one pair in the file includes <key,slot_id,value>
  size_t key_size = sizeof(TypeHashKey);
  size_t slot_id_size = sizeof(size_t);
  size_t value_size = sizeof(float) * embedding_vec_size;

  size_t offset = 0;
  for (size_t id = 0; id < local_gpu_count; id++) {
    for (unsigned int k = 0; k < count[id]; k++) {
      memcpy(key_ptr + offset, h_hash_table_key[id] + k, key_size);
      memcpy(slot_id_ptr + offset, h_hash_table_slot_id[id] + k, slot_id_size);
      memcpy(embedding_ptr + offset * embedding_vec_size,
             h_hash_table_value[id] + k * embedding_vec_size, value_size);
      offset += 1;
    }

    // std::cout << std::endl;
    MESSAGE_("Write hash table <key,slot_id,value> pairs to file");
  }

  *num = offset;

  // MESSAGE_("Done");

  for (size_t id = 0; id < local_gpu_count; id++) {
    if (count[id] == 0) {
      continue;
    }

    context.set_device(Base::get_local_gpu(id).get_device_id());

    CK_CUDA_THROW_(hipHostFree(h_hash_table_key[id]));
    CK_CUDA_THROW_(hipFree(d_hash_table_key[id]));
    CK_CUDA_THROW_(hipHostFree(h_hash_table_slot_id[id]));
    CK_CUDA_THROW_(hipFree(d_hash_table_slot_id[id]));
    CK_CUDA_THROW_(hipHostFree(h_hash_table_value[id]));
  }

  return;
}

template <typename TypeHashKey, typename TypeEmbeddingComp>
void LocalizedSlotSparseEmbeddingOneHot<TypeHashKey, TypeEmbeddingComp>::init_embedding(
    const std::vector<size_t> slot_sizes, size_t embedding_vec_size,
    std::vector<Tensors2<float>> &hash_table_value_tensors,
    Tensors2<size_t> &hash_table_slot_id_tensors) {
  size_t local_gpu_count = Base::get_resource_manager().get_local_gpu_count();
  size_t total_gpu_count = Base::get_resource_manager().get_global_gpu_count();

#ifndef NDEBUG
  MESSAGE_("local_gpu_count=" + std::to_string(local_gpu_count) +
           ", total_gpu_count=" + std::to_string(total_gpu_count));
#endif

#pragma omp parallel num_threads(Base::get_resource_manager().get_local_gpu_count())
  {
    size_t id = omp_get_thread_num();
    size_t device_id = Base::get_local_gpu(id).get_device_id();
    size_t global_id = Base::get_local_gpu(id).get_global_id();

#ifndef NDEBUG
    MESSAGE_("id=" + std::to_string(id) + ", device_id=" + std::to_string(device_id) +
             ", global_id=" + std::to_string(global_id));
#endif

    functors_.init_embedding_per_gpu(global_id, total_gpu_count, slot_sizes, embedding_vec_size,
                                     hash_table_value_tensors[id], hash_table_slot_id_tensors[id],
                                     Base::get_local_gpu(id));

    CK_CUDA_THROW_(hipStreamSynchronize(Base::get_local_gpu(id).get_stream()));
    MESSAGE_("gpu" + std::to_string(id) + " init embedding done");
  }

  return;
}

template <typename TypeHashKey, typename TypeEmbeddingComp>
void LocalizedSlotSparseEmbeddingOneHot<TypeHashKey, TypeEmbeddingComp>::reset() {
  CudaDeviceContext context;
  for (size_t i = 0; i < Base::get_resource_manager().get_local_gpu_count(); i++) {
    functors_.init_embedding_per_gpu(
        Base::get_local_gpu(i).get_global_id(), Base::get_resource_manager().get_global_gpu_count(),
        slot_size_array_, Base::get_embedding_vec_size(), value_table_tensors_[i],
        hash_table_slot_id_tensors_[i], Base::get_local_gpu(i));
  }

  for (size_t i = 0; i < Base::get_resource_manager().get_local_gpu_count(); i++) {
    CK_CUDA_THROW_(hipStreamSynchronize(Base::get_local_gpu(i).get_stream()));
  }
}

template class LocalizedSlotSparseEmbeddingOneHot<unsigned int, float>;
template class LocalizedSlotSparseEmbeddingOneHot<long long, float>;
template class LocalizedSlotSparseEmbeddingOneHot<unsigned int, __half>;
template class LocalizedSlotSparseEmbeddingOneHot<long long, __half>;

}  // namespace HugeCTR
