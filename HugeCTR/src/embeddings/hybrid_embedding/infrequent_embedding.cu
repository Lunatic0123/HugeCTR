#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hip/hip_runtime.h>

#include <algorithm>
#include <hipcub/hipcub.hpp>
#include <iostream>
#include <utility>
#include <vector>

#include "HugeCTR/include/common.hpp"
#include "HugeCTR/include/data_simulator.hpp"
#include "HugeCTR/include/embeddings/hybrid_embedding/data.hpp"
#include "HugeCTR/include/embeddings/hybrid_embedding/infrequent_embedding.hpp"
#include "HugeCTR/include/embeddings/hybrid_embedding/model.hpp"
#include "HugeCTR/include/embeddings/hybrid_embedding/update.cuh"
#include "HugeCTR/include/embeddings/hybrid_embedding/utils.cuh"
#include "HugeCTR/include/embeddings/hybrid_embedding/utils.hpp"
#include "HugeCTR/include/shuffle/shuffle.cuh"
#include "HugeCTR/include/tensor2.hpp"
#include "HugeCTR/include/utils.hpp"

namespace HugeCTR {

namespace hybrid_embedding {

namespace infrequent_embedding_kernels {

template <typename dtype, typename emtype>
__global__ void hier_update_model(
    const uint32_t* __restrict__ model_indices, const uint32_t* __restrict__ model_indices_offsets,
    const dtype* __restrict__ samples, const dtype* __restrict__ category_location,
    const emtype* __restrict__ gradients, float* __restrict__ embedding_vectors,
    uint32_t embedding_vec_size, uint32_t num_instances, uint32_t local_samples_size,
    uint32_t local_comm_buff_size, const float* __restrict__ lr_ptr, const float scale) {
  float lr = __ldg(lr_ptr) / scale;
  const uint32_t num_indices = model_indices_offsets[num_instances];

  // Load offset only when the network_id changes
  uint32_t previous_network_id = 0;
  uint32_t offset = 0;

  for (uint32_t i = blockIdx.x; i < num_indices; i += gridDim.x) {
    uint32_t index = model_indices[i];
    dtype category = samples[index];
    dtype location = category_location[2 * category + 1];
    uint32_t network_id = index / local_samples_size;
    if (network_id != previous_network_id) {
      offset = model_indices_offsets[network_id];
      previous_network_id = network_id;
    }
    atomicAdd(
        embedding_vectors + location * embedding_vec_size + threadIdx.x,
        -lr * TypeConvertFunc<float, emtype>::convert(
                  gradients[embedding_vec_size * (network_id * local_comm_buff_size + i - offset) +
                            threadIdx.x]));
  }
}

template <typename dtype, typename emtype>
__global__ void infrequent_update_model_direct(
    const emtype* const* __restrict__ gradients_pointers, float* embedding_vectors,
    const uint32_t* __restrict__ model_indices, const uint32_t* __restrict__ model_indices_offsets,
    const dtype* __restrict__ samples, const dtype* __restrict__ category_location,
    uint32_t num_instances, uint32_t model_id, uint32_t embedding_vec_size,
    uint32_t local_samples_size, const float* __restrict__ lr_ptr, const float scale) {
  float lr = __ldg(lr_ptr) / scale;
  // Shift pattern
  const uint32_t offset = __ldg(model_indices_offsets + model_id + 1);
  const uint32_t num_model_indices = __ldg(model_indices_offsets + num_instances);

  for (uint32_t i = blockIdx.x; i < num_model_indices; i += gridDim.x) {
    uint32_t vid = (i + offset) % num_model_indices;

    uint32_t index = model_indices[vid];
    uint32_t network_id = index / local_samples_size;
    uint32_t local_index = index % local_samples_size;
    dtype category = samples[index];
    uint32_t location = category_location[2 * category + 1];

    const emtype* gradients = gradients_pointers[network_id];

    atomicAdd(embedding_vectors + location * embedding_vec_size + threadIdx.x,
              -lr * TypeConvertFunc<float, emtype>::convert(
                        gradients[local_index * embedding_vec_size + threadIdx.x]));
  }
}

template <typename dtype>
__global__ void calculate_network_indices_mask(const dtype* __restrict__ local_samples,
                                               const dtype* __restrict__ category_location,
                                               bool* mask, uint32_t local_samples_size,
                                               uint32_t num_instances) {
  for (uint32_t i = blockIdx.x * blockDim.x + threadIdx.x; i < local_samples_size;
       i += gridDim.x * blockDim.x) {
    dtype category = local_samples[i];
    uint32_t model_id = static_cast<uint32_t>(category_location[2 * category]);
    for (uint32_t section_id = 0; section_id < num_instances; section_id++) {
      mask[local_samples_size * section_id + i] = (model_id == section_id);
    }
  }
}

}  // namespace infrequent_embedding_kernels

template <typename dtype, typename emtype>
InfrequentEmbedding<dtype, emtype>::InfrequentEmbedding(const Data<dtype>& data_train,
                                                        const Data<dtype>& data_evaluate,
                                                        const Model<dtype>& model,
                                                        const GPUResource& gpu_resource,
                                                        uint32_t embedding_vec_size)
    : model_(model),
      data_train_(data_train),
      data_evaluate_(data_evaluate),
      data_(data_train),  // Temporary
      gpu_resource(gpu_resource),
      embedding_vec_size_(embedding_vec_size) {
  auto buf = GeneralBuffer2<CudaAllocator>::create();
  auto managed_buf = GeneralBuffer2<CudaManagedAllocator>::create();

  size_t universe_batch_size = std::max(data_train.batch_size, data_evaluate.batch_size);
  buf->reserve({ceildiv<size_t>(model.num_categories, model.num_instances), embedding_vec_size_},
               &infrequent_embedding_vectors_);
  buf->reserve({universe_batch_size, data_train.table_sizes.size()}, &model_indices_);
  managed_buf->reserve({model.num_instances + 1, 1}, &model_indices_offsets_);
  buf->reserve({model_.num_instances}, &model_indices_sizes_);
  buf->reserve({model_.num_instances},
               &model_indices_sizes_ptrs_);  // TODO: should be local instances
  buf->reserve(
      {ceildiv<size_t>(universe_batch_size, model.num_instances), data_train.table_sizes.size()},
      &network_indices_);
  managed_buf->reserve({model.num_instances + 1, 1}, &network_indices_offsets_);
  buf->reserve({model_.num_instances}, &network_indices_sizes_);
  buf->reserve({model_.num_instances}, &network_indices_sizes_ptrs_);

  // Temporary storage
  calculate_model_indices_temp_storage_bytes();
  calculate_network_indices_temp_storage_bytes();
  buf->reserve({model_indices_temp_storage_bytes, 1}, &model_indices_temp_storage_);
  buf->reserve({network_indices_temp_storage_bytes, 1}, &network_indices_temp_storage_);

  buf->reserve({model.num_instances, 1}, &interaction_layer_input_pointers_train_);
  buf->reserve({model.num_instances, 1}, &interaction_layer_input_pointers_eval_);
  buf->reserve({model.num_instances, 1}, &gradients_pointers_);
  buf->allocate();

  managed_buf->allocate();
  int current_device;
  CK_CUDA_THROW_(hipGetDevice(&current_device));
  CK_CUDA_THROW_(hipMemAdvise(managed_buf->get_ptr(), managed_buf->get_size_in_bytes(),
                               hipMemAdviseSetReadMostly, current_device));
}

template <typename dtype, typename emtype>
void InfrequentEmbedding<dtype, emtype>::initialize_embedding_vectors() {
  CudaDeviceContext context(gpu_resource.get_device_id());

  const size_t num_tables = data_.table_sizes.size();
  for (size_t i = 0; i < num_tables; i++) {
    float up_bound = sqrt(1.f / data_.table_sizes[i]);

    const size_t offset = embedding_vec_size_ * model_.h_infrequent_model_table_offsets[i];
    const size_t number_of_vectors =
        model_.h_infrequent_model_table_offsets[i + 1] - model_.h_infrequent_model_table_offsets[i];
    UniformGenerator::fill(
        infrequent_embedding_vectors_.get_ptr() + offset, embedding_vec_size_ * number_of_vectors,
        -up_bound, up_bound, gpu_resource.get_sm_count(),
        gpu_resource.get_replica_variant_curand_generator(), gpu_resource.get_stream());
  }
}

template <typename dtype, typename emtype>
void InfrequentEmbedding<dtype, emtype>::forward_model(emtype* message_buffer,
                                                       hipStream_t stream) {
  auto model_indices = model_indices_.get_ptr();
  auto samples = data_.samples.get_ptr();
  auto category_location = model_.category_location.get_ptr();
  auto infrequent_embedding_vectors = infrequent_embedding_vectors_.get_ptr();
  auto embedding_vec_size = embedding_vec_size_;

  auto copy_desc = CopyDescriptors::make_OneToOne<float, emtype, 1>(
      model_indices_offsets_.get_ptr() + model_.num_instances, embedding_vec_size,
      [=] __device__(size_t i) -> CopyDescriptors::CopyDetails<float, emtype, 1> {
        uint32_t index = model_indices[i];
        dtype category = samples[index];
        dtype location = category_location[2 * category + 1];

        return {infrequent_embedding_vectors + location * embedding_vec_size,
                {message_buffer + i * embedding_vec_size},
                {true}};
      });

  shuffle(copy_desc, stream, data_.samples.get_num_elements() / model_.num_instances / 8);
  CK_CUDA_THROW_(hipPeekAtLastError());
}

template <typename dtype, typename emtype>
void InfrequentEmbedding<dtype, emtype>::fused_intra_forward_model(emtype** message_buffer,
                                                                   hipStream_t stream) {
  auto model_indices = model_indices_.get_ptr();
  auto model_indices_offsets = model_indices_offsets_.get_ptr();
  auto samples = data_.samples.get_ptr();
  auto category_location = model_.category_location.get_ptr();
  auto infrequent_embedding_vectors = infrequent_embedding_vectors_.get_ptr();
  size_t embedding_vec_size = embedding_vec_size_;
  auto local_instance_id = model_.instance_id;
  auto num_instances = model_.num_instances;
  auto per_node_instances = num_instances / model_.h_num_instances_per_node.size();
  uint32_t local_samples_size =
      ceildiv<uint32_t>(data_.batch_size, num_instances) * data_.table_sizes.size();

  uint32_t local_comm_buff_size =
      ceildiv<uint32_t>(max_num_infrequent_per_batch_, model_.num_instances);

  auto copy_desc = CopyDescriptors::make_OneToOne<float, emtype, 1>(
      model_indices_offsets_.get_ptr() + num_instances, embedding_vec_size,
      [=] __device__(size_t i) -> CopyDescriptors::CopyDetails<float, emtype, 1> {
        uint32_t num_selected = model_indices_offsets[num_instances];
        uint32_t vid = (i + model_indices_offsets[(local_instance_id + 1) % per_node_instances]) %
                       num_selected;
        uint32_t index = model_indices[vid];
        uint32_t network_id = (index / local_samples_size);
        dtype category = samples[index];
        dtype location = category_location[2 * category + 1];
        uint32_t local_network_id = (network_id % per_node_instances);
        emtype* output_ptr =
            &message_buffer[local_network_id][(network_id - local_network_id + local_instance_id) *
                                              local_comm_buff_size * embedding_vec_size];

        return {infrequent_embedding_vectors + location * embedding_vec_size,
                {output_ptr + (vid - model_indices_offsets[network_id]) * embedding_vec_size},
                {true}};
      });

  shuffle(copy_desc, stream, data_.samples.get_num_elements() / model_.num_instances / 8);
  CK_CUDA_THROW_(hipPeekAtLastError());
}

template <typename dtype, typename emtype>
void InfrequentEmbedding<dtype, emtype>::forward_network(const emtype* message_buffer,
                                                         emtype* interaction_layer_input,
                                                         hipStream_t stream) {
  auto network_indices = network_indices_.get_ptr();
  auto embedding_vec_size = embedding_vec_size_;

  auto copy_desc = CopyDescriptors::make_OneToOne<emtype, emtype, 1>(
      network_indices_offsets_.get_ptr() + model_.num_instances, embedding_vec_size,
      [=] __device__(size_t i) -> CopyDescriptors::CopyDetails<emtype, emtype, 1> {
        uint32_t index = network_indices[i];
        return {message_buffer + i * embedding_vec_size,
                {interaction_layer_input + index * embedding_vec_size},
                {true}};
      });

  shuffle(copy_desc, stream, data_.samples.get_num_elements() / model_.num_instances / 8);
  CK_CUDA_THROW_(hipPeekAtLastError());
}

template <typename dtype, typename emtype>
void InfrequentEmbedding<dtype, emtype>::hier_forward_network(const emtype* message_buffer,
                                                              emtype* interaction_layer_input,
                                                              hipStream_t stream) {
  auto network_indices = network_indices_.get_ptr();
  auto network_indices_offsets = network_indices_offsets_.get_ptr();
  auto embedding_vec_size = embedding_vec_size_;
  uint32_t local_samples_size =
      ceildiv<uint32_t>(data_.batch_size, model_.num_instances) * data_.table_sizes.size();
  uint32_t local_comm_buff_size =
      ceildiv<uint32_t>(max_num_infrequent_per_batch_, model_.num_instances);

  auto copy_desc = CopyDescriptors::make_OneToOne<emtype, emtype, 1>(
      network_indices_offsets_.get_ptr() + model_.num_instances, embedding_vec_size,
      [=] __device__(size_t i) -> CopyDescriptors::CopyDetails<emtype, emtype, 1> {
        uint32_t index = network_indices[i];

        // Find model id and offset
        uint32_t model_id = 0;
        uint32_t offset = 0;
        uint32_t next_offset = network_indices_offsets[1];
        while (next_offset <= i) {
          offset = next_offset;
          model_id++;
          next_offset = network_indices_offsets[model_id + 1];
        }

        return {
            message_buffer + (model_id * local_comm_buff_size + i - offset) * embedding_vec_size,
            {interaction_layer_input + index * embedding_vec_size},
            {true}};
      });

  shuffle(copy_desc, stream, data_.samples.get_num_elements() / model_.num_instances / 8);
  CK_CUDA_THROW_(hipPeekAtLastError());
}

/** Forward network for single GPU (no communications) */
template <typename dtype, typename emtype>
void InfrequentEmbedding<dtype, emtype>::forward_network_direct(bool is_train,
                                                                hipStream_t stream) {
  const uint32_t num_instances = model_.num_instances;
  const uint32_t model_id = model_.global_instance_id;
  uint32_t local_samples_size =
      ceildiv<uint32_t>(data_.batch_size, num_instances) * data_.table_sizes.size();

  auto interaction_layer_input_pointers = is_train
                                              ? interaction_layer_input_pointers_train_.get_ptr()
                                              : interaction_layer_input_pointers_eval_.get_ptr();
  auto model_indices = model_indices_.get_ptr();
  auto model_indices_offsets = model_indices_offsets_.get_ptr();
  auto category_location = model_.category_location.get_ptr();
  auto samples = data_.samples.get_ptr();
  auto model_table = infrequent_embedding_vectors_.get_ptr();
  auto embedding_vec_size = embedding_vec_size_;

  auto copy_desc = CopyDescriptors::make_OneToOne<float, emtype, 1>(
      model_indices_offsets + num_instances, embedding_vec_size,
      [=] __device__(size_t i) -> CopyDescriptors::CopyDetails<float, emtype, 1> {
        const uint32_t offset = model_indices_offsets[model_id + 1];
        const uint32_t num_model_indices = model_indices_offsets[num_instances];
        const uint32_t vid = (i + offset) % num_model_indices;
        const uint32_t index = model_indices[vid];

        const dtype category = samples[index];
        const dtype location = category_location[2 * category + 1];

        const uint32_t network_id = index / local_samples_size;
        const uint32_t local_index = index % local_samples_size;

        emtype* interaction_layer_input = interaction_layer_input_pointers[network_id];

        return {model_table + location * embedding_vec_size,
                {interaction_layer_input + local_index * embedding_vec_size},
                {true}};
      });

  PROFILE_RECORD("inf_forward_network_direct.forward_network_direct.start", stream, false);
  shuffle(copy_desc, stream, local_samples_size / 10);
  CK_CUDA_THROW_(hipPeekAtLastError());
  PROFILE_RECORD("inf_forward_network_direct.forward_network_direct.stop", stream, false);
}

template <typename dtype, typename emtype>
void InfrequentEmbedding<dtype, emtype>::update_network(const emtype* gradients,
                                                        emtype* message_buffer,
                                                        hipStream_t stream) {
  auto network_indices = network_indices_.get_ptr();
  auto embedding_vec_size = embedding_vec_size_;

  auto copy_desc = CopyDescriptors::make_OneToOne<emtype, emtype, 1>(
      network_indices_offsets_.get_ptr() + model_.num_instances, embedding_vec_size,
      [=] __device__(size_t i) -> CopyDescriptors::CopyDetails<emtype, emtype, 1> {
        uint32_t index = network_indices[i];

        return {gradients + index * embedding_vec_size,
                {message_buffer + i * embedding_vec_size},
                {true}};
      });

  shuffle(copy_desc, stream, data_.samples.get_num_elements() / model_.num_instances / 8);
  CK_CUDA_THROW_(hipPeekAtLastError());
}

template <typename dtype, typename emtype>
void InfrequentEmbedding<dtype, emtype>::fused_intra_update_network(const emtype* gradients,
                                                                    emtype** message_buffer,
                                                                    hipStream_t stream) {
  auto network_indices = network_indices_.get_ptr();
  auto network_indices_offsets = network_indices_offsets_.get_ptr();
  size_t embedding_vec_size = embedding_vec_size_;
  auto local_instance_id = model_.instance_id;
  auto num_instances = model_.num_instances;
  auto per_node_instances = num_instances / model_.h_num_instances_per_node.size();
  uint32_t local_comm_buff_size =
      ceildiv<uint32_t>(max_num_infrequent_per_train_batch_, model_.num_instances);

  auto copy_desc = CopyDescriptors::make_OneToOne<emtype, emtype, 1>(
      network_indices_offsets_.get_ptr() + model_.num_instances, embedding_vec_size,
      [=] __device__(size_t i) -> CopyDescriptors::CopyDetails<emtype, emtype, 1> {
        uint32_t num_selected = network_indices_offsets[num_instances];
        uint32_t vid = (i + network_indices_offsets[(local_instance_id + 1) % per_node_instances]) %
                       num_selected;
        uint32_t index = network_indices[vid];

        uint32_t model_id;
        for (model_id = 0; model_id < num_instances && network_indices_offsets[model_id + 1] <= vid;
             model_id++)
          ;

        uint32_t local_model_id = (model_id % per_node_instances);
        emtype* output_ptr =
            &message_buffer[local_model_id][(model_id - local_model_id + local_instance_id) *
                                            local_comm_buff_size * embedding_vec_size];

        return {gradients + index * embedding_vec_size,
                {output_ptr + (vid - network_indices_offsets[model_id]) * embedding_vec_size},
                {true}};
      });

  shuffle(copy_desc, stream, data_.samples.get_num_elements() / model_.num_instances / 8);
  CK_CUDA_THROW_(hipPeekAtLastError());
}

template <typename dtype, typename emtype>
void InfrequentEmbedding<dtype, emtype>::update_model(const emtype* message_buffer,
                                                      float* dev_lr, float scale,
                                                      hipStream_t stream) {
  const uint32_t* __restrict__ model_indices = model_indices_.get_ptr();
  const dtype* __restrict__ samples = data_.samples.get_ptr();
  const dtype* __restrict__ category_location = model_.category_location.get_ptr();

  uint32_t n_blocks = gpu_resource.get_sm_count();

  sgd_atomic_update(message_buffer, infrequent_embedding_vectors_.get_ptr(),
                    model_indices_offsets_.get_ptr() + model_.num_instances,
                    [model_indices, samples, category_location] __device__(uint32_t i) {
                      uint32_t index = model_indices[i];
                      dtype category = samples[index];
                      return category_location[2 * category + 1];
                    },
                    n_blocks, embedding_vec_size_, dev_lr, scale, stream);
}

template <typename dtype, typename emtype>
void InfrequentEmbedding<dtype, emtype>::hier_update_model(const emtype* message_buffer,
                                                           float* dev_lr, float scale,
                                                           hipStream_t stream) {
  const uint32_t& num_instances = model_.num_instances;
  uint32_t local_samples_size =
      ceildiv<uint32_t>(data_.batch_size, num_instances) * data_.table_sizes.size();
  uint32_t local_comm_buff_size =
      ceildiv<uint32_t>(max_num_infrequent_per_train_batch_, model_.num_instances);
  const uint32_t* __restrict__ model_indices = model_indices_.get_ptr();
  const dtype* __restrict__ samples = data_.samples.get_ptr();
  const dtype* __restrict__ category_location = model_.category_location.get_ptr();

  int num_sm = gpu_resource.get_sm_count();
  int n_blocks = 16 * num_sm;  // TODO: better heuristics

  infrequent_embedding_kernels::hier_update_model<<<n_blocks, embedding_vec_size_, 0, stream>>>(
      model_indices_.get_ptr(), model_indices_offsets_.get_ptr(), data_.samples.get_ptr(),
      model_.category_location.get_ptr(), message_buffer, infrequent_embedding_vectors_.get_ptr(),
      embedding_vec_size_, num_instances, local_samples_size, local_comm_buff_size, dev_lr, scale);
  CK_CUDA_THROW_(hipPeekAtLastError());
}

/** Update model for single GPU (no communications), lr is a device variable */
template <typename dtype, typename emtype>
void InfrequentEmbedding<dtype, emtype>::update_model_direct(float* dev_lr, float scale,
                                                             hipStream_t stream) {
  const uint32_t& num_instances = model_.num_instances;
  uint32_t local_samples_size =
      ceildiv<uint32_t>(data_.batch_size, num_instances) * data_.table_sizes.size();

  int num_sm = gpu_resource.get_sm_count();
  int n_blocks = 16 * num_sm;  // TODO: better heuristics

  /* Each model reads from the gradients of each network */
  PROFILE_RECORD("inf_update_model_direct.infrequent_update_model_direct.start", stream, false);
  infrequent_embedding_kernels::
      infrequent_update_model_direct<<<n_blocks, embedding_vec_size_, 0, stream>>>(
          gradients_pointers_.get_ptr(), infrequent_embedding_vectors_.get_ptr(),
          model_indices_.get_ptr(), model_indices_offsets_.get_ptr(), data_.samples.get_ptr(),
          model_.category_location.get_ptr(), model_.num_instances, model_.global_instance_id,
          embedding_vec_size_, local_samples_size, dev_lr, scale);
  CK_CUDA_THROW_(hipPeekAtLastError());
  PROFILE_RECORD("inf_update_model_direct.infrequent_update_model_direct.stop", stream, false);
}

template <typename dtype>
struct ModelIndicesSelectOp {
  const dtype* samples;
  const dtype* category_location;
  uint32_t my_model_id;
  __host__ __device__ __forceinline__ ModelIndicesSelectOp(const dtype* samples,
                                                           const dtype* category_location,
                                                           uint32_t my_model_id)
      : samples(samples), category_location(category_location), my_model_id(my_model_id) {}
  __device__ __forceinline__ bool operator()(const uint32_t& idx) const {
    dtype category = __ldg(samples + idx);
    dtype model_id = __ldg(category_location + 2 * category);
    return model_id == my_model_id;
  }
};

template <typename dtype, typename emtype>
void InfrequentEmbedding<dtype, emtype>::calculate_model_indices_temp_storage_bytes() {
  size_t max_batch_size = std::max(data_train_.batch_size, data_evaluate_.batch_size);

  hipcub::CountingInputIterator<uint32_t> counting(0);
  ModelIndicesSelectOp<dtype> select_op(nullptr, nullptr, 0);
  hipcub::DeviceSelect::If(nullptr, model_indices_temp_storage_bytes, counting, (uint32_t*)nullptr,
                        (uint32_t*)nullptr, max_batch_size * data_.table_sizes.size(), select_op,
                        0);
}

template <typename dtype, typename emtype>
void InfrequentEmbedding<dtype, emtype>::calculate_model_indices(hipStream_t stream) {
  const uint32_t& num_instances = model_.num_instances;

  size_t local_batch_size = ceildiv<size_t>(data_.batch_size, num_instances);

  // Select indices of infrequent categories belonging to this model
  hipcub::CountingInputIterator<uint32_t> counting(0);
  ModelIndicesSelectOp<dtype> select_op(data_.samples.get_ptr(), model_.category_location.get_ptr(),
                                        model_.global_instance_id);
  PROFILE_RECORD("inf_calculate_model_indices.device_select_if.start", stream, false);
  hipcub::DeviceSelect::If(reinterpret_cast<void*>(model_indices_temp_storage_.get_ptr()),
                        model_indices_temp_storage_bytes, counting, model_indices_.get_ptr(),
                        model_indices_offsets_.get_ptr() + num_instances,
                        data_.batch_size * data_.table_sizes.size(), select_op, stream);
  PROFILE_RECORD("inf_calculate_model_indices.device_select_if.stop", stream, false);

  // Compute offsets
  constexpr size_t TPB = 256;
  const size_t n_blocks = ceildiv<size_t>(num_instances, TPB);
  PROFILE_RECORD("inf_calculate_model_indices.offsets_kernel.start", stream, false);
  offsets_kernel<<<n_blocks, TPB, 0, stream>>>(model_indices_.get_ptr(),
                                               model_indices_offsets_.get_ptr(), num_instances,
                                               local_batch_size * data_.table_sizes.size());
  PROFILE_RECORD("inf_calculate_model_indices.offsets_kernel.stop", stream, false);
  CK_CUDA_THROW_(hipPeekAtLastError());
}

static __global__ void offsets_to_sizes(size_t* sizes, uint32_t* offsets, size_t element_size,
                                        uint32_t num_instances) {
  for (int t = blockIdx.x * blockDim.x + threadIdx.x; t < num_instances;
       t += gridDim.x * blockDim.x) {
    sizes[t] = (offsets[t + 1] - offsets[t]) * element_size;
  }
}

template <typename dtype, typename emtype>
void InfrequentEmbedding<dtype, emtype>::calculate_model_indices_sizes_from_offsets(
    hipStream_t stream) {
  constexpr size_t TPB = 256;
  const size_t n_blocks = ceildiv<size_t>(model_.num_instances, TPB);
  offsets_to_sizes<<<n_blocks, TPB, 0, stream>>>(
      model_indices_sizes_.get_ptr(), model_indices_offsets_.get_ptr(),
      embedding_vec_size_ * sizeof(emtype), model_.num_instances);
}

template <typename dtype, typename emtype>
void InfrequentEmbedding<dtype, emtype>::calculate_network_indices_temp_storage_bytes() {
  size_t max_batch_size = std::max(data_train_.batch_size, data_evaluate_.batch_size);
  const uint32_t num_instances = model_.num_instances;
  uint32_t samples_size = max_batch_size * data_.table_sizes.size();
  uint32_t local_samples_size = ceildiv<uint32_t>(samples_size, num_instances);

  // Calculate select bytes
  size_t select_bytes = 0;
  hipcub::CountingInputIterator<uint32_t> counting(0);
  hipcub::DeviceSelect::Flagged(nullptr, select_bytes, counting, (bool*)nullptr, (uint32_t*)nullptr,
                             (uint32_t*)nullptr, samples_size, 0);

  // Total size
  constexpr uint32_t align = 256;
  network_indices_temp_storage_bytes =
      alignTo<size_t>(sizeof(bool) * samples_size, align) + select_bytes;
}

template <typename dtype, typename emtype>
void InfrequentEmbedding<dtype, emtype>::calculate_network_indices(hipStream_t stream) {
  const uint32_t num_instances = model_.num_instances;
  uint32_t samples_size = data_.batch_size * data_.table_sizes.size();
  uint32_t local_samples_size = ceildiv<uint32_t>(samples_size, num_instances);

  // Temporary storage
  constexpr uint32_t align = 256;
  char* scratch_ptr = network_indices_temp_storage_.get_ptr();
  size_t scratch_offset = 0;
  bool* d_mask = reinterpret_cast<bool*>(scratch_ptr + scratch_offset);
  scratch_offset += alignTo<size_t>(sizeof(bool) * samples_size, align);
  void* d_temp_storage = reinterpret_cast<void*>(scratch_ptr + scratch_offset);
  size_t temp_storage_bytes = network_indices_temp_storage_bytes - scratch_offset;

  // Compute mask (for each source GPU, whether each element in the batch is located there)
  constexpr uint32_t TPB_mask = 256;
  uint32_t n_blocks_mask = ceildiv<uint32_t>(local_samples_size, TPB_mask);
  PROFILE_RECORD("inf_calculate_network_indices.calculate_network_indices_mask.start", stream,
                 false);
  infrequent_embedding_kernels::
      calculate_network_indices_mask<<<n_blocks_mask, TPB_mask, 0, stream>>>(
          data_.samples.get_ptr() + model_.global_instance_id * local_samples_size,
          model_.category_location.get_ptr(), d_mask, local_samples_size, num_instances);
  CK_CUDA_THROW_(hipPeekAtLastError());
  PROFILE_RECORD("inf_calculate_network_indices.calculate_network_indices_mask.stop", stream,
                 false);

  // Select indices according to the mask
  hipcub::CountingInputIterator<uint32_t> counting(0);
  PROFILE_RECORD("inf_calculate_network_indices.device_select_flagged.start", stream, false);
  hipcub::DeviceSelect::Flagged(
      d_temp_storage, temp_storage_bytes, counting, d_mask, network_indices_.get_ptr(),
      network_indices_offsets_.get_ptr() + num_instances, samples_size, stream);
  PROFILE_RECORD("inf_calculate_network_indices.device_select_flagged.stop", stream, false);

  // Compute offsets
  constexpr uint32_t TPB_offsets = 256;
  uint32_t n_blocks_offsets = ceildiv<uint32_t>(num_instances, TPB_offsets);
  PROFILE_RECORD("inf_calculate_network_indices.offsets_kernel.start", stream, false);
  offsets_kernel<<<n_blocks_offsets, TPB_offsets, 0, stream>>>(network_indices_.get_ptr(),
                                                               network_indices_offsets_.get_ptr(),
                                                               num_instances, local_samples_size);
  CK_CUDA_THROW_(hipPeekAtLastError());
  PROFILE_RECORD("inf_calculate_network_indices.offsets_kernel.stop", stream, false);

  // Re-map indices between 0 and local_samples_size - 1
  uint32_t TPB_remap = 256;
  uint32_t n_blocks_remap = gpu_resource.get_sm_count();
  PROFILE_RECORD("inf_calculate_network_indices.modulo_kernel.start", stream, false);
  modulo_kernel<<<n_blocks_remap, TPB_remap, 0, stream>>>(
      network_indices_.get_ptr(), network_indices_offsets_.get_ptr() + num_instances,
      local_samples_size);
  CK_CUDA_THROW_(hipPeekAtLastError());
  PROFILE_RECORD("inf_calculate_network_indices.modulo_kernel.stop", stream, false);
}

template <typename dtype, typename emtype>
void InfrequentEmbedding<dtype, emtype>::calculate_network_indices_sizes_from_offsets(
    hipStream_t stream) {
  constexpr size_t TPB = 256;
  const size_t n_blocks = ceildiv<size_t>(model_.num_instances, TPB);
  offsets_to_sizes<<<n_blocks, TPB, 0, stream>>>(
      network_indices_sizes_.get_ptr(), network_indices_offsets_.get_ptr(),
      embedding_vec_size_ * sizeof(emtype), model_.num_instances);
}

template class InfrequentEmbedding<uint32_t, __half>;
template class InfrequentEmbedding<uint32_t, float>;
template class InfrequentEmbedding<long long, __half>;
template class InfrequentEmbedding<long long, float>;
}  // namespace hybrid_embedding

}  // namespace HugeCTR
