#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <hipcub/hipcub.hpp>

#include "HugeCTR/include/utils.hpp"
#include "all2all_embedding_collection.hpp"
namespace embedding {
namespace tf {

namespace {

template <typename offset_t>
__global__ void reorder_row_lengths_kernel(const offset_t *row_lengths, int num_row_lengths,
                                           offset_t *bucket_range, int batch_size_per_gpu,
                                           int num_gpu, int num_embedding) {
  for (int tid = threadIdx.x + blockIdx.x * blockDim.x; tid < num_row_lengths;
       tid += blockDim.x * gridDim.x) {
    int gpu_id = tid / (batch_size_per_gpu * num_embedding);
    int embedding_id = (tid / batch_size_per_gpu) % num_embedding;
    int batch_id = tid % batch_size_per_gpu;

    int reorder_id =
        embedding_id * batch_size_per_gpu * num_gpu + gpu_id * batch_size_per_gpu + batch_id;
    bucket_range[1 + reorder_id] = row_lengths[tid];
  }
  if (threadIdx.x + blockIdx.x * blockDim.x == 0) {
    bucket_range[0] = 0;
  }
}

template <typename key_t, typename offset_t>
__global__ void reorder_key_kernel(const key_t *key, const offset_t *row_offsets,
                                   int num_row_lengths, const offset_t *bucket_range,
                                   key_t *reorder_key, int batch_size_per_gpu, int num_gpu,
                                   int num_embedding) {
  for (int tid = threadIdx.x + blockIdx.x * blockDim.x; tid < num_row_lengths;
       tid += blockDim.x * gridDim.x) {
    int gpu_id = tid / (batch_size_per_gpu * num_embedding);
    int embedding_id = (tid / batch_size_per_gpu) % num_embedding;
    int batch_id = tid % batch_size_per_gpu;

    int reorder_id =
        embedding_id * batch_size_per_gpu * num_gpu + gpu_id * batch_size_per_gpu + batch_id;
    offset_t start = (tid == 0) ? 0 : row_offsets[tid];
    offset_t end = row_offsets[tid + 1];
    for (offset_t r = 0; r < (end - start); ++r) {
      reorder_key[bucket_range[reorder_id] + r] = key[start + r];
    }
  }
}

}  // namespace

All2AllEmbeddingCollectionSwizzleKey::All2AllEmbeddingCollectionSwizzleKey(
    std::shared_ptr<CoreResourceManager> core)
    : core_(core) {}

void All2AllEmbeddingCollectionSwizzleKey::sparse_forward_per_gpu(
    const std::vector<Tensor> &keys, const std::vector<Tensor> &row_lengths,
    Tensor &key_all_gather_send_buffer, Tensor &row_lengths_all_gather_send_buffer) {
  size_t key_bytes_offset = 0;
  size_t row_lengths_bytes_offset = 0;
  key_all_gather_send_buffer.get();
  for (size_t i = 0; i < keys.size(); ++i) {
    keys[i].get();
    HCTR_LIB_THROW(hipMemcpyAsync(
        reinterpret_cast<char *>(key_all_gather_send_buffer.get()) + key_bytes_offset,
        keys[i].get(), keys[i].nbytes(), hipMemcpyDeviceToDevice,
        core_->get_local_gpu()->get_stream()));
    key_bytes_offset += keys[i].nbytes();

    HCTR_LIB_THROW(
        hipMemcpyAsync(reinterpret_cast<char *>(row_lengths_all_gather_send_buffer.get()) +
                            row_lengths_bytes_offset,
                        row_lengths[i].get(), row_lengths[i].nbytes(), hipMemcpyDeviceToDevice,
                        core_->get_local_gpu()->get_stream()));
    row_lengths_bytes_offset += row_lengths[i].nbytes();
  }
}

All2AllEmbeddingCollectionModelForward::All2AllEmbeddingCollectionModelForward(
    std::shared_ptr<CoreResourceManager> core, const UniformModelParallelEmbeddingMeta &meta)
    : core_(core), meta_(meta) {}

std::vector<size_t> All2AllEmbeddingCollectionModelForward::get_model_comm_buffer_size(
    int batch_size) {
  int num_gpus = core_->get_global_gpu_count();
  size_t num_ev_elements = 0;
  int batch_size_per_gpu = batch_size / num_gpus;
  for (int lookup_id : meta_.h_local_lookup_id_list_) {
    int ev_size = meta_.h_ev_size_list_[lookup_id];
    num_ev_elements += ev_size * batch_size_per_gpu;
  }
  return std::vector<size_t>(num_gpus, num_ev_elements);
}

void All2AllEmbeddingCollectionModelForward::sparse_forward_per_gpu(
    const Tensor &key_all_gather_recv_buffer, const Tensor &row_lengths_all_gather_recv_buffer,
    ILookup *emb_storage, std::vector<Tensor> &emb_vec_model_buffer, int64_t *num_model_key,
    int64_t *num_model_offsets) {
  HugeCTR::CudaDeviceContext context(core_->get_device_id());

  int num_gpus = core_->get_global_gpu_count();
  hipStream_t stream = core_->get_local_gpu()->get_stream();
  int batch_size = row_lengths_all_gather_recv_buffer.get_num_elements() / meta_.num_lookup_;

  Tensor keys, bucket_range;
  size_t num_keys = static_cast<size_t>(key_all_gather_recv_buffer.get_num_elements());
  // the shape of key_all_gather is (num_gpus, num_embedding, batch_size_per_gpu)
  // the shape of key is (num_embedding, batch_size)
  auto reorder_from_all_gather_input = [&] {
    Tensor all_gather_row_offsets;

    auto buffer_ptr = GetBuffer(core_);
    keys = buffer_ptr->reserve({key_all_gather_recv_buffer.get_num_elements()},
                               key_all_gather_recv_buffer.device(),
                               key_all_gather_recv_buffer.dtype());
    bucket_range = buffer_ptr->reserve({row_lengths_all_gather_recv_buffer.get_num_elements() + 1},
                                       row_lengths_all_gather_recv_buffer.device(),
                                       row_lengths_all_gather_recv_buffer.dtype());
    all_gather_row_offsets = buffer_ptr->reserve(
        {row_lengths_all_gather_recv_buffer.get_num_elements() + 1},
        row_lengths_all_gather_recv_buffer.device(), row_lengths_all_gather_recv_buffer.dtype());
    buffer_ptr->allocate();

    auto get_bucket_range = [&] {
      DISPATCH_INTEGRAL_FUNCTION(row_lengths_all_gather_recv_buffer.dtype().type(), offset_t, [&] {
        constexpr int block_size = 256;
        int grid_size =
            (row_lengths_all_gather_recv_buffer.get_num_elements() - 1) / block_size + 1;

        reorder_row_lengths_kernel<<<grid_size, block_size, 0, stream>>>(
            row_lengths_all_gather_recv_buffer.get<offset_t>(),
            row_lengths_all_gather_recv_buffer.get_num_elements(), bucket_range.get<offset_t>(),
            batch_size / num_gpus, num_gpus, meta_.num_lookup_);

        size_t temp_bytes = 0;
        Tensor temp_scan_storage;
        hipcub::DeviceScan::InclusiveSum(nullptr, temp_bytes, (offset_t *)nullptr, (offset_t *)nullptr,
                                      bucket_range.get_num_elements());
        temp_scan_storage =
            buffer_ptr->reserve(temp_bytes, DeviceType::GPU, TensorScalarType::Void);
        buffer_ptr->allocate();

        hipcub::DeviceScan::InclusiveSum(temp_scan_storage.get(), temp_bytes,
                                      bucket_range.get<offset_t>(), bucket_range.get<offset_t>(),
                                      bucket_range.get_num_elements(), stream);

        // HCTR_LIB_THROW(hipStreamSynchronize(stream));

        // std::vector<offset_t> gpu_bucket_range;
        // bucket_range.to(&gpu_bucket_range);
        // std::cout << "gpu_bucket_range:\n";
        // for (auto i : gpu_bucket_range) {
        //   std::cout << i << " ";
        // }
        // std::cout << "\n";
      });
    };

    auto scan_row_lengths = [&] {
      DISPATCH_INTEGRAL_FUNCTION(row_lengths_all_gather_recv_buffer.dtype().type(), offset_t, [&] {
        size_t temp_bytes = 0;
        Tensor temp_scan_storage;
        hipcub::DeviceScan::InclusiveSum(nullptr, temp_bytes, (offset_t *)nullptr, (offset_t *)nullptr,
                                      row_lengths_all_gather_recv_buffer.get_num_elements() + 1);
        temp_scan_storage =
            buffer_ptr->reserve(temp_bytes, DeviceType::GPU, TensorScalarType::Void);
        buffer_ptr->allocate();

        hipcub::DeviceScan::InclusiveSum(
            temp_scan_storage.get(), temp_bytes, row_lengths_all_gather_recv_buffer.get<offset_t>(),
            all_gather_row_offsets.get<offset_t>() + 1,
            row_lengths_all_gather_recv_buffer.get_num_elements(), stream);
        // HCTR_LIB_THROW(hipStreamSynchronize(stream));

        // std::vector<offset_t> gpu_row_lengths;
        // row_lengths_all_gather_recv_buffer.to(&gpu_row_lengths);
        // std::cout << "gpu_row_lengths:\n";
        // for (auto i : gpu_row_lengths) {
        //   std::cout << i << " ";
        // }
        // std::cout << "\n";

        // std::vector<offset_t> gpu_row_offsets;
        // all_gather_row_offsets.to(&gpu_row_offsets);
        // std::cout << "gpu_row_offsets:\n";
        // for (auto i : gpu_row_offsets) {
        //   std::cout << i << " ";
        // }
        // std::cout << "\n";
      });
    };

    auto reorder_key = [&] {
      DISPATCH_INTEGRAL_FUNCTION(key_all_gather_recv_buffer.dtype().type(), key_t, [&] {
        DISPATCH_INTEGRAL_FUNCTION(
            row_lengths_all_gather_recv_buffer.dtype().type(), offset_t, [&] {
              constexpr int block_size = 256;
              int grid_size =
                  (row_lengths_all_gather_recv_buffer.get_num_elements() - 1) / block_size + 1;
              reorder_key_kernel<<<grid_size, block_size, 0, stream>>>(
                  key_all_gather_recv_buffer.get<key_t>(), all_gather_row_offsets.get<offset_t>(),
                  row_lengths_all_gather_recv_buffer.get_num_elements(),
                  bucket_range.get<offset_t>(), keys.get<key_t>(), batch_size / num_gpus, num_gpus,
                  meta_.num_lookup_);
              // HCTR_LIB_THROW(hipStreamSynchronize(stream));

              // std::vector<key_t> gpu_all_gather_key;
              // key_all_gather_recv_buffer.to(&gpu_all_gather_key);
              // std::cout << "gpu_all_gather_key:\n";
              // for (auto i : gpu_all_gather_key) {
              //   std::cout << i << " ";
              // }
              // std::cout << "\n";

              // std::vector<offset_t> gpu_reorder_key;
              // keys.to(&gpu_reorder_key);
              // std::cout << "gpu_reorder_key:\n";
              // for (auto i : gpu_reorder_key) {
              //   std::cout << i << " ";
              // }
              // std::cout << "\n";
            });
      });
    };

    get_bucket_range();
    scan_row_lengths();
    reorder_key();
  };
  reorder_from_all_gather_input();

  DataType key_type = key_all_gather_recv_buffer.dtype();
  model_index_calculation_ =
      ModelIndexCalculation(core_, meta_.num_local_lookup_, meta_.num_local_hotness_,
                            meta_.hotness_sum_, batch_size, key_type);

  Tensor model_key, model_offsets;
  size_t num_model_key_;
  model_index_calculation_.compute(keys, bucket_range, num_keys, meta_.d_local_lookup_id_list_,
                                   meta_.d_local_shard_id_list_, meta_.d_local_num_shards_list_,
                                   batch_size, &model_key, &model_offsets, &num_model_key_);

  compress_offset_ = CompressOffset(core_, meta_.num_local_lookup_ + 1);
  Tensor num_key_per_lookup_offset;
  compress_offset_.compute(model_offsets, batch_size, &num_key_per_lookup_offset);

  HCTR_LIB_THROW(hipStreamSynchronize(stream));
  TensorList embedding_vec = TensorList(core_.get(), key_all_gather_recv_buffer.get_num_elements(),
                                        DeviceType::GPU, TensorScalarType::Float32);
  emb_storage->lookup(model_key, num_model_key_, num_key_per_lookup_offset,
                      meta_.num_local_lookup_ + 1, meta_.d_local_table_id_list_, embedding_vec);

  model_forward_ = ModelForward(core_, num_gpus, meta_.h_local_lookup_id_list_);

  TensorList model_comm_buffer{core_.get(), emb_vec_model_buffer, DeviceType::GPU,
                               emb_vec_model_buffer[0].dtype(), stream};
  model_forward_.compute(embedding_vec, model_offsets, model_comm_buffer,
                         meta_.d_local_ev_size_list_, meta_.d_local_ev_size_offset_, batch_size,
                         meta_.max_ev_size_);

  model_key_ = model_key;
  model_offsets_ = model_offsets;
  *num_model_key = static_cast<int64_t>(num_model_key_);
  *num_model_offsets = model_offsets.get_num_elements();
}

void All2AllEmbeddingCollectionModelForward::copy_model_keys_and_offsets(Tensor &model_key,
                                                                         Tensor &model_offsets) {
  HCTR_LIB_THROW(hipMemcpyAsync(model_key.get(), model_key_.get(), model_key.nbytes(),
                                 hipMemcpyDeviceToDevice, core_->get_local_gpu()->get_stream()));
  HCTR_LIB_THROW(hipMemcpyAsync(model_offsets.get(), model_offsets_.get(), model_offsets.nbytes(),
                                 hipMemcpyDeviceToDevice, core_->get_local_gpu()->get_stream()));
}

All2AllEmbeddingCollectionNetworkForward::All2AllEmbeddingCollectionNetworkForward(
    std::shared_ptr<CoreResourceManager> core, const UniformModelParallelEmbeddingMeta &meta)
    : core_(core), meta_(meta) {
  int num_gpus = core->get_global_gpu_count();
  network_forward_ = NetworkForward(core, num_gpus);
}

void All2AllEmbeddingCollectionNetworkForward::sparse_forward_per_gpu(
    const std::vector<Tensor> &emb_vec_network_buffer, const std::vector<Tensor> &row_lengths,
    std::vector<Tensor> &forward_emb_vec) {
  HugeCTR::CudaDeviceContext context(core_->get_device_id());
  hipStream_t stream = core_->get_local_gpu()->get_stream();
  int num_gpus = core_->get_global_gpu_count();
  int batch_size_per_gpu = row_lengths[0].get_num_elements();
  int batch_size = batch_size_per_gpu * num_gpus;
  int global_gpu_id = core_->get_global_gpu_id();

  TensorList row_lengths_buffer{core_.get(), row_lengths, DeviceType::GPU, row_lengths[0].dtype(),
                                stream};
  TensorList network_comm_buffer{core_.get(), emb_vec_network_buffer, DeviceType::GPU,
                                 emb_vec_network_buffer[0].dtype(), stream};
  TensorList output_buffer{core_.get(), forward_emb_vec, DeviceType::GPU,
                           forward_emb_vec[0].dtype(), stream};
  network_forward_.compute(row_lengths_buffer, meta_.d_combiner_list_, network_comm_buffer,
                           meta_.network_ids_, meta_.network_gpu_ids_, meta_.network_offsets_,
                           meta_.network_dst_lookup_ids_, meta_.network_ev_sizes_,
                           meta_.network_ev_offsets_, output_buffer, meta_.d_ev_size_offset_,
                           batch_size, meta_.max_ev_size_);
}

All2AllEmbeddingCollectionNetworkBackward::All2AllEmbeddingCollectionNetworkBackward(
    std::shared_ptr<CoreResourceManager> core, const UniformModelParallelEmbeddingMeta &meta)
    : core_(core), meta_(meta) {
  int num_gpus = core->get_global_gpu_count();
  network_backward_ = NetworkBackward(core, num_gpus);
}

void All2AllEmbeddingCollectionNetworkBackward::backward_per_gpu(
    const std::vector<Tensor> &top_grad, const std::vector<Tensor> &row_lengths,
    std::vector<Tensor> &emb_vec_network_buffer) {
  HugeCTR::CudaDeviceContext context(core_->get_device_id());
  hipStream_t stream = core_->get_local_gpu()->get_stream();
  // int batch_size = (top_grad[0].get_num_elements() * core_->get_global_gpu_count()) /
  // meta_.h_ev_size_list_[0];
  int num_gpus = core_->get_global_gpu_count();
  int batch_size_per_gpu = row_lengths[0].get_num_elements();
  int batch_size = batch_size_per_gpu * num_gpus;

  TensorList row_lengths_buffer{core_.get(), row_lengths, DeviceType::GPU, row_lengths[0].dtype(),
                                stream};
  TensorList network_comm_buffer{core_.get(), emb_vec_network_buffer, DeviceType::GPU,
                                 emb_vec_network_buffer[0].dtype(), stream};
  TensorList top_grad_buffer{core_.get(), top_grad, DeviceType::GPU, top_grad[0].dtype(), stream};

  network_backward_.compute(row_lengths_buffer, meta_.d_combiner_list_, top_grad_buffer,
                            meta_.network_ids_, meta_.network_gpu_ids_, meta_.network_offsets_,
                            meta_.network_dst_lookup_ids_, meta_.network_ev_sizes_,
                            meta_.network_ev_offsets_, network_comm_buffer, meta_.d_ev_size_offset_,
                            batch_size, meta_.max_ev_size_);
}

All2AllEmbeddingCollectionModelBackward::All2AllEmbeddingCollectionModelBackward(
    std::shared_ptr<CoreResourceManager> core, const UniformModelParallelEmbeddingMeta &meta)
    : core_(core), meta_(meta) {}

void All2AllEmbeddingCollectionModelBackward::sparse_backward_per_gpu(
    const std::vector<Tensor> &emb_vec_model_buffer, const Tensor &model_key,
    const Tensor &model_offsets, std::vector<int> *num_unique_key_per_table,
    std::vector<int> *table_id_list) {
  HugeCTR::CudaDeviceContext context(core_->get_device_id());
  int num_gpus = core_->get_global_gpu_count();
  hipStream_t stream = core_->get_local_gpu()->get_stream();
  int batch_size = (model_offsets.get_num_elements() - 1) / meta_.num_local_lookup_;
  size_t num_model_key = static_cast<size_t>(model_key.get_num_elements());

  Tensor num_key_per_lookup_offset;
  CompressOffset compress_offset{core_, meta_.num_local_lookup_ + 1};
  compress_offset.compute(model_offsets, batch_size, &num_key_per_lookup_offset);

  model_backward_index_calculation_ = ModelBackwardIndexCalculation(
      core_, num_gpus, meta_.num_local_lookup_, meta_.h_local_hotness_list_,
      meta_.h_local_table_id_list_, meta_.h_local_ev_size_list_, batch_size, model_key.dtype());

  Tensor continous_unique_key, wgrad_idx_offset, sorted_bucket_id_list, sorted_bucket_id_offset,
      d_table_id_list, num_unique_key_per_table_offset, continous_grad_emb_ev, coordinate_key,
      coordinate_wgrad_dst_idx;
  size_t num_unique_key;
  model_backward_index_calculation_.compute(
      model_key, num_model_key, model_offsets, num_key_per_lookup_offset,
      meta_.d_local_table_id_list_, batch_size, &continous_unique_key, &num_unique_key,
      &wgrad_idx_offset, &sorted_bucket_id_list, &sorted_bucket_id_offset, &d_table_id_list,
      &num_unique_key_per_table_offset, &coordinate_key, &coordinate_wgrad_dst_idx);

  model_backward_ =
      ModelBackward(core_, num_gpus, meta_.num_local_lookup_, meta_.h_local_hotness_list_,
                    meta_.h_local_ev_size_list_, batch_size, meta_.max_ev_size_, meta_.num_sms_);

  TensorList model_comm_buffer{core_.get(), emb_vec_model_buffer, DeviceType::GPU,
                               emb_vec_model_buffer[0].dtype(), stream};
  model_backward_.compute(model_comm_buffer, wgrad_idx_offset, sorted_bucket_id_list,
                          sorted_bucket_id_offset, num_unique_key, coordinate_key,
                          coordinate_wgrad_dst_idx, meta_.d_local_ev_size_offset_, batch_size,
                          meta_.max_ev_size_, num_model_key, &continous_grad_emb_ev);
  d_table_id_list.to(table_id_list, stream);
  continous_unique_key_ = continous_unique_key;
  continous_emb_vec_ = continous_grad_emb_ev;
  HCTR_LIB_THROW(hipStreamSynchronize(stream));
  std::vector<uint32_t> gpu_num_key_per_table_offset;
  num_unique_key_per_table_offset.to(&gpu_num_key_per_table_offset);

  num_unique_key_per_table->resize(d_table_id_list.get_num_elements());
  for (int i = 0; i < d_table_id_list.get_num_elements(); ++i) {
    (*num_unique_key_per_table)[i] =
        gpu_num_key_per_table_offset[i + 1] - gpu_num_key_per_table_offset[i];
  }
}

void All2AllEmbeddingCollectionModelBackward::copy_backward_key_and_emb_vec(
    std::vector<Tensor> &unique_key, std::vector<Tensor> &emb_vec) {
  size_t nbytes_key_offsets = 0ul;
  size_t nbytes_emb_vec_offsets = 0ul;
  for (size_t i = 0; i < unique_key.size(); ++i) {
    HCTR_LIB_THROW(hipMemcpyAsync(
        unique_key[i].get(),
        reinterpret_cast<char *>(continous_unique_key_.get()) + nbytes_key_offsets,
        unique_key[i].nbytes(), hipMemcpyDeviceToDevice, core_->get_local_gpu()->get_stream()));
    HCTR_LIB_THROW(hipMemcpyAsync(
        emb_vec[i].get(),
        reinterpret_cast<char *>(continous_emb_vec_.get()) + nbytes_emb_vec_offsets,
        emb_vec[i].nbytes(), hipMemcpyDeviceToDevice, core_->get_local_gpu()->get_stream()));
    nbytes_key_offsets += unique_key[i].nbytes();
    nbytes_emb_vec_offsets += emb_vec[i].nbytes();
  }
}

}  // namespace tf
}  // namespace embedding
