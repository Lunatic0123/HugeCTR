/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "parameters/dumping_functions.h"
#include "common/include/forward_functions.cuh"

namespace SparseOperationKit {

void get_hash_value(size_t count, size_t embedding_vec_size, const size_t *value_index,
                const float *embedding_table, float *value_retrieved,
                hipStream_t stream) {
const size_t block_size = embedding_vec_size;
const size_t grid_size = count;

HugeCTR::get_hash_value_kernel<<<grid_size, block_size, 0, stream>>>(count, embedding_vec_size,
                                                value_index, embedding_table, value_retrieved);
}

} // namespace SparseOperationKit