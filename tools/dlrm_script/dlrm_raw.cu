#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <core23/logger.hpp>
#include <dlrm_raw_utils.hpp>

using namespace DLRM_RAW;

template <typename T>
auto dtype() {
  return cudf::data_type{cudf::type_to_id<T>()};
}

void process_kaggle_dataset(const std::string &input_dir_path, const std::string &output_dir_path,
                            const int num_numericals, const int num_categoricals) {
  int max_chunk_per_file = 10000;  // loop count, in a single binary data, store how many chunks

  bool process_output = true;
  bool write_out = true;

  // int32_t hash_bucket = 40000000;  // mod-idx
  // int max_cat_fea_cardi = 40000000;  // 40M
  // int avg_cat_fea_cardi = 1000000;    // 1M
  // int min_cat_fea_cardi = 1000000;    // 1M
  // std::vector<int32_t> hist_sizes = {max_cat_fea_cardi, avg_cat_fea_cardi, avg_cat_fea_cardi,
  // avg_cat_fea_cardi, avg_cat_fea_cardi,
  //                                    min_cat_fea_cardi, min_cat_fea_cardi, min_cat_fea_cardi,
  //                                    min_cat_fea_cardi, max_cat_fea_cardi, max_cat_fea_cardi,
  //                                    avg_cat_fea_cardi, min_cat_fea_cardi, min_cat_fea_cardi,
  //                                    avg_cat_fea_cardi, min_cat_fea_cardi, min_cat_fea_cardi,
  //                                    min_cat_fea_cardi, min_cat_fea_cardi, max_cat_fea_cardi,
  //                                    max_cat_fea_cardi, max_cat_fea_cardi, avg_cat_fea_cardi,
  //                                    min_cat_fea_cardi, min_cat_fea_cardi, min_cat_fea_cardi}; //
  //                                    mod-idx

  int min_cat_fea_cardi = 10000000;  // 10M
  int32_t hash_bucket = min_cat_fea_cardi;
  std::vector<int32_t> hist_sizes = {
      hash_bucket, hash_bucket, hash_bucket, hash_bucket, hash_bucket, hash_bucket, hash_bucket,
      hash_bucket, hash_bucket, hash_bucket, hash_bucket, hash_bucket, hash_bucket, hash_bucket,
      hash_bucket, hash_bucket, hash_bucket, hash_bucket, hash_bucket, hash_bucket, hash_bucket,
      hash_bucket, hash_bucket, hash_bucket, hash_bucket, hash_bucket};  // mod-idx

  size_t pool_alloc_size = (size_t)4 * 1024 * 1024 * 1024;  // 4 GB
  // std::vector<int> dev = {0};
  rmm::mr::device_memory_resource *base_mr = new rmm::mr::cuda_memory_resource();
  auto *p_mr =
      new rmm::mr::pool_memory_resource<rmm::mr::device_memory_resource>(base_mr, pool_alloc_size);
  rmm::mr::set_current_device_resource(p_mr);

  std::vector<cudf::data_type> column_dtypes;             // dtypes of label, dense, categorical
  std::vector<std::string> column_names;                  // names of label, dense, categorical
  std::vector<std::string> cat_column_names;              // names of categorical
  std::map<std::string, int32_t> column_name_to_col_idx;  // <col-name, idx>
  std::unordered_map<std::string, map_type<key_type, value_type> *>
      categorical_col_hash_tables;  // <name, <key, value>>

  // label
  column_dtypes.push_back(dtype<int32_t>());
  column_names.push_back("label");
  column_name_to_col_idx.insert(std::make_pair("label", 0));

  // dense-features
  for (int k = 1; k <= 13; k++) {
    column_dtypes.push_back(dtype<int32_t>());
    std::string name = "I" + std::to_string(k);
    column_names.push_back(name);
    column_name_to_col_idx.insert(std::make_pair(name, k));
  }

  // categorical-features
  for (int k = 1; k <= num_categoricals; k++) {
    column_dtypes.push_back(dtype<cudf::string_view>());
    std::string name = "C" + std::to_string(k);
    column_names.push_back(name);
    cat_column_names.push_back(name);
    column_name_to_col_idx.insert(std::make_pair(name, k + num_numericals - 1));

    auto cuda_map_obj =
        map_type<key_type, value_type>::create(compute_hash_table_size(hist_sizes[k - 1]))
            .release();
    ;
    categorical_col_hash_tables.insert(std::make_pair(name, cuda_map_obj));
  }

  int current_device = 0;
  hipDeviceProp_t prop;
  HCTR_LIB_THROW(hipGetDeviceProperties(&prop, current_device));

  size_t read_chunks = 128 * 1024 * 1024;  // read 128MB at one time

  uint32_t *accum_location = nullptr;                // slot-size
  HCTR_LIB_THROW(hipMalloc(&accum_location, 128));  // 128 Bytes = 32 * uint32_t
  HCTR_LIB_THROW(hipMemset(accum_location, 0, 128));

  // uint32_t *culled_index_count = nullptr;
  // HCTR_LIB_THROW(hipMalloc(&culled_index_count, 128)); // 128 Bytes = 32 * uint32_t

  size_t total_file_bytes_read = 0;
  const auto time_map_start = std::chrono::high_resolution_clock::now();

  // get file size, hard-coded filename
  std::string input_file_name = std::string(input_dir_path + "/train.txt");
  std::ifstream binary_reader(input_file_name, std::ios::binary);
  binary_reader.seekg(0, std::ios::end);
  size_t file_size = binary_reader.tellg();
  binary_reader.close();

  // csv arguments,
  // https://docs.rapids.ai/api/libcudf/stable/structcudf_1_1io_1_1read__csv__args.html
  cudf_io::csv_reader_options in_args =
      cudf_io::csv_reader_options::builder(cudf_io::source_info{input_file_name}).header(-1);
  in_args.set_dtypes(column_dtypes);
  in_args.set_names(column_names);
  in_args.set_delimiter('\t');
  in_args.set_byte_range_size(read_chunks);  // how many bytes to read at one time.
  in_args.set_skipfooter(0);
  in_args.set_skiprows(0);
  in_args.set_use_cols_names(cat_column_names);

  int32_t total_row_nums = 0;

  int loop_count = 0;
  while (true) {
    total_file_bytes_read += in_args.get_byte_range_size();
#if defined(CUDF_VERSION_MAJOR) && CUDF_VERSION_MAJOR == 23 && defined(CUDF_VERSION_MINOR) && \
    CUDF_VERSION_MINOR >= 12
    cudf_io::table_with_metadata tbl_w_metadata =
        cudf_io::read_csv(in_args, cudf::get_default_stream(), p_mr);
#elif defined(CUDF_VERSION_MAJOR) && CUDF_VERSION_MAJOR > 23
    auto tbl_w_metadata = cudf_io::read_csv(in_args, cudf::get_default_stream(), mr);
#else
    cudf_io::table_with_metadata tbl_w_metadata = cudf_io::read_csv(in_args, p_mr);
#endif
    total_row_nums += tbl_w_metadata.tbl->num_rows();

    dim3 block(prop.maxThreadsPerBlock, 1, 1);
    dim3 grid((tbl_w_metadata.tbl->num_rows() - 1) / block.x + 1, 1, 1);

    // categorical-features
    for (unsigned int k = 0; k < cat_column_names.size(); ++k) {
      auto col = std::move(tbl_w_metadata.tbl->get_column(k));
      if (col.type().id() == cudf::type_id::STRING) {
        auto str_col = cudf::strings_column_view(col.view());
        int64_t num_strings = str_col.size();
        char *char_array = const_cast<char *>(str_col.chars().data<char>());
        int32_t *offsets = const_cast<int32_t *>(str_col.offsets().data<int32_t>());

        build_categorical_index<key_type, value_type><<<grid, block>>>(
            char_array, offsets, num_strings,
            // *categorical_col_hash_tables[cat_column_names[k]], hash_bucket, &accum_location[k]);
            *categorical_col_hash_tables[cat_column_names[k]], hist_sizes[k], &accum_location[k]);

      } else if (col.type().id() == cudf::type_id::INT32) {
        key_type *data = const_cast<key_type *>(col.view().data<key_type>());
        bitmask_type *in_mask = const_cast<bitmask_type *>(col.view().null_mask());

        build_categorical_index_from_ints<key_type, value_type><<<grid, block>>>(
            data, in_mask, tbl_w_metadata.tbl->num_rows(),
            // *categorical_col_hash_tables[cat_column_names[k]], hash_bucket, &accum_location[k]);
            *categorical_col_hash_tables[cat_column_names[k]], hist_sizes[k], &accum_location[k]);

      } else {
        HCTR_LOG_S(ERROR, WORLD) << "col.type().id() != [STRING, INT32] " << HCTR_LOCATION()
                                 << std::endl;
      }
    }

    size_t new_byte_range_offset = in_args.get_byte_range_offset() + read_chunks;
    in_args.set_byte_range_offset(new_byte_range_offset);
    if (in_args.get_byte_range_offset() >= file_size) break;

    if ((in_args.get_byte_range_offset() + read_chunks) > file_size) {
      size_t new_byte_range_size = file_size - in_args.get_byte_range_offset();
      in_args.set_byte_range_size(new_byte_range_size);
    }

    ++loop_count;

    if (loop_count == max_chunk_per_file) break;
  }
  HCTR_LOG_S(INFO, ROOT) << input_file_name << "'s total rows number = " << total_row_nums
                         << std::endl;

  // show: slot size array
  std::vector<uint32_t> host_sz_per_fea(num_categoricals);
  HCTR_LIB_THROW(hipMemcpy(host_sz_per_fea.data(), accum_location,
                            num_categoricals * sizeof(uint32_t), hipMemcpyDeviceToHost));
  HCTR_LOG_S(INFO, ROOT) << "Slot size array in " << input_file_name
                         << ", missing value mapped to unused key: " << std::endl;
  for (auto c : host_sz_per_fea) std::cout << (c) << ", ";
  std::cout << "\b\b" << std::endl;

  const auto time_map_stop = std::chrono::high_resolution_clock::now();
  const auto time_map_build =
      std::chrono::duration_cast<std::chrono::milliseconds>(time_map_stop - time_map_start);
  HCTR_LOG_S(INFO, ROOT) << "Time used to build map: " << time_map_build.count() << " milliseconds."
                         << std::endl;

  double read_bw = double(total_file_bytes_read) / (1024.0 * 1024.0 * 1024.0);
  read_bw = (read_bw / time_map_build.count()) * 1000.f;
  HCTR_LOG_S(INFO, ROOT) << "Total bytes read: " << total_file_bytes_read
                         << " Effective Read B/W: " << read_bw << " GB/s." << std::endl;

  // HCTR_LIB_THROW(hipFree(culled_index_count));
  HCTR_LIB_THROW(hipFree(accum_location));

  // starting to do the conversion
  if (process_output) {
    uint32_t *dev_slot_size_array = nullptr;
    size_t slot_size_array_size = num_categoricals * sizeof(uint32_t);
    HCTR_LIB_THROW(hipMalloc(&dev_slot_size_array, slot_size_array_size));
    HCTR_LIB_THROW(hipMemcpy(dev_slot_size_array, host_sz_per_fea.data(), slot_size_array_size,
                              hipMemcpyHostToDevice));

    int32_t *dev_out_buffer = nullptr;
    int32_t *host_out_buffer = nullptr;

    size_t sz_output_buffer = 128 * 1024 * 1024;  // 128 MB, = read_chunks
    HCTR_LIB_THROW(hipMalloc(&dev_out_buffer, sz_output_buffer));
    HCTR_LIB_THROW(hipHostMalloc(&host_out_buffer, sz_output_buffer));

    int64_t *dev_int_col_ptrs = nullptr;
    int64_t *dev_int_col_nullmask_ptrs = nullptr;
    int64_t *dev_cat_col_nullmask_ptrs = nullptr;
    int64_t *dev_categorical_col_hash_obj = nullptr;
    int64_t *dev_char_ptrs = nullptr;
    int64_t *dev_offset_ptrs = nullptr;

    size_t sz_dev_int_col = num_numericals * sizeof(int64_t);
    size_t sz_dev_cat_hash_obj = num_categoricals * sizeof(map_type<key_type, value_type>);
    size_t sz_dev_str_ptrs = num_categoricals * sizeof(int64_t);

    HCTR_LIB_THROW(hipMalloc(&dev_int_col_ptrs, sz_dev_int_col));
    HCTR_LIB_THROW(hipMalloc(&dev_int_col_nullmask_ptrs, sz_dev_int_col));
    HCTR_LIB_THROW(hipMalloc(&dev_cat_col_nullmask_ptrs, sz_dev_str_ptrs));
    HCTR_LIB_THROW(hipMalloc(&dev_categorical_col_hash_obj, sz_dev_cat_hash_obj));
    HCTR_LIB_THROW(hipMalloc(&dev_char_ptrs, sz_dev_str_ptrs));
    HCTR_LIB_THROW(hipMalloc(&dev_offset_ptrs, sz_dev_str_ptrs));

    // encode and write out binary
    int maxbytes = 96 * 1024;  // dynamic shared memory size 96 KB
    hipFuncSetAttribute(reinterpret_cast<const void*>(process_data_rows<key_type), value_type>,
                         hipFuncAttributeMaxDynamicSharedMemorySize, maxbytes);

    std::vector<map_type<key_type, value_type>> categorical_col_hash_obj;
    for (auto c : cat_column_names) {
      categorical_col_hash_obj.push_back(*categorical_col_hash_tables[c]);
    }

    HCTR_LIB_THROW(hipMemcpy((void *)dev_categorical_col_hash_obj,
                              (void *)categorical_col_hash_obj.data(), sz_dev_cat_hash_obj,
                              hipMemcpyHostToDevice));

    if (process_output) {
      std::ofstream *binary_writer = nullptr;

      if (write_out)
        binary_writer =
            new std::ofstream(std::string(output_dir_path + "/train_data.bin"), std::ios::binary);
      size_t sz_total_output_binary = 0;

      const auto time_convert_start = std::chrono::high_resolution_clock::now();

      // train_data.bin
      {
        int32_t rows_begin_train = 0, rows_end_train = 36672493;  // train.txt [:36672493)
        std::string input_file_path = std::string(input_dir_path + "/train.txt");
        sz_total_output_binary = convert_input_binaries<key_type, value_type>(
            p_mr, input_file_path, column_dtypes, column_names, hash_bucket, max_chunk_per_file, 0,
            false, dev_int_col_ptrs, dev_int_col_nullmask_ptrs, dev_cat_col_nullmask_ptrs,
            dev_categorical_col_hash_obj, dev_char_ptrs, dev_offset_ptrs, dev_out_buffer,
            host_out_buffer, binary_writer, dev_slot_size_array, rows_begin_train, rows_end_train,
            3);

        HCTR_LOG_S(INFO, ROOT) << "Processed file: " << input_file_path << " for /train_data.bin"
                               << std::endl;
        HCTR_LOG_S(INFO, ROOT) << "Size of train_data.bin: " << sz_total_output_binary << " Bytes."
                               << std::endl;

        if (binary_writer) binary_writer->close();
      }

      // validation-data and testing-data
      {
        int32_t rows_begin_val = 36672493,
                rows_end_val = 41256555;  // train.txt [36672493, 41256555)
        int32_t rows_begin_test = 41256555,
                rows_end_test = 45840617;  // train.txt [41256555, 45840617]
        std::string input_file_path = std::string(input_dir_path + "/train.txt");

        // val
        std::ofstream *binary_writer_val = nullptr;
        if (write_out)
          binary_writer_val =
              new std::ofstream(std::string(output_dir_path + "/val_data.bin"), std::ios::binary);

        sz_total_output_binary = convert_input_binaries<key_type, value_type>(
            p_mr, input_file_path, column_dtypes, column_names, hash_bucket, max_chunk_per_file, 0,
            false, dev_int_col_ptrs, dev_int_col_nullmask_ptrs, dev_cat_col_nullmask_ptrs,
            dev_categorical_col_hash_obj, dev_char_ptrs, dev_offset_ptrs, dev_out_buffer,
            host_out_buffer, binary_writer_val, dev_slot_size_array, rows_begin_val, rows_end_val,
            3);

        HCTR_LOG_S(INFO, ROOT) << "Size of val_data.bin: " << sz_total_output_binary << " Bytes."
                               << std::endl;

        if (binary_writer_val) binary_writer_val->close();

        // test
        std::ofstream *binary_writer_test = nullptr;
        if (write_out)
          binary_writer_test =
              new std::ofstream(std::string(output_dir_path + "/test_data.bin"), std::ios::binary);

        sz_total_output_binary = convert_input_binaries<key_type, value_type>(
            p_mr, input_file_path, column_dtypes, column_names, hash_bucket, max_chunk_per_file, 0,
            false, dev_int_col_ptrs, dev_int_col_nullmask_ptrs, dev_cat_col_nullmask_ptrs,
            dev_categorical_col_hash_obj, dev_char_ptrs, dev_offset_ptrs, dev_out_buffer,
            host_out_buffer, binary_writer_test, dev_slot_size_array, rows_begin_test,
            rows_end_test, 3);

        HCTR_LOG_S(INFO, ROOT) << "Size of test_data.bin: " << sz_total_output_binary << " Bytes."
                               << std::endl;

        if (binary_writer_test) binary_writer_test->close();
        HCTR_LOG_S(INFO, ROOT) << "Processed file: " << input_file_path
                               << " for val_data.bin and test_data.bin" << std::endl;
      }

      const auto time_convert_stop = std::chrono::high_resolution_clock::now();
      const auto time_convert_total = std::chrono::duration_cast<std::chrono::milliseconds>(
          time_convert_stop - time_convert_start);
      HCTR_LOG_S(INFO, ROOT) << "Time to process binaries: " << time_convert_total.count()
                             << " milliseconds." << std::endl;
      double p_read_bw = (double)process_read_bytes / (1024.0 * 1024.0 * 1024.0);
      p_read_bw = (p_read_bw / time_convert_total.count()) * 1000.f;

      double p_write_bw = (double)process_write_bytes / (1024.0 * 1024.0 * 1024.0);
      p_write_bw = (p_write_bw / time_convert_total.count()) * 1000.f;

      size_t total_second_pass_bytes = process_read_bytes + process_write_bytes;
      double p_2nd_bw = (double)total_second_pass_bytes / (1024.0 * 1024.0 * 1024.0);
      p_2nd_bw = (p_2nd_bw / time_convert_total.count()) * 1000.f;

      HCTR_LOG_S(INFO, ROOT) << "Convert Bytes reading: " << process_read_bytes
                             << ", Effective reading B/W: " << p_read_bw << " GB/s." << std::endl;
      HCTR_LOG_S(INFO, ROOT) << "Convert Bytes writing: " << process_write_bytes
                             << ", Effective reading B/W: " << p_write_bw << " GB/s." << std::endl;
      HCTR_LOG_S(INFO, ROOT) << "Convert Bytes total: " << total_second_pass_bytes
                             << ", Effective reading B/W: " << p_2nd_bw << " GB/s." << std::endl;
    }

    const auto program_end_time = std::chrono::high_resolution_clock::now();
    const auto application_time =
        std::chrono::duration_cast<std::chrono::milliseconds>(program_end_time - time_map_start);
    double app_bw = (double)total_file_bytes_read / (1024.0 * 1024.0 * 1024.0);
    app_bw = (app_bw / application_time.count()) * 1000.f;

    HCTR_LOG_S(INFO, ROOT) << "Application process B/W: " << app_bw << " GB/s." << std::endl;

    HCTR_LIB_THROW(hipFree(dev_out_buffer));
    HCTR_LIB_THROW(hipHostFree(host_out_buffer));

    HCTR_LIB_THROW(hipFree(dev_int_col_ptrs));
    HCTR_LIB_THROW(hipFree(dev_int_col_nullmask_ptrs));
    HCTR_LIB_THROW(hipFree(dev_categorical_col_hash_obj));
    HCTR_LIB_THROW(hipFree(dev_char_ptrs));
    HCTR_LIB_THROW(hipFree(dev_offset_ptrs));
    HCTR_LIB_THROW(hipFree(dev_slot_size_array));
    HCTR_LIB_THROW(hipFree(dev_cat_col_nullmask_ptrs));
  }
  // destroy map objects
  for (auto c : categorical_col_hash_tables) c.second->destroy();

  delete p_mr;
  p_mr = nullptr;
}

void process_terabyte_dataset(const std::string &input_dir_path, const std::string &output_dir_path,
                              const int num_numericals, const int num_categoricals,
                              const std::vector<std::string> &train_days,
                              const std::vector<std::string> &test_days) {
  int max_chunk_per_file = 10000;  // loop count, in a single binary data, store how many chunks

  bool process_output = true;
  bool write_out = true;

  int min_cat_fea_cardi = 40000000;  // 40M
  int32_t hash_bucket = min_cat_fea_cardi;
  std::vector<int32_t> hist_sizes = {
      hash_bucket, hash_bucket, hash_bucket, hash_bucket, hash_bucket, hash_bucket, hash_bucket,
      hash_bucket, hash_bucket, hash_bucket, hash_bucket, hash_bucket, hash_bucket, hash_bucket,
      hash_bucket, hash_bucket, hash_bucket, hash_bucket, hash_bucket, hash_bucket, hash_bucket,
      hash_bucket, hash_bucket, hash_bucket, hash_bucket, hash_bucket};  // mod-idx

  size_t pool_alloc_size = (size_t)10 * 1024 * 1024 * 1024;  // 10 GB
  rmm::mr::device_memory_resource *base_mr = new rmm::mr::cuda_memory_resource();
  auto *p_mr =
      new rmm::mr::pool_memory_resource<rmm::mr::device_memory_resource>(base_mr, pool_alloc_size);
  rmm::mr::set_current_device_resource(p_mr);

  std::vector<cudf::data_type> column_dtypes;             // dtypes of label, dense, categorical
  std::vector<std::string> column_names;                  // names of label, dense, categorical
  std::vector<std::string> cat_column_names;              // names of categorical
  std::map<std::string, int32_t> column_name_to_col_idx;  // <col-name, idx>
  std::unordered_map<std::string, map_type<key_type, value_type> *>
      categorical_col_hash_tables;  // <name, <key, value>>

  // label
  column_dtypes.push_back(dtype<int32_t>());
  column_names.push_back("label");
  column_name_to_col_idx.insert(std::make_pair("label", 0));

  // dense-features
  for (int k = 1; k <= 13; k++) {
    column_dtypes.push_back(dtype<int32_t>());
    std::string name = "I" + std::to_string(k);
    column_names.push_back(name);
    column_name_to_col_idx.insert(std::make_pair(name, k));
  }

  // categorical-features
  for (int k = 1; k <= num_categoricals; k++) {
    column_dtypes.push_back(dtype<cudf::string_view>());
    std::string name = "C" + std::to_string(k);
    column_names.push_back(name);
    cat_column_names.push_back(name);
    column_name_to_col_idx.insert(std::make_pair(name, k + num_numericals - 1));

    auto cuda_map_obj =
        map_type<key_type, value_type>::create(compute_hash_table_size(hist_sizes[k - 1]))
            .release();
    ;
    categorical_col_hash_tables.insert(std::make_pair(name, cuda_map_obj));
  }

  int current_device = 0;
  hipDeviceProp_t prop;
  HCTR_LIB_THROW(hipGetDeviceProperties(&prop, current_device));

  size_t read_chunks = 128 * 1024 * 1024;  // read 128MB at one time

  uint32_t *accum_location = nullptr;                // slot-size
  HCTR_LIB_THROW(hipMalloc(&accum_location, 128));  // 128 Bytes = 32 * uint32_t
  HCTR_LIB_THROW(hipMemset(accum_location, 0, 128));

  // uint32_t *culled_index_count = nullptr;
  // HCTR_LIB_THROW(hipMalloc(&culled_index_count, 128)); // 128 Bytes = 32 * uint32_t

  size_t total_file_bytes_read = 0;
  const auto time_map_start = std::chrono::high_resolution_clock::now();

  // iteration on each day's data, including training and testing.
  std::vector<std::string> all_days;
  all_days.insert(all_days.end(), train_days.begin(), train_days.end());
  all_days.insert(all_days.end(), test_days.begin(), test_days.end());

  std::vector<size_t> sample_nums;
  for (const auto &day : all_days) {
    // get file size
    std::string input_file_name = input_dir_path + "/day_" + day;
    std::ifstream binary_reader(input_file_name, std::ios::binary);
    binary_reader.seekg(0, std::ios::end);
    size_t file_size = binary_reader.tellg();
    binary_reader.close();

    // csv arguments,
    // https://docs.rapids.ai/api/libcudf/stable/structcudf_1_1io_1_1read__csv__args.html
    cudf_io::csv_reader_options in_args =
        cudf_io::csv_reader_options::builder(cudf_io::source_info{input_file_name}).header(-1);

    in_args.set_dtypes(column_dtypes);
    in_args.set_names(column_names);
    in_args.set_delimiter('\t');
    in_args.set_byte_range_size(read_chunks);  // how many bytes to read at one time.
    in_args.set_skipfooter(0);
    in_args.set_skiprows(0);
    in_args.set_use_cols_names(cat_column_names);

    int32_t total_row_nums = 0;

    int loop_count = 0;
    while (true) {
      total_file_bytes_read += in_args.get_byte_range_size();
#if defined(CUDF_VERSION_MAJOR) && CUDF_VERSION_MAJOR == 23 && defined(CUDF_VERSION_MINOR) && \
    CUDF_VERSION_MINOR >= 12
      cudf_io::table_with_metadata tbl_w_metadata =
          cudf_io::read_csv(in_args, cudf::get_default_stream(), p_mr);
#elif defined(CUDF_VERSION_MAJOR) && CUDF_VERSION_MAJOR > 23
      auto tbl_w_metadata = cudf_io::read_csv(in_args, cudf::get_default_stream(), mr);
#else
      cudf_io::table_with_metadata tbl_w_metadata = cudf_io::read_csv(in_args, p_mr);

#endif
      total_row_nums += tbl_w_metadata.tbl->num_rows();

      dim3 block(prop.maxThreadsPerBlock, 1, 1);
      dim3 grid((tbl_w_metadata.tbl->num_rows() - 1) / block.x + 1, 1, 1);

      // categorical-features
      for (unsigned int k = 0; k < cat_column_names.size(); ++k) {
        auto col = std::move(tbl_w_metadata.tbl->get_column(k));
        if (col.type().id() == cudf::type_id::STRING) {
          auto str_col = cudf::strings_column_view(col.view());
          int64_t num_strings = str_col.size();
          char *char_array = const_cast<char *>(str_col.chars().data<char>());
          int32_t *offsets = const_cast<int32_t *>(str_col.offsets().data<int32_t>());

          build_categorical_index<key_type, value_type><<<grid, block>>>(
              char_array, offsets, num_strings,
              // *categorical_col_hash_tables[cat_column_names[k]], hash_bucket,
              // &accum_location[k]);
              *categorical_col_hash_tables[cat_column_names[k]], hist_sizes[k], &accum_location[k]);

        } else if (col.type().id() == cudf::type_id::INT32) {
          key_type *data = const_cast<key_type *>(col.view().data<key_type>());
          bitmask_type *in_mask = const_cast<bitmask_type *>(col.view().null_mask());

          build_categorical_index_from_ints<key_type, value_type><<<grid, block>>>(
              data, in_mask, tbl_w_metadata.tbl->num_rows(),
              // *categorical_col_hash_tables[cat_column_names[k]], hash_bucket,
              // &accum_location[k]);
              *categorical_col_hash_tables[cat_column_names[k]], hist_sizes[k], &accum_location[k]);

        } else {
          HCTR_LOG_S(ERROR, WORLD)
              << "col.type().id() != [STRING, INT32] " << HCTR_LOCATION() << std::endl;
        }
      }

      size_t new_byte_range_offset = in_args.get_byte_range_offset() + read_chunks;
      in_args.set_byte_range_offset(new_byte_range_offset);
      if (in_args.get_byte_range_offset() >= file_size) break;

      if ((in_args.get_byte_range_offset() + read_chunks) > file_size) {
        size_t new_byte_range_size = file_size - in_args.get_byte_range_offset();
        in_args.set_byte_range_size(new_byte_range_size);
      }

      ++loop_count;

      if (loop_count == max_chunk_per_file) break;
    }
    HCTR_LOG_S(INFO, ROOT) << input_file_name << "'s total rows number = " << total_row_nums
                           << std::endl;
    sample_nums.push_back(total_row_nums);

  }  // end for all_days

  // show: slot size array
  std::vector<uint32_t> host_sz_per_fea(num_categoricals);
  HCTR_LIB_THROW(hipMemcpy(host_sz_per_fea.data(), accum_location,
                            num_categoricals * sizeof(uint32_t), hipMemcpyDeviceToHost));
  HCTR_LOG(INFO, ROOT, "Slot size array, missing value mapped to unused key: \n");
  {
    auto log = HCTR_LOG_S(INFO, WORLD);
    for (auto c : host_sz_per_fea) {
      log << (c) << ", ";
    }
    log << "\b\b" << std::endl;
  }

  const auto time_map_stop = std::chrono::high_resolution_clock::now();
  const auto time_map_build =
      std::chrono::duration_cast<std::chrono::milliseconds>(time_map_stop - time_map_start);
  HCTR_LOG_S(INFO, ROOT) << "Time used to build map: " << time_map_build.count() << " milliseconds."
                         << std::endl;

  double read_bw = double(total_file_bytes_read) / (1024.0 * 1024.0 * 1024.0);
  read_bw = (read_bw / time_map_build.count()) * 1000.f;
  HCTR_LOG_S(INFO, ROOT) << "Total bytes read: " << total_file_bytes_read
                         << " Effective Read B/W: " << read_bw << " GB/s." << std::endl;

  // HCTR_LIB_THROW(hipFree(culled_index_count));
  HCTR_LIB_THROW(hipFree(accum_location));

  // starting to do the conversion
  if (process_output) {
    uint32_t *dev_slot_size_array = nullptr;
    size_t slot_size_array_size = num_categoricals * sizeof(uint32_t);
    HCTR_LIB_THROW(hipMalloc(&dev_slot_size_array, slot_size_array_size));
    HCTR_LIB_THROW(hipMemcpy(dev_slot_size_array, host_sz_per_fea.data(), slot_size_array_size,
                              hipMemcpyHostToDevice));

    int32_t *dev_out_buffer = nullptr;
    int32_t *host_out_buffer = nullptr;

    size_t sz_output_buffer = 128 * 1024 * 1024;  // 128 MB, = read_chunks
    HCTR_LIB_THROW(hipMalloc(&dev_out_buffer, sz_output_buffer));
    HCTR_LIB_THROW(hipHostMalloc(&host_out_buffer, sz_output_buffer));

    int64_t *dev_int_col_ptrs = nullptr;
    int64_t *dev_int_col_nullmask_ptrs = nullptr;
    int64_t *dev_cat_col_nullmask_ptrs = nullptr;
    int64_t *dev_categorical_col_hash_obj = nullptr;
    int64_t *dev_char_ptrs = nullptr;
    int64_t *dev_offset_ptrs = nullptr;

    size_t sz_dev_int_col = num_numericals * sizeof(int64_t);
    size_t sz_dev_cat_hash_obj = num_categoricals * sizeof(map_type<key_type, value_type>);
    size_t sz_dev_str_ptrs = num_categoricals * sizeof(int64_t);

    HCTR_LIB_THROW(hipMalloc(&dev_int_col_ptrs, sz_dev_int_col));
    HCTR_LIB_THROW(hipMalloc(&dev_int_col_nullmask_ptrs, sz_dev_int_col));
    HCTR_LIB_THROW(hipMalloc(&dev_cat_col_nullmask_ptrs, sz_dev_str_ptrs));
    HCTR_LIB_THROW(hipMalloc(&dev_categorical_col_hash_obj, sz_dev_cat_hash_obj));
    HCTR_LIB_THROW(hipMalloc(&dev_char_ptrs, sz_dev_str_ptrs));
    HCTR_LIB_THROW(hipMalloc(&dev_offset_ptrs, sz_dev_str_ptrs));

    // encode and write out binary
    int maxbytes = 96 * 1024;  // dynamic shared memory size 96 KB
    HCTR_LIB_THROW(hipFuncSetAttribute(reinterpret_cast<const void*>(process_data_rows<key_type), value_type>,
                                        hipFuncAttributeMaxDynamicSharedMemorySize, maxbytes));

    std::vector<map_type<key_type, value_type>> categorical_col_hash_obj;
    for (auto c : cat_column_names) {
      categorical_col_hash_obj.push_back(*categorical_col_hash_tables[c]);
    }

    HCTR_LIB_THROW(hipMemcpy((void *)dev_categorical_col_hash_obj,
                              (void *)categorical_col_hash_obj.data(), sz_dev_cat_hash_obj,
                              hipMemcpyHostToDevice));

    if (process_output) {
      const auto time_convert_start = std::chrono::high_resolution_clock::now();

      std::ofstream *binary_writer = nullptr;
      if (write_out)
        binary_writer =
            new std::ofstream(std::string(output_dir_path + "/train_data.bin"), std::ios::binary);
      size_t sz_total_output_binary = 0;

      // train_data.bin
      size_t saved_samples_num = 0;
      for (size_t i = 0; i < train_days.size(); i++) {
        const auto &day = train_days[i];
        size_t needed_samples_num = 4195197692 - saved_samples_num;  // total should be 4195197692
        int32_t rows_begin_train = -1, rows_end_train = -1;          // train.txt [:36672000)
        if (needed_samples_num < sample_nums[i]) rows_end_train = needed_samples_num;

        std::string input_file_path = input_dir_path + "/day_" + day;
        sz_total_output_binary += convert_input_binaries<key_type, value_type>(
            p_mr, input_file_path, column_dtypes, column_names, hash_bucket, max_chunk_per_file, 0,
            false, dev_int_col_ptrs, dev_int_col_nullmask_ptrs, dev_cat_col_nullmask_ptrs,
            dev_categorical_col_hash_obj, dev_char_ptrs, dev_offset_ptrs, dev_out_buffer,
            host_out_buffer, binary_writer, dev_slot_size_array, rows_begin_train, rows_end_train,
            1);

        HCTR_LOG_S(INFO, ROOT) << "Processed file: " << input_file_path << " for /train_data.bin"
                               << std::endl;

        if (needed_samples_num < sample_nums[i]) {
          saved_samples_num += needed_samples_num;
          break;
        } else {
          saved_samples_num += sample_nums[i];
        }

      }  // end for train_days
      HCTR_LOG_S(INFO, ROOT) << "Size of train_data.bin: " << sz_total_output_binary << " Bytes."
                             << std::endl;
      if (binary_writer) binary_writer->close();

      // testing-data
      {
        // test_data.bin
        std::ofstream *binary_writer_test = nullptr;
        if (write_out)
          binary_writer_test =
              new std::ofstream(std::string(output_dir_path + "/test_data.bin"), std::ios::binary);

        sz_total_output_binary = 0;
        size_t saved_samples_num = 0;
        for (size_t i = 0; i < test_days.size(); ++i) {
          const auto &day = test_days[i];
          size_t needed_samples_num = 89137319 - saved_samples_num;  // total should be 89137319
          int32_t rows_begin_test = -1, rows_end_test = -1;
          if (needed_samples_num < sample_nums[train_days.size() + i])
            rows_end_test = needed_samples_num;

          // rows_begin_test = 89137318; rows_end_test = -1; // [89137318: ), second half

          std::string input_file_path = input_dir_path + "/day_" + day;
          sz_total_output_binary += convert_input_binaries<key_type, value_type>(
              p_mr, input_file_path, column_dtypes, column_names, hash_bucket, max_chunk_per_file,
              0, false, dev_int_col_ptrs, dev_int_col_nullmask_ptrs, dev_cat_col_nullmask_ptrs,
              dev_categorical_col_hash_obj, dev_char_ptrs, dev_offset_ptrs, dev_out_buffer,
              host_out_buffer, binary_writer_test, dev_slot_size_array, rows_begin_test,
              rows_end_test, 1);

          HCTR_LOG_S(INFO, ROOT) << "Processed file: " << input_file_path << " for /test_data.bin"
                                 << std::endl;

          if (needed_samples_num < sample_nums[train_days.size() + i]) {
            saved_samples_num += needed_samples_num;
            break;
          } else {
            saved_samples_num += sample_nums[train_days.size() + i];
          }

        }  // end for test_days

        HCTR_LOG_S(INFO, ROOT) << "Size of test_data.bin: " << sz_total_output_binary << " Bytes."
                               << std::endl;

        if (binary_writer_test) binary_writer_test->close();
      }

      const auto time_convert_stop = std::chrono::high_resolution_clock::now();
      const auto time_convert_total = std::chrono::duration_cast<std::chrono::milliseconds>(
          time_convert_stop - time_convert_start);
      HCTR_LOG_S(INFO, ROOT) << "Time to process binaries: " << time_convert_total.count()
                             << " milliseconds." << std::endl;
      double p_read_bw = (double)process_read_bytes / (1024.0 * 1024.0 * 1024.0);
      p_read_bw = (p_read_bw / time_convert_total.count()) * 1000.f;

      double p_write_bw = (double)process_write_bytes / (1024.0 * 1024.0 * 1024.0);
      p_write_bw = (p_write_bw / time_convert_total.count()) * 1000.f;

      size_t total_second_pass_bytes = process_read_bytes + process_write_bytes;
      double p_2nd_bw = (double)total_second_pass_bytes / (1024.0 * 1024.0 * 1024.0);
      p_2nd_bw = (p_2nd_bw / time_convert_total.count()) * 1000.f;

      HCTR_LOG_S(INFO, ROOT) << "Convert Bytes reading: " << process_read_bytes
                             << ", Effective reading B/W: " << p_read_bw << " GB/s." << std::endl;
      HCTR_LOG_S(INFO, ROOT) << "Convert Bytes writing: " << process_write_bytes
                             << ", Effective reading B/W: " << p_write_bw << " GB/s." << std::endl;
      HCTR_LOG_S(INFO, ROOT) << "Convert Bytes total: " << total_second_pass_bytes
                             << ", Effective reading B/W: " << p_2nd_bw << " GB/s." << std::endl;
    }

    const auto program_end_time = std::chrono::high_resolution_clock::now();
    const auto application_time =
        std::chrono::duration_cast<std::chrono::milliseconds>(program_end_time - time_map_start);
    double app_bw = (double)total_file_bytes_read / (1024.0 * 1024.0 * 1024.0);
    app_bw = (app_bw / application_time.count()) * 1000.f;

    HCTR_LOG_S(INFO, ROOT) << "Application process B/W: " << app_bw << " GB/s." << std::endl;

    HCTR_LIB_THROW(hipFree(dev_out_buffer));
    HCTR_LIB_THROW(hipHostFree(host_out_buffer));

    HCTR_LIB_THROW(hipFree(dev_int_col_ptrs));
    HCTR_LIB_THROW(hipFree(dev_int_col_nullmask_ptrs));
    HCTR_LIB_THROW(hipFree(dev_categorical_col_hash_obj));
    HCTR_LIB_THROW(hipFree(dev_char_ptrs));
    HCTR_LIB_THROW(hipFree(dev_offset_ptrs));
    HCTR_LIB_THROW(hipFree(dev_slot_size_array));
    HCTR_LIB_THROW(hipFree(dev_cat_col_nullmask_ptrs));
  }
  // destroy map objects
  for (auto c : categorical_col_hash_tables) c.second->destroy();

  delete p_mr;
  p_mr = nullptr;
}

int main(const int argc, const char *argv[]) {
  if (argc < 3) {
    HCTR_LOG(INFO, ROOT, "Need min 2 args: input_dir output_dir\n");
    HCTR_LOG(INFO, ROOT, "Usage for Kaggle Datasets: ./dlrm_raw input_dir output_dir\n");
    HCTR_LOG(
        INFO, ROOT,
        "Usage for TeraBytes Datasets: ./dlrm_raw input_dir output_dir --train [days for training] "
        "--test [days for testing], those days are separated with comma, no whitespace.\n");
    return -1;
  }

  const int num_numericals = 14;    // label + 13 int-dense-feature
  const int num_categoricals = 26;  // 26 int-categorical-feature

  std::string input_dir_path(argv[1]);
  std::string output_dir_path(argv[2]);

  switch (argc) {
    case 3: {
      HCTR_LOG_S(INFO, ROOT) << "Processing Kaggle datasets" << std::endl;
      HCTR_LOG_S(INFO, ROOT) << "input_dir: " << input_dir_path << std::endl;
      HCTR_LOG_S(INFO, ROOT) << "output_dir: " << output_dir_path << std::endl;

      process_kaggle_dataset(input_dir_path, output_dir_path, num_numericals, num_categoricals);
      break;
    }

    case 7: {
      if (argc == 7 &&
          (std::strcmp(argv[3], "--train") != 0 || std::strcmp(argv[5], "--test") != 0)) {
        HCTR_LOG(INFO, ROOT,
                 "Usage for TeraBytes Datasets: ./dlrm_raw input_dir output_dir --train [days for "
                 "training] "
                 "--test [days for testing], those days are separated with comma, no "
                 "whitespace.\n");
        HCTR_LOG(INFO, ROOT, "For example: ./dlrm_raw ./ ./ --train 0,1,2,3,4 --test 5,6,7\n");
        return -1;
      }

      const std::vector<std::string> train_days = split_string(std::string(argv[4]), ",");
      const std::vector<std::string> test_days = split_string(std::string(argv[6]), ",");

      HCTR_LOG_S(INFO, ROOT) << "Processing TeraBytes datasets." << std::endl;
      HCTR_LOG_S(INFO, ROOT) << "input_dir: " << input_dir_path << std::endl;
      HCTR_LOG_S(INFO, ROOT) << "output_dir: " << output_dir_path << std::endl;
      HCTR_LOG_S(INFO, ROOT) << "days for training: " << argv[4] << std::endl;
      HCTR_LOG_S(INFO, ROOT) << "days for testing: " << argv[6] << std::endl;

      process_terabyte_dataset(input_dir_path, output_dir_path, num_numericals, num_categoricals,
                               train_days, test_days);
      break;
    }

    default: {
      HCTR_LOG(INFO, ROOT, "Usage for Kaggle Datasets: ./dlrm_raw input_dir output_dir\n");
      HCTR_LOG(INFO, ROOT,
               "Usage for TeraBytes Datasets: ./dlrm_raw input_dir output_dir --train [days for "
               "training] --test [days for testing], those days are separated with comma, no "
               "whitespace.\n");
      return -1;
      break;
    }
  }

  HCTR_LOG(INFO, ROOT, "Done.\n");
  return 0;
}
