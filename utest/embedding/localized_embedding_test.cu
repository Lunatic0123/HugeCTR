/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "HugeCTR/include/data_parser.hpp"
#include "HugeCTR/include/data_reader.hpp"
//#include "HugeCTR/include/embeddings/localized_slot_sparse_embedding_hash.hpp"
#include "HugeCTR/include/embedding.hpp"
#include "utest/embedding/sparse_embedding_hash_cpu.hpp"
#include "utest/embedding/embedding_test_utils.hpp"
#include "utest/test_utils.h"
#include "gtest/gtest.h"
#include "nvToolsExt.h"
#include <sys/time.h>
#include <fstream>
#include <functional>
#include <nccl.h>

using namespace HugeCTR;
using namespace embedding_test;

namespace {

//---------------------------------------------------------------------------------------
// global params for all testing 
const std::vector<int> device_list = {0};
// const std::vector<int> device_list = {0,1};
//const std::vector<int> device_list = {0,3};
// const std::vector<int> device_list = {0,1,2,3};
// const std::vector<int> device_list = {0,1,2,3,4,5,6,7};
const int batch_num = 2;  // can not more than 32
const int batchsize = 1024;
const long long num_records = batchsize * batch_num;
const int slot_num = 26; 
const int max_nnz_per_slot = 10;
const int max_feature_num = max_nnz_per_slot * slot_num;  // max_feature_num in a sample
const long long vocabulary_size = 100;
const int embedding_vec_size = 16;
const int combiner = 0;   // 0-sum, 1-mean
const int optimizer = 2;  // 0-adam, 1-momentum_sgd, 2-nesterov
const bool global_update = true; // true-embedding table global update; fase-embedding table local update 
// const bool global_update = false;
const float scaler = 1.0f; // used in mixed precision training 
const float lr = 0.01;
const long long label_dim = 1;
const long long dense_dim = 0;
typedef long long T;

// In order to not allocate the total size of hash table on each GPU, the users need to set the
// size of max_vocabulary_size_per_gpu, which should be more than vocabulary_size/gpu_count,
// eg: 1.25x of that.
const float load_factor = 0.75;  // CAUSION: this is a very important param for performance

const int num_chunks = 1; // must be 1 for CPU and GPU results comparation 
const int num_threads = 1; // must be 1 for CPU and GPU results comparation 
const int num_files = 1;
const Check_t CHK = Check_t::Sum; // Check_t::Sum
const std::string file_list_name("sample_file_list.txt");
const std::string prefix("./data_reader_test_data/temp_dataset_");

const std::string plan_file(PROJECT_HOME_ + "utest/all2all_plan_dgx_{0}.json"); // for device_list {0} testing
// const std::string plan_file(PROJECT_HOME_ + "utest/all2all_plan_dgx_{0,1}.json"); // for device_list {0,3} testing
// const std::string plan_file(PROJECT_HOME_ + "utest/all2all_plan_dgx_{0,3}.json"); // for device_list {0,3} testing
// const std::string plan_file(PROJECT_HOME_ + "utest/all2all_plan_dgx_{0,1,2,3}.json"); // for device_list {0,3} testing
// const std::string plan_file(PROJECT_HOME_ + "utest/all2all_plan_dgx_{0,1,2,3,4,5,6,7}.json");

const char *hash_table_file_name = "localized_hash_table.bin";
bool init_hash_table = true;  // true: init hash_table and upload_to_device
                              // false: don't init hash_table or upload_to_device, just use an
                              //        empty hash_table to train

//-----------------------------------------------------------------------------------------

#if 0
TEST(localized_sparse_embedding_hash_test, forward_reorder) {
  int local_gpu_count = 4; // 4,2 pass 
  int embedding_vec_size = 4;
  int batch_size = 16; // 8,16 pass 
  int samples_per_gpu = batch_size / local_gpu_count;
  int slot_num = 10; // 8,10 pass 
  int slots_per_sample = (slot_num + local_gpu_count - 1) / local_gpu_count; 
  int size_per_gpu = batch_size * slots_per_sample * embedding_vec_size;

  float * h_src, * d_src, * h_dst, * d_dst;
  hipHostMalloc(&h_src, size_per_gpu*sizeof(float));
  hipHostMalloc(&h_dst, size_per_gpu*sizeof(float));
  hipMalloc(&d_src, size_per_gpu*sizeof(float));
  hipMalloc(&d_dst, size_per_gpu*sizeof(float));

  int stride = samples_per_gpu * slots_per_sample * embedding_vec_size;
  for(int i = 0; i < samples_per_gpu; i++) {
    int offset = i * slots_per_sample * embedding_vec_size;
    for(int j = 0; j < slot_num; j++) {
      int addr = offset + (j/local_gpu_count) * embedding_vec_size + (j%local_gpu_count) * stride;
      //printf("sample_id=%d, slot_id=%d, addr=%d\n", i, j, addr);
      for(int k = 0; k < embedding_vec_size; k++) {
        h_src[addr+k] = (float)j;
      }
    }
  }

  std::cout << "original dataset:" << std::endl;
  for(int i = 0; i < batch_size; i++) {
    for(int j = 0; j < slots_per_sample; j++) {
      for(int k = 0; k < embedding_vec_size; k++) {
        int addr = i*slots_per_sample*embedding_vec_size+j*embedding_vec_size+k;
        //std::cout << "addr[" << addr << "]=" << h_src[addr] << ", ";
        std::cout << h_src[addr] << ", ";
      }
    }
    std::cout << std::endl;
  }
  std::cout << std::endl;

  dim3 blockSize(embedding_vec_size, 1, 1);
  dim3 gridSize(batch_size/local_gpu_count, 1, 1);

  hipMemcpy(d_src, h_src, size_per_gpu * sizeof(float), hipMemcpyHostToDevice);

  forward_reorder_kernel<float><<<gridSize, blockSize>>>(batch_size,
                                                  slot_num,
                                                  embedding_vec_size,
                                                  local_gpu_count,
                                                  d_src,
                                                  d_dst);

  hipMemcpy(h_dst, d_dst, size_per_gpu * sizeof(float), hipMemcpyDeviceToHost);

  std::cout << "reodered dataset:" << std::endl;
  for(int i = 0; i < samples_per_gpu; i++) {
    std::cout << "sample " << i << ":" << std::endl;
    for(int j = 0; j < slot_num; j++) {
      for(int k = 0; k < embedding_vec_size; k++) {
        int addr = i*slot_num*embedding_vec_size+j*embedding_vec_size+k;
        std::cout << h_dst[addr] << ", ";
      }
    }
    std::cout << std::endl;
  }
  std::cout << std::endl;         

  // check results 
  bool results = true;
  for(int i = 0; i < samples_per_gpu; i++) {
    for(int j = 0; j < slot_num; j++) {
      for(int k = 0; k < embedding_vec_size; k++) {
        int addr = i*slot_num*embedding_vec_size+j*embedding_vec_size+k;
        if(!compare_element(h_dst[addr], float(j))) {
          results = false;
          j = slot_num;
          i = samples_per_gpu;
          break;
        }
      }
    }
  }

  ASSERT_EQ(results, true);
  
  hipHostFree(h_src);
  hipHostFree(h_dst);
  hipFree(d_src);
  hipFree(d_dst);
}
#endif 

#if 0
TEST(localized_sparse_embedding_hash_test, forward_all2all_reorder_single_node) {
  std::vector<int> device_list = {0,1,2,3}; // 4,8 gpus pass
  int local_gpu_count = device_list.size();
  int embedding_vec_size = 1;
  int batch_size = 8; // 8,16 pass
  int samples_per_gpu = batch_size / local_gpu_count;
  int slot_num = 10;  // 8,10 pass
  int slots_per_sample = (slot_num + local_gpu_count - 1) / local_gpu_count; 
  int size_per_gpu = batch_size * slots_per_sample * embedding_vec_size;

  std::vector<std::vector<int>> vvgpu;
  vvgpu.push_back(device_list);
  std::shared_ptr<DeviceMap> device_map(new DeviceMap(vvgpu, 0));
  std::shared_ptr<GPUResourceGroup> gpu_resource_group(new GPUResourceGroup(device_map));
  CudaDeviceContext context((*gpu_resource_group)[0]->get_device_id());
  
  SparseEmbeddingHashFunctors functors;

  std::vector<float *> h_src(local_gpu_count);
  std::vector<float *> h_mid(local_gpu_count);
  std::vector<float *> h_dst(local_gpu_count);
  for(int id = 0; id < local_gpu_count; id++) {
    hipHostMalloc(&h_src[id], size_per_gpu*sizeof(float));
    hipHostMalloc(&h_mid[id], size_per_gpu*sizeof(float));
    hipHostMalloc(&h_dst[id], size_per_gpu*sizeof(float));
  }

  Tensors<float> d_src;
  Tensors<float> d_mid;
  Tensors<float> d_dst;
  GeneralBuffers<float> buf;
  for (int i = 0; i < local_gpu_count; i++) {
    int cur_device = (*gpu_resource_group)[i]->get_device_id();
    context.set_device(cur_device);
    std::cout << "GPU " << cur_device << std::endl;

    buf.emplace_back(new GeneralBuffer<float>());

    std::vector<int> dims = {batch_size, slots_per_sample, embedding_vec_size};
    std::cout << "\tdims[" << dims[0] << " " << dims[1] << " " << dims[2] << "]" << std::endl;

    d_src.emplace_back(new Tensor<float>(dims, buf.back(), TensorFormat_t::HSW));
    d_mid.emplace_back(new Tensor<float>(dims, buf.back(), TensorFormat_t::HSW));
    d_dst.emplace_back(new Tensor<float>(dims, buf.back(), TensorFormat_t::HSW));

    buf.back()->init(cur_device);
    std::cout << "\tbuf size:" << buf.back()->get_size() << std::endl;
  }

  // init src
  for(int id = 0; id < local_gpu_count; id++) {
    for(int sample_id = 0; sample_id < batch_size; sample_id++) {
      for(int slot_id = 0; slot_id < slots_per_sample; slot_id++) {
        int index = sample_id * slots_per_sample + slot_id;
        int value = id + slot_id * local_gpu_count;
        if(value < slot_num) {
          for(int k = 0; k < embedding_vec_size; k++) {
            h_src[id][index * embedding_vec_size + k] = value;
          }
        }
      }
    }
  }

  std::cout << "original dataset:" << std::endl;
  for(int id = 0; id < local_gpu_count; id++) {
    std::cout << "gpu " << id << ": " << std::endl;
    for(int sample_id = 0; sample_id < batch_size; sample_id++) {
      std::cout << "\tsample " << sample_id << ": ";
      for(int slot_id = 0; slot_id < slots_per_sample; slot_id++) {
        int index = sample_id * slots_per_sample + slot_id;
        for(int k = 0; k < embedding_vec_size; k++) {
          std::cout << h_src[id][index * embedding_vec_size + k] << ", ";
        }
      }
      std::cout << std::endl;
    }
    std::cout << std::endl;
  }

  // memcpy from CPU to GPU
  std::cout << "memcpy from CPU to GPU:" << std::endl;
  for(int id = 0; id < local_gpu_count; id++) {
    int cur_device = (*gpu_resource_group)[id]->get_device_id();
    context.set_device(cur_device);

    hipMemcpyAsync(d_src[id]->get_ptr(), h_src[id], size_per_gpu * sizeof(float), \
      hipMemcpyHostToDevice, (*gpu_resource_group)[id]->get_stream());
  }
  functors.sync_all_gpus(gpu_resource_group, context);

  // all2all 
  using comm_handler_traits = FasterGossipComm::FasterGossipCommAll2AllTraits<float>;
  using comm_handler = FasterGossipComm::FasterGossipComm<float, comm_handler_traits>;
  std::unique_ptr<comm_handler> all2all;
  const std::string plan_file = PROJECT_HOME_ + "utest/all2all_plan.json";

  const size_t element_per_send = samples_per_gpu * slots_per_sample * embedding_vec_size;
  std::cout << "all2all init" << std::endl;
  functors.all2all_init(all2all, plan_file, element_per_send, d_src, d_mid, gpu_resource_group);
  std::cout << "all2all sync" << std::endl;
  functors.all2all_exec(all2all);

  // check results of all2all
  for(int id = 0; id < local_gpu_count; id++) {
    int cur_device = (*gpu_resource_group)[id]->get_device_id();
    context.set_device(cur_device);

    hipMemcpyAsync(h_mid[id], d_mid[id]->get_ptr(), size_per_gpu * sizeof(float), \
      hipMemcpyDeviceToHost, (*gpu_resource_group)[id]->get_stream());
  }
  functors.sync_all_gpus(gpu_resource_group, context);

  std::cout << "all2all dataset:" << std::endl;
  for(int id = 0; id < local_gpu_count; id++) {
    std::cout << "gpu " << id << ": " << std::endl;
    for(int sample_id = 0; sample_id < batch_size; sample_id++) {
      std::cout << "\t";
      for(int slot_id = 0; slot_id < slots_per_sample; slot_id++) {
        int index = sample_id * slots_per_sample + slot_id;
        for(int k = 0; k < embedding_vec_size; k++) {
          std::cout << h_mid[id][index * embedding_vec_size + k] << ", ";
        }
      }
      std::cout << std::endl;
    }
    std::cout << std::endl;
  }   

  // reorder
  std::cout << "reorder" << std::endl;
  dim3 blockSize(embedding_vec_size, 1, 1);
  dim3 gridSize(batch_size/local_gpu_count, 1, 1);
  for(int id = 0; id < local_gpu_count; id++) {
    context.set_device((*gpu_resource_group)[id]->get_device_id());
    forward_reorder_kernel<float><<<gridSize, blockSize, 0, (*gpu_resource_group)[id]->get_stream()>>>(batch_size,
                                                    slot_num,
                                                    embedding_vec_size,
                                                    local_gpu_count,
                                                    d_mid[id]->get_ptr(),
                                                    d_dst[id]->get_ptr());
  }

  // memcpy from GPU to CPU
  std::cout << "memcpy from GPU to CPU" << std::endl;
  for(int id = 0; id < local_gpu_count; id++) {
    int cur_device = (*gpu_resource_group)[id]->get_device_id();
    context.set_device(cur_device);

    hipMemcpyAsync(h_dst[id], d_dst[id]->get_ptr(), size_per_gpu * sizeof(float), \
      hipMemcpyDeviceToHost, (*gpu_resource_group)[id]->get_stream());
  }
  functors.sync_all_gpus(gpu_resource_group, context);

  std::cout << "reodered dataset:" << std::endl;
  for(int id = 0; id < local_gpu_count; id++) {
    std::cout << "gpu " << id << ": " << std::endl;
    for(int sample_id = 0; sample_id < samples_per_gpu; sample_id++) {
      std::cout << "\tsample " << id*samples_per_gpu+sample_id << ": ";
      for(int slot_id = 0; slot_id < slot_num; slot_id++) {
        int index = sample_id * slot_num + slot_id;
        for(int k = 0; k < embedding_vec_size; k++) {
          std::cout << h_dst[id][index * embedding_vec_size + k] << ", ";
        }
      }
      std::cout << std::endl;
    }
    std::cout << std::endl;
  }     

  // check results 
  bool results = true;
  for(int id = 0; id < local_gpu_count; id++) {
    for(int sample_id = 0; sample_id < samples_per_gpu; sample_id++) {
      for(int slot_id = 0; slot_id < slot_num; slot_id++) {
        int index = sample_id * slot_num + slot_id;
        for(int k = 0; k < embedding_vec_size; k++) {
          if(!compare_element(h_dst[id][index * embedding_vec_size + k], float(slot_id))) {
            results = false;
            id = local_gpu_count;
            sample_id = samples_per_gpu;
            slot_id = slot_num;
            break;
          }
        }
      }
    }
  } 

  ASSERT_EQ(results, true);
  
  for(int id = 0; id < local_gpu_count; id++) {
    hipHostFree(h_src[id]);
    hipHostFree(h_mid[id]);
    hipHostFree(h_dst[id]);
  }
}
#endif 

#if 0
// localized_sparse_embedding_hash upload_params() and download_params() testing
TEST(localized_sparse_embedding_hash_test, upload_and_download_params) {

  const SparseEmbeddingHashParams embedding_params = {
      batchsize, vocabulary_size, load_factor, embedding_vec_size, 
      max_feature_num, slot_num, 0, 0};

  int numprocs = 1, pid = 0;
  std::vector<std::vector<int>> vvgpu;
#ifdef ENABLE_MPI
  test::mpi_init();
  MPI_Comm_rank(MPI_COMM_WORLD, &pid);
  MPI_Comm_size(MPI_COMM_WORLD, &numprocs);
#endif
  for (int i = 0; i < numprocs; i++) {
    vvgpu.push_back(device_list);
  }
  std::shared_ptr<DeviceMap> device_map(new DeviceMap(vvgpu, pid));
  std::shared_ptr<GPUResourceGroup> gpu_resource_group(new GPUResourceGroup(device_map));

  if(pid == 0) {
#if 1
    // re-generate the dataset files 
    std::ifstream file(file_list_name);
    if(file.good()) {
      std::remove(file_list_name.c_str());
    }
#endif 
    // data generation: key's corresponding slot_id=(key%slot_num)
    HugeCTR::data_generation_for_localized_test<T, CHK>(file_list_name, prefix, num_files, num_records, slot_num,
        vocabulary_size, label_dim, dense_dim, max_nnz_per_slot);
  }

#ifdef ENABLE_MPI
  MPI_Barrier(MPI_COMM_WORLD);
  std::cout << "This is rank: " << pid << std::endl; 
#endif 

  //setup a data reader
  const DataReaderSparseParam param = {DataReaderSparse_t::Localized, max_nnz_per_slot*slot_num, slot_num};
  std::vector<DataReaderSparseParam> params;
  params.push_back(param);
  DataReader<T> * data_reader = new DataReader<T>(file_list_name, batchsize, label_dim, dense_dim, CHK, params, 
                            gpu_resource_group, num_chunks, num_threads);

  // define object
  // Embedding<T>* embedding = new LocalizedSlotSparseEmbeddingHash<T>(\
  //     data_reader->get_row_offsets_tensors(), data_reader->get_value_tensors(), \
  //     embedding_params, plan_file, gpu_resource_group);

  Embedding<T> *embedding = EmbeddingCreator::create_localized_sparse_embedding_hash(data_reader->get_row_offsets_tensors(),
        data_reader->get_value_tensors(),
        embedding_params, plan_file, gpu_resource_group);

  // init hash table file
  const std::string hash_table_upload("localized_hash_table_upload.bin");
  const std::string hash_table_download("localized_hash_table_download.bin");       


  if(pid == 0) {
    std::ofstream weight_stream(hash_table_upload);
    if(!weight_stream.is_open()) {
      ERROR_MESSAGE_("Error: file not open for writing");
    }
    //UnifiedDataSimulator<T> ldata_sim(0, vocabulary_size-1); // for key 
    UnifiedDataSimulator<T> ldata_sim(0, slot_num-1); // for slot_id
    UnifiedDataSimulator<float> fdata_sim(0, vocabulary_size-1); // for value
    T * p_key = (T *)malloc(vocabulary_size * sizeof(T));
    UnorderedKeyGenerator<T> unorderedKey;
    unorderedKey.fill_unique(p_key, vocabulary_size);
    // key + slot_id + value
    for(int i = 0; i < vocabulary_size; i++) {
      //T key = (T)i;
      //T key = ldata_sim.get_num(); // CAUSION: can not get correct results when testing by the case with duplicated keys
      //weight_stream.write((char *)&key, sizeof(T));
      weight_stream.write((char *)&p_key[i], sizeof(T));
      T slot_id = ldata_sim.get_num();
      weight_stream.write((char *)&slot_id, sizeof(T));
      //float val = (float)i;
      float val = fdata_sim.get_num();
      for(int j = 0; j < embedding_vec_size; j++) {
        weight_stream.write((char *)&val, sizeof(float));
      }

      // just for debug 
      // std::cout << "i=" << i << ":key=" << p_key[i] << " slot_id=" << slot_id << " val=" << val << std::endl;
    }
    weight_stream.close();
    free(p_key);
  }

#ifdef ENABLE_MPI
  MPI_Barrier(MPI_COMM_WORLD);
#endif 

  // upload data from host to device
  std::ifstream i_weight_stream(hash_table_upload);
  printf("start updaload_params_to_device()\n");
  embedding->upload_params_to_device(i_weight_stream);
  i_weight_stream.close();

#ifdef ENABLE_MPI
  MPI_Barrier(MPI_COMM_WORLD);
#endif 

  // download data from device to host
  std::ofstream o_weight_stream(hash_table_download);
  printf("start download_params_to_host()\n");
  embedding->download_params_to_host(o_weight_stream);
  o_weight_stream.close();

#ifdef ENABLE_MPI
  MPI_Barrier(MPI_COMM_WORLD);
#endif 

  // comapre the read file with the written file
  typedef struct TypeHashValue_{
  	float data[embedding_vec_size];
  } TypeHashValue;

  printf("start compare_localized_hash_table_files()\n");
  bool rtn = compare_localized_hash_table_files<T, T, TypeHashValue>(hash_table_upload, hash_table_download);
  ASSERT_EQ(true, rtn);
}
#endif 

#if 1
// localized_sparse_embedding_hash correctness testing: forward->backward->update_params
TEST(localized_sparse_embedding_hash_test, training_correctness) {

  OptHyperParams hyper_params;
  hyper_params.adam.beta1 = 0.9f;
  hyper_params.adam.beta2 = 0.999f;
  hyper_params.adam.epsilon = 1e-8f;
  hyper_params.momentum.factor = 0.9f;
  hyper_params.nesterov.mu = 0.9f;

  const OptParams opt_params = {optimizer, lr, hyper_params, global_update};

  const SparseEmbeddingHashParams embedding_params = {
      batchsize, vocabulary_size, load_factor, embedding_vec_size, 
      max_feature_num, slot_num, combiner, opt_params, scaler};

  int numprocs = 1, pid = 0;
  std::vector<std::vector<int>> vvgpu;
  test::mpi_init();
#ifdef ENABLE_MPI
  MPI_Comm_rank(MPI_COMM_WORLD, &pid);
  MPI_Comm_size(MPI_COMM_WORLD, &numprocs);
#endif

  // if there are multi-node, we assume each node has the same gpu device_list
  for (int i = 0; i < numprocs; i++) {
    vvgpu.push_back(device_list);
  }
  std::shared_ptr<DeviceMap> device_map(new DeviceMap(vvgpu, pid));
  std::shared_ptr<GPUResourceGroup> gpu_resource_group(new GPUResourceGroup(device_map));

  if(pid == 0) {
    std::cout << "rank " << pid << " is generating data" << std::endl; 
#if 1
    // re-generate the dataset files 
    std::ifstream file(file_list_name);
    if(file.good()) {
      std::remove(file_list_name.c_str());
    }
#endif
    // data generation: key's corresponding slot_id=(key%slot_num)
    HugeCTR::data_generation_for_localized_test<T, CHK>(file_list_name, prefix, num_files, num_records, slot_num,
        vocabulary_size, label_dim, dense_dim, max_nnz_per_slot);
  }

#ifdef ENABLE_MPI
  MPI_Barrier(MPI_COMM_WORLD);
  std::cout << "This is rank: " << pid << std::endl; 
#endif 

  //setup a data reader
  const DataReaderSparseParam param = {DataReaderSparse_t::Localized, max_nnz_per_slot*slot_num, slot_num};
  std::vector<DataReaderSparseParam> params;
  params.push_back(param);
  DataReader<T> * data_reader = new DataReader<T>(file_list_name, batchsize, label_dim, dense_dim, CHK, params, 
                            gpu_resource_group, num_chunks, num_threads);

  // Embedding<T> *embedding = new LocalizedSlotSparseEmbeddingHash<T>(data_reader->get_row_offsets_tensors(),
  //                                                      data_reader->get_value_tensors(),
  //                                                      embedding_params, plan_file, gpu_resource_group);


  Embedding<T> *embedding = EmbeddingCreator::create_localized_sparse_embedding_hash(
                  data_reader->get_row_offsets_tensors(),
								   data_reader->get_value_tensors(),
                   embedding_params, plan_file, gpu_resource_group);

  if (init_hash_table) {
    // generate hashtable
    if(pid == 0) {
      // init hash table file: <key, solt_id, value>
      std::ofstream weight_stream(hash_table_file_name);
      if (!weight_stream.is_open()) {
        ERROR_MESSAGE_("Error: file not open for writing");
      }
      //UnifiedDataSimulator<T> ldata_sim(0, slot_num-1); // for slot_id
      UnifiedDataSimulator<float> fdata_sim(-0.1f, 0.1f); // for value
      for (long long i = 0; i < vocabulary_size; i++) {
        T key = (T)i;
        // T key = ldata_sim.get_num();
        // CAUSION: can not set random keys here, because we need to ensure that:
        // 1) we can find keys in the data file from this hash table
        // 2) there are no repeated keys
        weight_stream.write((char *)&key, sizeof(T));
        //T slot_id = ldata_sim.get_num();
        T slot_id = key%slot_num; // CAUSION: need to dedicate the slot_id for each key for correctness verification
        weight_stream.write((char *)&slot_id, sizeof(T));
        // float val = (float)i;
        float val = 0.1f;
        //float val = fdata_sim.get_num();
        for (int j = 0; j < embedding_vec_size; j++) {
          weight_stream.write((char *)&val, sizeof(float));
        }
      }
      weight_stream.close();
    }

#ifdef ENABLE_MPI
    MPI_Barrier(MPI_COMM_WORLD);
#endif 

    // upload hash table to device
    std::ifstream i_weight_stream(hash_table_file_name);
    embedding->upload_params_to_device(i_weight_stream);
    i_weight_stream.close();
  }

  // for SparseEmbeddingCpu
  SparseEmbeddingHashCpu<T> *embedding_cpu = new SparseEmbeddingHashCpu<T>(
    batchsize, max_feature_num, vocabulary_size, embedding_vec_size, slot_num, 
    label_dim, dense_dim, CHK, num_records, combiner, optimizer, lr, 
    file_list_name, hash_table_file_name, SparseEmbedding_t::Localized, global_update);

  float *embedding_feature_from_cpu = embedding_cpu->get_forward_results();
  float *wgrad_from_cpu = embedding_cpu->get_backward_results();
  T *hash_table_key_from_cpu = embedding_cpu->get_hash_table_key_ptr();
  float *hash_table_value_from_cpu = embedding_cpu->get_hash_table_value_ptr();

  // for results check
  float *embedding_feature_from_gpu =
      (float *)malloc(batchsize * slot_num * embedding_vec_size * sizeof(float));
  float *wgrad_from_gpu = (float *)malloc(batchsize * slot_num * embedding_vec_size * sizeof(float));
  T *hash_table_key_from_gpu = (T *)malloc(vocabulary_size * sizeof(T));
  float *hash_table_value_from_gpu =
      (float *)malloc(vocabulary_size * (long long)embedding_vec_size * sizeof(float));

  typedef struct TypeHashValue_ {
    float data[embedding_vec_size];
  } TypeHashValue;

  for (int i = 0; i < batch_num; i++) {
    printf("Rank%d: Round %d start:\n", pid, i);

    // call read a batch
    printf("Rank%d: data_reader->read_a_batch_to_device()\n", pid);
    data_reader->read_a_batch_to_device();

    // GPU forward
    printf("Rank%d: embedding->forward()\n", pid);
    embedding->forward();

    // check the result of forward
    printf("Rank%d: embedding->get_forward_results()\n", pid);
    embedding->get_forward_results(embedding_feature_from_gpu);  // memcpy from GPU to CPU

    if(pid == 0) {
      // CPU forward
      printf("Rank0: embedding_cpu->forward()\n");
      embedding_cpu->forward();

      // // just for debug 
      // for(int l=0; l<10; l++) {
      //   for(int j=0; j<slot_num; j++) {
      //     for(int k=0; k<embedding_vec_size; k++) {
      //       if(k == 0) {
      //         std::cout << "  emb_fea_cpu=" << embedding_feature_from_cpu[l*slot_num*embedding_vec_size+j*embedding_vec_size+k]
      //                   << ",emb_fea_gpu=" << embedding_feature_from_gpu[l*slot_num*embedding_vec_size+j*embedding_vec_size+k]
      //                   << std::endl;
      //       }
      //     }
      //   }
      // }
      // std::cout << std::endl;

      printf("Rank0: check forward results\n");
      ASSERT_EQ(true,
                compare_embedding_feature(batchsize * slot_num * embedding_vec_size,
                                          embedding_feature_from_gpu, embedding_feature_from_cpu));
    }

#ifdef ENABLE_MPI
    MPI_Barrier(MPI_COMM_WORLD);
#endif 

    // GPU backward
    printf("Rank%d: embedding->backward()\n", pid);
    embedding->backward();

    // check the result of backward
    printf("Rank%d: embedding->get_backward_results()\n", pid);
    embedding->get_backward_results(wgrad_from_gpu, 0);

    if(pid == 0) {
      // CPU backward
      printf("Rank0: embedding_cpu->backward()\n");
      embedding_cpu->backward();

      // // just for debug 
      // for(int j = 0; j < (batchsize * slot_num * embedding_vec_size); j++) {
      //   printf("cpu:%f, gpu:%f\n", wgrad_from_cpu[j], wgrad_from_gpu[j]);
      // }

      printf("Rank0: check backward results: GPU and CPU\n");
      ASSERT_EQ(true, compare_wgrad(batchsize * slot_num * embedding_vec_size, 
                                    wgrad_from_gpu, wgrad_from_cpu));
    }

#ifdef ENABLE_MPI
  MPI_Barrier(MPI_COMM_WORLD);
#endif 

    // GPU update_params
    printf("Rank%d: embedding->update_params()\n", pid);
    embedding->update_params();

    // check the results of update params
    printf("Rank%d: embedding->get_update_params_results()\n", pid);
    embedding->get_update_params_results(hash_table_key_from_gpu,
                                  hash_table_value_from_gpu);  // memcpy from GPU to CPU

    if(pid == 0) {                 
      // CPU update_params
      printf("Rank0: embedding_cpu->update_params()\n");
      embedding_cpu->update_params();

      // // just for debug 
      // std::cout << "hash_table_key_from_gpu: " << std::endl;
      // for(int i = 0; i < (vocabulary_size+1); i++) {
      //   std::cout << hash_table_key_from_gpu[i] << ", ";
      //   if((i+1)%10 == 0) {
      //     std::cout << std::endl;
      //   }
      // }
      // std::cout << std::endl;
      // std::cout << "hash_table_key_from_cpu: " << std::endl;
      // for(int i = 0; i < (vocabulary_size+1); i++) {
      //   std::cout << hash_table_key_from_cpu[i] << ", ";
      //   if((i+1)%10 == 0) {
      //     std::cout << std::endl;
      //   }
      // }
      // std::cout << std::endl;

      printf("Rank0: check update_params results\n");
      bool rtn = compare_hash_table<T, TypeHashValue>(
          vocabulary_size, (T *)hash_table_key_from_gpu, (TypeHashValue *)hash_table_value_from_gpu,
          (T *)hash_table_key_from_cpu, (TypeHashValue *)hash_table_value_from_cpu);
      ASSERT_EQ(true, rtn);
    }

#ifdef ENABLE_MPI
  MPI_Barrier(MPI_COMM_WORLD);
#endif 

    printf("Rank%d: Round %d end:\n", pid, i);
  }

  test::mpi_finialize();

  // release resources
  free(embedding_feature_from_gpu);
  free(wgrad_from_gpu);
  free(hash_table_value_from_gpu);
  free(hash_table_key_from_gpu);
}
#endif

}
