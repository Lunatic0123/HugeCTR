#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <sys/time.h>
#include <fstream>
#include <functional>
#include <unordered_set>
#include "HugeCTR/include/data_parser.hpp"
#include "HugeCTR/include/data_reader.hpp"
#include "HugeCTR/include/embedding.hpp"
#include "gtest/gtest.h"
#include "nvToolsExt.h"
#include "utest/embedding/embedding_test_utils.hpp"
#include "utest/embedding/sparse_embedding_hash_cpu.hpp"
#include "utest/test_utils.h"

using namespace HugeCTR;
using namespace embedding_test;

namespace {
//---------------------------------------------------------------------------------------
// global params for all testing
const int train_batch_num = 10;  // can not more than 32
const int test_batch_num = 1;
const int train_batchsize = 1024;
const int test_batchsize = 2560;
const int slot_num = 26;
const int max_nnz_per_slot = 1;
const int max_feature_num = max_nnz_per_slot * slot_num;  // max_feature_num in a sample
const long long vocabulary_size = 100000;
const int embedding_vec_size = 64;
const int combiner = 0;  // 0-sum, 1-mean
const long long label_dim = 1;
const long long dense_dim = 0;
typedef long long T;

const float scaler = 1.0f;  // used in mixed precision training

// In order to not allocate the total size of hash table on each GPU, the users need to set the
// size of max_vocabulary_size_per_gpu, which should be more than vocabulary_size/gpu_count,
// eg: 1.25x of that.

const int num_chunk_threads = 1;  // must be 1 for CPU and GPU results comparation
const int num_files = 1;
const Check_t CHK = Check_t::Sum;  // Check_t::Sum
const char *train_file_list_name = "train_file_list.txt";
const char *test_file_list_name = "test_file_list.txt";
const char *prefix = "./data_reader_test_data/temp_dataset_";
const char *hash_table_file_name = "distributed_hash_table.bin";
//-----------------------------------------------------------------------------------------

template <typename TypeEmbeddingComp>
void train_and_test(const std::vector<int> &device_list, const Optimizer_t &optimizer,
                    bool global_update) {
  OptHyperParams<TypeEmbeddingComp> hyper_params;
  hyper_params.adam.beta1 = 0.9f;
  hyper_params.adam.beta2 = 0.999f;
  hyper_params.adam.epsilon = 1e-7f;
  hyper_params.momentum.factor = 0.9f;
  hyper_params.nesterov.mu = 0.9f;

  const float lr = optimizer == Optimizer_t::Adam ? 0.001f : 0.01f;

  const OptParams<TypeEmbeddingComp> opt_params = {optimizer, lr, hyper_params, global_update,
                                                   scaler};

  int numprocs = 1, pid = 0;
  std::vector<std::vector<int>> vvgpu;
  test::mpi_init();
#ifdef ENABLE_MPI
  MPI_Comm_rank(MPI_COMM_WORLD, &pid);
  MPI_Comm_size(MPI_COMM_WORLD, &numprocs);
#endif
  for (int i = 0; i < numprocs; i++) {
    vvgpu.push_back(device_list);
  }
  std::shared_ptr<DeviceMap> device_map(new DeviceMap(vvgpu, pid));
  std::shared_ptr<GPUResourceGroup> gpu_resource_group(new GPUResourceGroup(device_map));

  if (pid == 0) {
    // re-generate the dataset files
    {
      std::ifstream fs(train_file_list_name);
      if (fs.good()) {
        std::remove(train_file_list_name);
      }
    }
    {
      std::ifstream fs(test_file_list_name);
      if (fs.good()) {
        std::remove(test_file_list_name);
      }
    }
    // data generation
    HugeCTR::data_generation_for_test<T, CHK>(
        train_file_list_name, prefix, num_files, train_batch_num * train_batchsize, slot_num,
        vocabulary_size, label_dim, dense_dim, max_nnz_per_slot);
    HugeCTR::data_generation_for_test<T, CHK>(
        test_file_list_name, prefix, num_files, test_batch_num * test_batchsize, slot_num,
        vocabulary_size, label_dim, dense_dim, max_nnz_per_slot);
  }

#ifdef ENABLE_MPI
  MPI_Barrier(MPI_COMM_WORLD);
#endif

  // setup a data reader
  const DataReaderSparseParam param = {DataReaderSparse_t::Distributed, max_nnz_per_slot * slot_num,
                                       max_nnz_per_slot, slot_num};
  std::vector<DataReaderSparseParam> params;
  params.push_back(param);

  std::unique_ptr<DataReader<T>> train_data_reader(
      new DataReader<T>(train_file_list_name, train_batchsize, label_dim, dense_dim, CHK, params,
                        gpu_resource_group, num_chunk_threads));

  std::unique_ptr<DataReader<T>> test_data_reader(
      new DataReader<T>(test_file_list_name, test_batchsize, label_dim, dense_dim, CHK, params,
                        gpu_resource_group, num_chunk_threads));

  // init hash table file
  if (pid == 0) {
    std::ofstream fs(hash_table_file_name);
    if (!fs.is_open()) {
      ERROR_MESSAGE_("Error: file not open for writing");
    }
    UnifiedDataSimulator<float> fdata_sim(-0.1f, 0.1f);
    for (long long i = 0; i < vocabulary_size; i++) {
      T key = (T)i;
      // T key = ldata_sim.get_num();
      // CAUSION: can not set random keys here, because we need to ensure that:
      // 1) we can find keys in the data file from this hash table
      // 2) there are no repeated keys
      fs.write((char *)&key, sizeof(T));
      float val = fdata_sim.get_num();
      for (int j = 0; j < embedding_vec_size; j++) {
        fs.write((char *)&val, sizeof(float));
      }
    }
    fs.close();
  }

#ifdef ENABLE_MPI
  MPI_Barrier(MPI_COMM_WORLD);
#endif

  const SparseEmbeddingHashParams<TypeEmbeddingComp> embedding_params = {
      train_batchsize, test_batchsize, vocabulary_size, {},        embedding_vec_size,
      max_feature_num, slot_num,       combiner,        opt_params};

  std::unique_ptr<Embedding<T, TypeEmbeddingComp>> embedding(
      EmbeddingCreator::create_distributed_sparse_embedding_hash(
          train_data_reader->get_row_offsets_tensors(), train_data_reader->get_value_tensors(),
          train_data_reader->get_nnz_array(), test_data_reader->get_row_offsets_tensors(),
          test_data_reader->get_value_tensors(), test_data_reader->get_nnz_array(),
          embedding_params, gpu_resource_group));

  {
    // upload hash table to device
    std::ifstream fs(hash_table_file_name);
    embedding->upload_params_to_device(fs);
    fs.close();
  }

  // for SparseEmbeddingCpu
  std::unique_ptr<SparseEmbeddingHashCpu<T, TypeEmbeddingComp>> embedding_cpu(
      new SparseEmbeddingHashCpu<T, TypeEmbeddingComp>(
          train_batchsize, max_feature_num, vocabulary_size, embedding_vec_size, slot_num,
          label_dim, dense_dim, CHK, train_batch_num * train_batchsize, combiner, optimizer, lr,
          train_file_list_name, hash_table_file_name, SparseEmbedding_t::Distributed, global_update,
          scaler));

  // for results check
  std::unique_ptr<TypeEmbeddingComp[]> embedding_feature_from_gpu(
      new TypeEmbeddingComp[train_batchsize * slot_num * embedding_vec_size]);
  TypeEmbeddingComp *embedding_feature_from_cpu = embedding_cpu->get_forward_results();
  std::vector<std::unique_ptr<TypeEmbeddingComp[]>> wgrad_from_gpu(device_list.size());
  for (size_t i = 0; i < device_list.size(); i++) {
    wgrad_from_gpu[i].reset(new TypeEmbeddingComp[train_batchsize * slot_num * embedding_vec_size]);
  }
  TypeEmbeddingComp *wgrad_from_cpu = embedding_cpu->get_backward_results();
  std::unique_ptr<T[]> hash_table_key_from_gpu(new T[vocabulary_size]);
  std::unique_ptr<float[]> hash_table_value_from_gpu(
      new float[vocabulary_size * embedding_vec_size]);
  T *hash_table_key_from_cpu = embedding_cpu->get_hash_table_key_ptr();
  float *hash_table_value_from_cpu = embedding_cpu->get_hash_table_value_ptr();

  typedef struct TypeHashValue_ {
    float data[embedding_vec_size];
  } TypeHashValue;

  embedding->train();

  for (int i = 0; i < train_batch_num; i++) {
    printf("Rank%d: Round %d start training:\n", pid, i);

    // call read a batch
    printf("Rank%d: data_reader->read_a_batch_to_device()\n", pid);
    train_data_reader->read_a_batch_to_device();

    // GPU forward
    printf("Rank%d: embedding->forward()\n", pid);
    embedding->forward();

    // check the result of forward
    printf("Rank%d: embedding->get_forward_results()\n", pid);
    embedding->get_forward_results(embedding_feature_from_gpu.get());  // memcpy from GPU to CPU

    if (pid == 0) {
      // CPU forward
      printf("Rank0: embedding_cpu->forward()\n");
      embedding_cpu->forward();

      printf("Rank0: check forward results\n");
      ASSERT_EQ(true, compare_embedding_feature(train_batchsize * slot_num * embedding_vec_size,
                                                embedding_feature_from_gpu.get(),
                                                embedding_feature_from_cpu));
    }

#ifdef ENABLE_MPI
    MPI_Barrier(MPI_COMM_WORLD);
#endif

    // GPU backward
    printf("Rank%d: embedding->backward()\n", pid);
    embedding->backward();

    // check the result of backward
    printf("Rank%d: embedding->get_backward_results()\n", pid);
    embedding->get_backward_results(wgrad_from_gpu[0].get(), 0);

    if (pid == 0) {
      // CPU backward
      printf("Rank0: embedding_cpu->backward()\n");
      embedding_cpu->backward();

      printf("Rank0: check backward results: GPU and CPU\n");
      ASSERT_EQ(true, compare_wgrad(train_batchsize * slot_num * embedding_vec_size,
                                    wgrad_from_gpu[0].get(), wgrad_from_cpu));
    }

#ifdef ENABLE_MPI
    MPI_Barrier(MPI_COMM_WORLD);
#endif

    // GPU update_params
    printf("Rank%d: embedding->update_params()\n", pid);
    embedding->update_params();

    // check the results of update params
    printf("Rank%d: embedding->get_update_params_results()\n", pid);
    embedding->get_update_params_results(
        hash_table_key_from_gpu.get(),
        hash_table_value_from_gpu.get());  // memcpy from GPU to CPU

    if (pid == 0) {
      // CPU update_params
      printf("Rank0: embedding_cpu->update_params()\n");
      embedding_cpu->update_params();

      printf("Rank0: check update_params results\n");
      bool rtn = compare_hash_table(
          vocabulary_size, hash_table_key_from_gpu.get(),
          reinterpret_cast<TypeHashValue *>(hash_table_value_from_gpu.get()),
          hash_table_key_from_cpu, reinterpret_cast<TypeHashValue *>(hash_table_value_from_cpu));
      ASSERT_EQ(true, rtn);
    }

#ifdef ENABLE_MPI
    MPI_Barrier(MPI_COMM_WORLD);
#endif

    printf("Rank%d: Round %d end:\n\n", pid, i);
  }

  ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  // create new obj for eval()

  {
    std::ofstream fs(hash_table_file_name);
    embedding->download_params_to_host(fs);
    fs.close();
  }

  // for SparseEmbeddingCpu eval
  std::unique_ptr<SparseEmbeddingHashCpu<T, TypeEmbeddingComp>> test_embedding_cpu(
      new SparseEmbeddingHashCpu<T, TypeEmbeddingComp>(
          test_batchsize, max_feature_num, vocabulary_size, embedding_vec_size, slot_num, label_dim,
          dense_dim, CHK, test_batch_num * test_batchsize, combiner, optimizer, lr,
          test_file_list_name, hash_table_file_name, SparseEmbedding_t::Distributed, global_update,
          scaler));

  TypeEmbeddingComp *embedding_feature_from_cpu_eval = test_embedding_cpu->get_forward_results();

  // for results check
  std::unique_ptr<TypeEmbeddingComp[]> embedding_feature_from_gpu_eval(
      new TypeEmbeddingComp[test_batchsize * slot_num * embedding_vec_size]);

  embedding->evaluate();
  {
    // eval
    printf("\nRank%d: start eval:\n", pid);

    // call read a batch
    printf("Rank%d: data_reader_eval->read_a_batch_to_device()\n", pid);
    test_data_reader->read_a_batch_to_device();

    // GPU forward
    printf("Rank%d: embedding_eval->forward()\n", pid);
    embedding->forward();

    // check the result of forward
    printf("Rank%d: embedding_eval->get_forward_results()\n", pid);
    embedding->get_forward_results(
        embedding_feature_from_gpu_eval.get());  // memcpy from GPU to CPU

    if (pid == 0) {
      // CPU forward
      printf("Rank0: embedding_cpu_eval->forward()\n");
      test_embedding_cpu->forward();

      printf("Rank0: check forward results\n");
      ASSERT_EQ(true, compare_embedding_feature(test_batchsize * slot_num * embedding_vec_size,
                                                embedding_feature_from_gpu_eval.get(),
                                                embedding_feature_from_cpu_eval));
    }
  }

  test::mpi_finialize();
}

}  // namespace

TEST(distributed_sparse_embedding_hash_test, fp32_sgd_1gpu) {
  train_and_test<float>({0}, Optimizer_t::SGD, false);
}

TEST(distributed_sparse_embedding_hash_test, fp32_sgd_8gpu) {
  train_and_test<float>({0, 1}, Optimizer_t::SGD, false);
}

TEST(distributed_sparse_embedding_hash_test, fp32_sgd_global_update_1gpu) {
  train_and_test<float>({0}, Optimizer_t::SGD, true);
}

TEST(distributed_sparse_embedding_hash_test, fp32_sgd_global_update_8gpu) {
  train_and_test<float>({0, 1, 2, 3, 4, 5, 6, 7}, Optimizer_t::SGD, true);
}

TEST(distributed_sparse_embedding_hash_test, fp16_sgd_1gpu) {
  train_and_test<__half>({0}, Optimizer_t::SGD, false);
}

TEST(distributed_sparse_embedding_hash_test, fp16_sgd_8gpu) {
  train_and_test<__half>({0, 1, 2, 3, 4, 5, 6, 7}, Optimizer_t::SGD, false);
}

TEST(distributed_sparse_embedding_hash_test, fp16_sgd_globl_update_1gpu) {
  train_and_test<__half>({0}, Optimizer_t::SGD, true);
}

TEST(distributed_sparse_embedding_hash_test, fp16_sgd_globl_update_8gpu) {
  train_and_test<__half>({0, 1, 2, 3, 4, 5, 6, 7}, Optimizer_t::SGD, true);
}

TEST(distributed_sparse_embedding_hash_test, fp32_adam_1gpu) {
  train_and_test<float>({0}, Optimizer_t::Adam, false);
}

TEST(distributed_sparse_embedding_hash_test, fp32_adam_8gpu) {
  train_and_test<float>({0, 1, 2, 3, 4, 5, 6, 7}, Optimizer_t::Adam, false);
}

TEST(distributed_sparse_embedding_hash_test, fp32_adam_global_update_1gpu) {
  train_and_test<float>({0}, Optimizer_t::Adam, true);
}

TEST(distributed_sparse_embedding_hash_test, fp32_adam_global_update_8gpu) {
  train_and_test<float>({0, 1, 2, 3, 4, 5, 6, 7}, Optimizer_t::Adam, true);
}

TEST(distributed_sparse_embedding_hash_test, fp16_adam_1gpu) {
  train_and_test<__half>({0}, Optimizer_t::Adam, false);
}

TEST(distributed_sparse_embedding_hash_test, fp16_adam_8gpu) {
  train_and_test<__half>({0, 1, 2, 3, 4, 5, 6, 7}, Optimizer_t::Adam, false);
}

TEST(distributed_sparse_embedding_hash_test, fp16_adam_globl_update_1gpu) {
  train_and_test<__half>({0}, Optimizer_t::Adam, true);
}

TEST(distributed_sparse_embedding_hash_test, fp16_adam_globl_update_8gpu) {
  train_and_test<__half>({0, 1, 2, 3, 4, 5, 6, 7}, Optimizer_t::Adam, true);
}