#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

 #include <gtest/gtest.h>
#include "utest/model_oversubscriber/mos_test_utils.hpp"
#include "HugeCTR/include/model_oversubscriber/parameter_server.hpp"

using namespace HugeCTR;
using namespace mos_test;

namespace {

const char* prefix = "./model_oversubscriber_test_data/tmp_";
const char* file_list_name_train = "file_list_train.txt";
const char* file_list_name_eval = "file_list_eval.txt";
const char* snapshot_src_file = "distributed_snapshot_src";
const char* snapshot_dst_file = "distributed_snapshot_dst";
const char* snapshot_bkp_file_unsigned = "distributed_snapshot_unsigned";
const char* snapshot_bkp_file_longlong = "distributed_snapshot_longlong";
const char* keyset_file_name = "keyset_file.bin";

const int batchsize = 4096;
const long long label_dim = 1;
const long long dense_dim = 0;
const int slot_num = 128;
const int max_nnz_per_slot = 1;
const int max_feature_num = max_nnz_per_slot * slot_num;
const long long vocabulary_size = 100000;
const int emb_vec_size = 64;
const int combiner = 0;
const float scaler = 1.0f;
const int num_workers = 1;
const int num_files = 1;

const Check_t check = Check_t::Sum;
const Update_t update_type = Update_t::Local;

// const int batch_num_train = 10;
const int batch_num_eval = 1;

template <typename TypeKey>
void do_upload_and_download_snapshot(
    size_t batch_num_train, bool use_host_ps, bool is_distributed) {
  Embedding_t embedding_type = is_distributed ? 
                               Embedding_t::DistributedSlotSparseEmbeddingHash :
                               Embedding_t::LocalizedSlotSparseEmbeddingHash;
  // create a resource manager for a single GPU
  std::vector<std::vector<int>> vvgpu;
  vvgpu.push_back({0});
  const auto resource_manager = ResourceManager::create(vvgpu, 0);

  generate_sparse_model<TypeKey, check>(snapshot_src_file, snapshot_dst_file,
      snapshot_bkp_file_unsigned, snapshot_bkp_file_longlong,
      file_list_name_train, file_list_name_eval, prefix, num_files, label_dim,
      dense_dim, slot_num, max_nnz_per_slot, max_feature_num,
      vocabulary_size, emb_vec_size, combiner, scaler, num_workers, batchsize,
      batch_num_train, batch_num_eval, update_type, resource_manager);
  copy_sparse_model(snapshot_src_file, snapshot_dst_file);

  auto get_ext_file = [](const std::string& sparse_model_file, std::string ext) {
    return std::string(sparse_model_file) + "/" + ext;
  };

  // Create a ParameterServer
  ParameterServer<TypeKey> parameter_server(use_host_ps, snapshot_dst_file,
      embedding_type, emb_vec_size, resource_manager);

  // Make a synthetic keyset files
  {
    size_t key_file_size_in_byte =
        fs::file_size(get_ext_file(snapshot_dst_file, "key"));
    size_t num_keys = key_file_size_in_byte / sizeof(long long);
    std::vector<long long> keys_in_file(num_keys);
    std::ifstream key_ifs(get_ext_file(snapshot_dst_file, "key"));
    key_ifs.read(reinterpret_cast<char *>(keys_in_file.data()),
                                          key_file_size_in_byte);
    TypeKey *key_ptr = nullptr;
    std::vector<TypeKey> key_vec;
    if (std::is_same<TypeKey, long long>::value) {
      key_ptr = reinterpret_cast<TypeKey*>(keys_in_file.data());
    } else {
      key_vec.resize(num_keys);
      std::transform(keys_in_file.begin(), keys_in_file.end(), key_vec.begin(),
                     [](long long key) { return static_cast<unsigned>(key); });
      key_ptr = key_vec.data();
    }
    std::ofstream key_ofs(keyset_file_name, std::ofstream::binary |
                                            std::ofstream::trunc);
    key_ofs.write(reinterpret_cast<char *>(key_ptr), num_keys * sizeof(TypeKey));
  }

  BufferBag buf_bag;
  {
    std::shared_ptr<GeneralBuffer2<CudaHostAllocator>> blobs_buff =
      GeneralBuffer2<CudaHostAllocator>::create();

    Tensor2<TypeKey> tensor_keys;
    Tensor2<size_t> tensor_slot_id;
    blobs_buff->reserve({vocabulary_size}, &tensor_keys);
    blobs_buff->reserve({vocabulary_size}, &tensor_slot_id);

    blobs_buff->reserve({vocabulary_size, emb_vec_size}, &(buf_bag.embedding));
    blobs_buff->allocate();

    buf_bag.keys = tensor_keys.shrink();
    buf_bag.slot_id = tensor_slot_id.shrink();
  }

  Timer timer_ps;
  timer_ps.start();

  parameter_server.load_keyset_from_file(keyset_file_name);

  size_t size_tmp = 0;
  parameter_server.pull(buf_bag, size_tmp);
  parameter_server.push(buf_bag, size_tmp);
  parameter_server.flush_emb_tbl_to_ssd();

  MESSAGE_("Batch_num=" + std::to_string(batch_num_train) +
           ", embedding_vec_size=" + std::to_string(emb_vec_size) +
           ", elapsed time=" + std::to_string(timer_ps.elapsedSeconds()) + "s");

  // Check if the result is correct
  ASSERT_TRUE(check_vector_equality(snapshot_src_file, snapshot_dst_file, "key"));
  ASSERT_TRUE(check_vector_equality(snapshot_src_file, snapshot_dst_file, "emb_vector"));
  if (!is_distributed)
    ASSERT_TRUE(check_vector_equality(snapshot_src_file, snapshot_dst_file, "slot_id"));
}

TEST(parameter_server_test, long_long_ssd_distributed) {
  do_upload_and_download_snapshot<long long>(30, false, true);
}

TEST(parameter_server_test, unsigned_host_distributed) {
  do_upload_and_download_snapshot<unsigned>(20, true, true);
}

TEST(parameter_server_test, long_long_ssd_localized) {
  do_upload_and_download_snapshot<long long>(30, false, false);
}

TEST(parameter_server_test, unsigned_host_localized) {
  do_upload_and_download_snapshot<unsigned>(20, true, false);
}

}  // namespace
